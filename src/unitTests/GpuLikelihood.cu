#include "hip/hip_runtime.h"
/*
 * GpuLikelihood.cpp
 *
 *  Created on: Feb 13, 2012
 *      Author: stsiab
 */
#include <stdexcept>
#include <string>
#include <iostream>
#include <sstream>
#include <vector>
#include <utility>
#include <cmath>
#include <math_functions.h>
#include <sys/time.h>

#include "GpuLikelihood.hpp"

// Constants
const float UNITY = 1.0;
const float ZERO = 0.0;

inline
double
timeinseconds(const timeval a, const timeval b)
{
  timeval result;
  timersub(&b, &a, &result);
  return result.tv_sec + result.tv_usec / 1000000.0;
}

class GpuRuntimeError : public std::exception
{
public:
  GpuRuntimeError(const std::string usrMsg, hipError_t cudaErr)
  {
    msg_ = "GPU Runtime Error: ";
    msg_ += usrMsg;
    msg_ += " (";
    msg_ += cudaErr;
    msg_ += ",";
    msg_ += hipGetErrorString(cudaErr);
    msg_ += ")";
  }
  ~GpuRuntimeError() throw ()
  {
  }
  ;
  const char*
  what() const throw ()
  {
    return msg_.c_str();
  }

private:
  std::string msg_;
};

#define checkCudaError(err)  __checkCudaError(err, __FILE__, __LINE__)

void
__checkCudaError(const hipError_t err, const char* file, const int line)
{
  if (err != hipSuccess)
    {
      std::stringstream s;
      s << file << "(" << line << ") : Cuda Runtime error ";
      throw GpuRuntimeError(s.str(), err);
    }
}

// CUDA kernels

template<typename T>
  struct Log
  {
    __host__ __device__
    T
    operator()(const T& val) const
    {
      return logf(val);
    }
  };



__global__ void
_calcIntegral(const int infecSize, int* DRowPtr, int* DColInd, float* D,
    float* eventTimes, const int eventTimesPitch, const float* susceptibility,
    const float* infectivity, const float gamma2, const float delta,
    const float obsTime, float* output)
{
  // Each warp calculates a row i of the sparse matrix

  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int row = tid / 32; // Global Warp id
  int lwid = threadIdx.x / 32; // Local warp ID
  int lane = tid & (32 - 1); // Id within a warp

  __shared__
  float buff[THREADSPERBLOCK];

  buff[threadIdx.x] = 0.0;

  if (row < infecSize)
    {

      int begin = DRowPtr[row];
      int end = DRowPtr[row+1];
      float Ii = eventTimes[row];
      float Ni = eventTimes[row + eventTimesPitch];
      float Ri = eventTimes[row + eventTimesPitch*2];

      float threadSum = 0.0f;
      for (int jj = begin + lane; jj < end; jj += 32)
        {
          // Integrated infection pressure
          float Ij = eventTimes[DColInd[jj]];
          float Nj = eventTimes[DColInd[jj] + eventTimesPitch];
          float jMaxSuscep;
          jMaxSuscep = fminf(Nj, Ij);
          jMaxSuscep = fminf(jMaxSuscep, obsTime);
          float betaij = fminf(Ni, jMaxSuscep)
              - fminf(Ii, jMaxSuscep);
          betaij += gamma2
              * (fminf(Ri, jMaxSuscep) - fminf(Ni, jMaxSuscep));

          // Apply distance kernel and suscep
          betaij *= delta / (delta * delta + D[jj]);
          betaij *= susceptibility[DColInd[jj]];
          threadSum += betaij;
        }
      buff[threadIdx.x] = threadSum*infectivity[row];
    }
  __syncthreads();

  // Reduce all warp sums and write to global memory.
  for (unsigned int size = blockDim.x/2; size > 32; size >>= 1)
    {
      if (threadIdx.x < size)
        buff[threadIdx.x] += buff[threadIdx.x + size];
      __syncthreads();
    }
  if (threadIdx.x < 32) {
      volatile float* vbuff = buff;
      vbuff[threadIdx.x] += vbuff[threadIdx.x + 32];
      vbuff[threadIdx.x] += vbuff[threadIdx.x + 16];
      vbuff[threadIdx.x] += vbuff[threadIdx.x +  8];
      vbuff[threadIdx.x] += vbuff[threadIdx.x +  4];
      vbuff[threadIdx.x] += vbuff[threadIdx.x +  2];
      vbuff[threadIdx.x] += vbuff[threadIdx.x +  1];
  }


  if (threadIdx.x == 0)
    {
      output[blockIdx.x] = buff[0];
    }
}


__global__ void
_calcProduct(const int infecSize, const int* DRowPtr,
    const int* DColInd, float* D, const float* eventTimes,
    const int eventTimesPitch, const float* susceptibility,
    const float* infectivity, const float epsilon,
    const float gamma1, const float gamma2, const float delta, float* prodCache)
{
  // Each warp calculates a row of the sparse matrix

  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int gwid = tid / 32; // Global Warp id
  int lane = tid & (32 - 1); // Id within a warp

  __shared__ float threadProdCache[THREADSPERBLOCK];
  threadProdCache[threadIdx.x] = 0.0f;
  int row = gwid;

  if (row < infecSize)
    {
      //threadProdCache[threadIdx.x] = 0.0f;
      int begin = DRowPtr[row];
      int end = DRowPtr[row + 1];

      float Ij = eventTimes[row];


      for (int ii = begin + lane; ii < end and DColInd[ii] < infecSize; ii += 32)
        {
          int i = DColInd[ii];
          float Ii = eventTimes[i];
          float Ni = eventTimes[eventTimesPitch + i];
          float Ri = eventTimes[eventTimesPitch * 2 + i];

          if (Ii < Ij and Ij <= Ni)
            threadProdCache[threadIdx.x] += infectivity[i]*delta / (delta*delta + D[ii]);
          else if (Ni < Ij and Ij <= Ri)
            threadProdCache[threadIdx.x] += gamma2*infectivity[i]*delta / (delta*delta + D[ii]);
          else
            threadProdCache[threadIdx.x] += 0.0f;
        }
      __syncthreads();

      // Reduce semi-products into productCache
      volatile float* vThreadProdCache = threadProdCache;
      if(lane < 16)
        {
          vThreadProdCache[threadIdx.x] += vThreadProdCache[threadIdx.x + 16];
          vThreadProdCache[threadIdx.x] += vThreadProdCache[threadIdx.x +  8];
          vThreadProdCache[threadIdx.x] += vThreadProdCache[threadIdx.x +  4];
          vThreadProdCache[threadIdx.x] += vThreadProdCache[threadIdx.x +  2];
          vThreadProdCache[threadIdx.x] += vThreadProdCache[threadIdx.x +  1];
        }
      __syncthreads();

      // Write out to global memory -- we're going to get a bank conflict here!
      if(lane == 0) prodCache[row] = threadProdCache[threadIdx.x]*susceptibility[row]*gamma1 + epsilon;
    }
}


__global__ void
calcSpecPow(const int size, const int nSpecies, float* specpow,
    const int specpowPitch, const float* animals, const int animalsPitch,
    const float* powers)
{
  int row = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < size)
    {
      for (unsigned int col = 0; col < nSpecies; ++col)
        {
          specpow[col * specpowPitch + row] = powf(
              animals[col * animalsPitch + row], powers[col]);
        }
    }
}

__global__ void
_updateInfectionTimeInteg(const int idx, const float newTime, int* DRowPtr, int* DColInd, float* D,
    float* eventTimes, const int eventTimesPitch, const float* susceptibility,
    const float* infectivity, const float gamma2, const float delta,
    const float obsTime)
{

}

GpuLikelihood::GpuLikelihood(const size_t realPopSize, const size_t popSize,
    const size_t numInfecs, const size_t nSpecies, const float obsTime,
    const size_t distanceNNZ) :
    realPopSize_(realPopSize), popSize_(popSize), numInfecs_(numInfecs), numSpecies_(
        nSpecies), obsTime_(obsTime), I1Time_(0.0), I1Idx_(0), sumI_(0), bgIntegral_(
        0.0), covariateCopies_(0), devAnimals_(NULL), animalsPitch_(0), devAnimalsInfPow_(
        NULL), devAnimalsSuscPow_(NULL), devEventTimes_(NULL), devSusceptibility_(
        NULL), devInfectivity_(NULL), devDVal_(NULL), devDRowPtr_(NULL), devDColInd_(
        NULL), dnnz_(distanceNNZ), epsilon_(
        0.0f), gamma1_(0.0f), gamma2_(0.0f), devXi_(NULL), devPsi_(NULL), devZeta_(
        NULL), devPhi_(NULL), delta_(0.0f)
{

  // Allocate Animals_
  checkCudaError(
      hipMallocPitch(&devAnimals_, &animalsPitch_, popSize_ * sizeof(float), numSpecies_));
  animalsPitch_ /= sizeof(float);
  checkCudaError(
      hipMallocPitch(&devAnimalsSuscPow_, &animalsSuscPowPitch_, popSize_ * sizeof(float), numSpecies_));
  animalsSuscPowPitch_ /= sizeof(float);
  checkCudaError(
      hipMallocPitch(&devAnimalsInfPow_, &animalsInfPowPitch_, numInfecs_ * sizeof(float), numSpecies_));
  animalsInfPowPitch_ /= sizeof(float);

  // Allocate Distance_ CRS matrix
  checkCudaError(hipMalloc(&devDVal_, dnnz_ * sizeof(float)));
  checkCudaError(hipMalloc(&devDRowPtr_, (popSize_ + 1) * sizeof(int)));
  checkCudaError(hipMalloc(&devDColInd_, dnnz_ * sizeof(float)));
  hostDRowPtr_ = new int[popSize_ + 1];

  // Set up reference counter to covariate data
  covariateCopies_ = new size_t;
  *covariateCopies_ = 1;

  // Allocate Event times - popSize_ * NUMEVENTS matrix
  checkCudaError(
      hipMallocPitch(&devEventTimes_, &eventTimesPitch_, popSize_ * sizeof(float), NUMEVENTS));
  eventTimesPitch_ /= sizeof(float);

  // Allocate intermediate infectivity and susceptibility
  checkCudaError(hipMalloc(&devSusceptibility_, popSize_ * sizeof(float)));
  checkCudaError(hipMalloc(&devInfectivity_, numInfecs_ * sizeof(float)));

  // Allocate product cache
  checkCudaError(hipMalloc(&devProduct_, numInfecs_ * sizeof(float)));

  // Allocate integral array
  int numRequiredThreads = numInfecs_*32; // One warp per infection
  integralBuffSize_ = (numRequiredThreads + THREADSPERBLOCK - 1) / THREADSPERBLOCK;
  checkCudaError(hipMalloc(&devIntegral_, integralBuffSize_*sizeof(float)));

  // Parameters
  checkCudaError(hipMalloc(&devXi_, numSpecies_ * sizeof(float)));
  checkCudaError(hipMalloc(&devPsi_, numSpecies_ * sizeof(float)));
  checkCudaError(hipMalloc(&devZeta_, numSpecies_ * sizeof(float)));
  checkCudaError(hipMalloc(&devPhi_, numSpecies_ * sizeof(float)));

  // BLAS handles
  blasStat_ = hipblasCreate(&cudaBLAS_);
  if (blasStat_ != HIPBLAS_STATUS_SUCCESS)
    throw std::runtime_error("CUBLAS init failed");

  sparseStat_ = hipsparseCreate(&cudaSparse_);
  if (sparseStat_ != HIPSPARSE_STATUS_SUCCESS)
    throw std::runtime_error("CUSPARSE init failed");

  sparseStat_ = hipsparseCreateMatDescr(&crsDescr_);
  if (sparseStat_ != HIPSPARSE_STATUS_SUCCESS)
    throw std::runtime_error("CUSPARSE matrix descriptor init failed");
  hipsparseSetMatType(crsDescr_, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(crsDescr_, HIPSPARSE_INDEX_BASE_ZERO);

}

// Copy constructor
GpuLikelihood::GpuLikelihood(const GpuLikelihood& other) :
    realPopSize_(other.realPopSize_), popSize_(other.popSize_), numInfecs_(
        other.numInfecs_), numSpecies_(other.numSpecies_), obsTime_(
        other.obsTime_), I1Time_(other.I1Time_), I1Idx_(other.I1Idx_), sumI_(
        other.sumI_), bgIntegral_(other.bgIntegral_), lp_(other.lp_), covariateCopies_(
        other.covariateCopies_), devAnimals_(other.devAnimals_), animalsPitch_(
        other.animalsPitch_), devDVal_(other.devDVal_), devDRowPtr_(
        other.devDRowPtr_), devDColInd_(other.devDColInd_), dnnz_(other.dnnz_),
        integralBuffSize_(other.integralBuffSize_), epsilon_(
        other.epsilon_), gamma1_(other.gamma1_), gamma2_(other.gamma2_), delta_(
        other.delta_)
{
  timeval start, end;
  gettimeofday(&start, NULL);
  // Allocate Animals_
  checkCudaError(
      hipMallocPitch(&devAnimalsInfPow_, &animalsInfPowPitch_, numInfecs_ * sizeof(float), numSpecies_));
  animalsInfPowPitch_ /= sizeof(float);
  checkCudaError(
      hipMemcpy2D(devAnimalsInfPow_,animalsInfPowPitch_*sizeof(float),other.devAnimalsInfPow_,other.animalsInfPowPitch_*sizeof(float),numInfecs_*sizeof(float),numSpecies_,hipMemcpyDeviceToDevice));

  checkCudaError(
      hipMallocPitch(&devAnimalsSuscPow_, &animalsSuscPowPitch_, popSize_ * sizeof(float), numSpecies_));
  animalsSuscPowPitch_ /= sizeof(float);
  checkCudaError(
      hipMemcpy2D(devAnimalsSuscPow_,animalsSuscPowPitch_*sizeof(float),other.devAnimalsSuscPow_,other.animalsSuscPowPitch_*sizeof(float),popSize_*sizeof(float),numSpecies_,hipMemcpyDeviceToDevice));

  // Allocate and copy event times - popSize_ * NUMEVENTS matrix
  checkCudaError(
      hipMallocPitch(&devEventTimes_, &eventTimesPitch_, popSize_ * sizeof(float), NUMEVENTS));
  eventTimesPitch_ /= sizeof(float);
  checkCudaError(
      hipMemcpy2D(devEventTimes_,eventTimesPitch_*sizeof(float),other.devEventTimes_,other.eventTimesPitch_*sizeof(float),numInfecs_*sizeof(float), NUMEVENTS, hipMemcpyDeviceToDevice));

  // Allocate and copy intermediate infectivity and susceptibility
  checkCudaError(hipMalloc(&devSusceptibility_, popSize_ * sizeof(float)));
  checkCudaError(
      hipMemcpy(devSusceptibility_, other.devSusceptibility_, popSize_ * sizeof(float),hipMemcpyDeviceToDevice));
  checkCudaError(hipMalloc(&devInfectivity_, numInfecs_ * sizeof(float)));
  checkCudaError(
      hipMemcpy(devInfectivity_, other.devInfectivity_, numInfecs_ * sizeof(float), hipMemcpyDeviceToDevice));

  // Allocate and copy product vector
  checkCudaError(hipMalloc(&devProduct_, numInfecs_ * sizeof(float)));
  checkCudaError(
      hipMemcpy(devProduct_, other.devProduct_, numInfecs_ * sizeof(float), hipMemcpyDeviceToDevice));

  // Allocate integral array
  checkCudaError(hipMalloc(&devIntegral_, integralBuffSize_*sizeof(float)));


  // Parameters -- Allocate and Copy
  checkCudaError(hipMalloc(&devXi_, numSpecies_ * sizeof(float)));
  checkCudaError(
      hipMemcpy(devXi_, other.devXi_, numSpecies_ * sizeof(float), hipMemcpyDeviceToDevice));
  checkCudaError(hipMalloc(&devPsi_, numSpecies_ * sizeof(float)));
  checkCudaError(
      hipMemcpy(devPsi_, other.devPsi_, numSpecies_ * sizeof(float), hipMemcpyDeviceToDevice));
  checkCudaError(hipMalloc(&devZeta_, numSpecies_ * sizeof(float)));
  checkCudaError(
      hipMemcpy(devZeta_, other.devZeta_, numSpecies_ * sizeof(float), hipMemcpyDeviceToDevice));
  checkCudaError(hipMalloc(&devPhi_, numSpecies_ * sizeof(float)));
  checkCudaError(
      hipMemcpy(devPhi_, other.devPhi_, numSpecies_ * sizeof(float), hipMemcpyDeviceToDevice));

  // BLAS handles
  cudaBLAS_ = other.cudaBLAS_;
  cudaSparse_ = other.cudaSparse_;
  crsDescr_ = other.crsDescr_;

  ++*covariateCopies_; // Increment copies of covariate data

  gettimeofday(&end, NULL);
  std::cerr << "Time (" << __PRETTY_FUNCTION__ << "): "
      << timeinseconds(start, end) << std::endl;

}

// Assignment constructor
const GpuLikelihood&
GpuLikelihood::operator=(const GpuLikelihood& other)
{
  timeval start, end;
  gettimeofday(&start, NULL);
  // Copy animal powers
  checkCudaError(
      hipMemcpy2DAsync(devAnimalsInfPow_,animalsInfPowPitch_*sizeof(float),other.devAnimalsInfPow_,other.animalsInfPowPitch_*sizeof(float),numInfecs_*sizeof(float),numSpecies_,hipMemcpyDeviceToDevice));
  checkCudaError(
      hipMemcpy2DAsync(devAnimalsSuscPow_,animalsSuscPowPitch_*sizeof(float),other.devAnimalsSuscPow_,other.animalsSuscPowPitch_*sizeof(float),popSize_*sizeof(float),numSpecies_,hipMemcpyDeviceToDevice));

  // copy event times
  checkCudaError(
      hipMemcpy2DAsync(devEventTimes_,eventTimesPitch_*sizeof(float),other.devEventTimes_,other.eventTimesPitch_*sizeof(float),numInfecs_*sizeof(float), NUMEVENTS, hipMemcpyDeviceToDevice));

  // copy intermediate infectivity and susceptibility
  checkCudaError(
      hipMemcpyAsync(devSusceptibility_, other.devSusceptibility_, popSize_ * sizeof(float),hipMemcpyDeviceToDevice));
  checkCudaError(
      hipMemcpyAsync(devInfectivity_, other.devInfectivity_, numInfecs_ * sizeof(float), hipMemcpyDeviceToDevice));

  // copy product vector
  checkCudaError(
      hipMemcpyAsync(devProduct_, other.devProduct_, numInfecs_ * sizeof(float), hipMemcpyDeviceToDevice));

  // Device Parameters Copy
  checkCudaError(
      hipMemcpyAsync(devXi_, other.devXi_, numSpecies_ * sizeof(float), hipMemcpyDeviceToDevice));
  checkCudaError(
      hipMemcpyAsync(devPsi_, other.devPsi_, numSpecies_ * sizeof(float), hipMemcpyDeviceToDevice));
  checkCudaError(
      hipMemcpyAsync(devZeta_, other.devZeta_, numSpecies_ * sizeof(float), hipMemcpyDeviceToDevice));
  checkCudaError(
      hipMemcpyAsync(devPhi_, other.devPhi_, numSpecies_ * sizeof(float), hipMemcpyDeviceToDevice));

  // Host Parameters Copy
  epsilon_ = other.epsilon_;
  gamma1_ = other.gamma1_;
  gamma2_ = other.gamma2_;
  delta_ = other.delta_;

  // Likelihood components
  integral_ = other.integral_;
  bgIntegral_ = other.bgIntegral_;
  lp_ = other.lp_;

  gettimeofday(&end, NULL);
  std::cerr << "Time (" << __PRETTY_FUNCTION__ << "): "
      << timeinseconds(start, end) << std::endl;

  return *this;
}

GpuLikelihood::~GpuLikelihood()
{
  if (*covariateCopies_ == 1) // We're the last copy to be destroyed
    {
      hipFree(devAnimals_);
      hipFree(devDVal_);
      hipFree(devDRowPtr_);
      hipFree(devDColInd_);
      delete[] hostDRowPtr_;
      hipblasDestroy(cudaBLAS_);
      hipsparseDestroy(cudaSparse_);

      delete covariateCopies_;
    }

  if (devEventTimes_)
    hipFree(devEventTimes_);
  if (devAnimalsSuscPow_)
    hipFree(devAnimalsSuscPow_);
  if (devAnimalsInfPow_)
    hipFree(devAnimalsInfPow_);
  if (devSusceptibility_)
    hipFree(devSusceptibility_);
  if (devInfectivity_)
    hipFree(devInfectivity_);
  if (devProduct_)
    hipFree(devProduct_);
  if (devIntegral_)
    hipFree(devIntegral_);

  if (devXi_)
    hipFree(devXi_);
  if (devPsi_)
    hipFree(devPsi_);
  if (devZeta_)
    hipFree(devZeta_);
  if (devPhi_)
    hipFree(devPhi_);

}

void
GpuLikelihood::SetEvents(const float* data)
{
  // Get event times into GPU memory
  hipError_t rv = hipMemcpy2D(devEventTimes_,
      eventTimesPitch_ * sizeof(float), data, popSize_ * sizeof(float),
      popSize_ * sizeof(float), NUMEVENTS, hipMemcpyHostToDevice);
  if (rv != hipSuccess)
    throw GpuRuntimeError("Copying event times to device failed", rv);


}

void
GpuLikelihood::SetSpecies(const float* data)
{
  // Loads species data assuming **COL MAJOR**
  hipError_t rv = hipMemcpy2D(devAnimals_, animalsPitch_ * sizeof(float),
      data, popSize_ * sizeof(float), popSize_ * sizeof(float), numSpecies_,
      hipMemcpyHostToDevice);
  if (rv != hipSuccess)
    throw GpuRuntimeError("Failed copying species data to device", rv);

  CalcInfectivity();
  CalcSusceptibility();
}

void
GpuLikelihood::SetDistance(const float* data, const int* rowptr,
    const int* colind)
{
  // Loads distance data into memory
  checkCudaError(hipMemcpy(devDVal_, data, dnnz_ * sizeof(float),
      hipMemcpyHostToDevice));
  checkCudaError(hipMemcpy(devDRowPtr_, rowptr, (popSize_ + 1) * sizeof(int),
      hipMemcpyHostToDevice));
  checkCudaError(hipMemcpy(devDColInd_, colind, dnnz_ * sizeof(int),
      hipMemcpyHostToDevice));
}

void
GpuLikelihood::SetParameters(float* epsilon, float* gamma1, float* gamma2,
    float* xi, float* psi, float* zeta, float* phi, float* delta)
{
  epsilon_ = *epsilon;
  gamma1_ = *gamma1;
  gamma2_ = *gamma2;
  delta_ = *delta;

  checkCudaError(hipMemcpy(devXi_, xi, numSpecies_ * sizeof(float),
      hipMemcpyHostToDevice));

  checkCudaError(hipMemcpy(devPsi_, psi, numSpecies_ * sizeof(float),
      hipMemcpyHostToDevice));

  checkCudaError(hipMemcpy(devZeta_, zeta, numSpecies_ * sizeof(float),
      hipMemcpyHostToDevice));

  checkCudaError(hipMemcpy(devPhi_, phi, numSpecies_ * sizeof(float),
      hipMemcpyHostToDevice));
}


inline
void
GpuLikelihood::CalcInfectivityPow()
{
  int dimBlock(THREADSPERBLOCK);
  int dimGrid((numInfecs_ + THREADSPERBLOCK - 1) / THREADSPERBLOCK);
calcSpecPow<<<dimGrid, dimBlock>>>(numInfecs_,numSpecies_,devAnimalsInfPow_, animalsInfPowPitch_,devAnimals_,animalsPitch_,devPsi_);
          checkCudaError(hipGetLastError());
}

inline
void
GpuLikelihood::CalcInfectivity()
{

  // Now calculate infectivity
  blasStat_ = hipblasSgemv(cudaBLAS_, HIPBLAS_OP_N, numInfecs_, numSpecies_,
      &UNITY, devAnimalsInfPow_, animalsInfPowPitch_, devXi_, 1, &ZERO,
      devInfectivity_, 1);
  if (blasStat_ != HIPBLAS_STATUS_SUCCESS)
    {
      std::cerr << "Error in infectivity: " << blasStat_ << std::endl;
    }

}

inline
void
GpuLikelihood::CalcSusceptibilityPow()
{
  int dimBlock(THREADSPERBLOCK);
  int dimGrid((popSize_ + THREADSPERBLOCK - 1) / THREADSPERBLOCK);
calcSpecPow<<<dimGrid, dimBlock>>>(popSize_,numSpecies_,devAnimalsSuscPow_,animalsSuscPowPitch_, devAnimals_,animalsPitch_,devPhi_);
          checkCudaError(hipGetLastError());
}

inline
void
GpuLikelihood::CalcSusceptibility()
{
  // Calculates susceptibility powers and sums over suscept.
  blasStat_ = hipblasSgemv(cudaBLAS_, HIPBLAS_OP_N, popSize_, numSpecies_, &UNITY,
      devAnimalsSuscPow_, animalsSuscPowPitch_, devZeta_, 1, &ZERO,
      devSusceptibility_, 1);
  if (blasStat_ != HIPBLAS_STATUS_SUCCESS)
    {
      std::cerr << "Error in susceptibility: " << blasStat_ << std::endl;
    }
}


void
GpuLikelihood::CalcBgIntegral()
{
  // Get I1Time

  thrust::device_ptr<float> v(devEventTimes_); // REQUIRES COL MAJOR!!
  thrust::device_ptr<float> myMin = thrust::min_element(v, v + numInfecs_);
  I1Idx_ = myMin - v;
  I1Time_ = *myMin;
  sumI_ = thrust::reduce(v, v + numInfecs_, 0.0f, thrust::plus<float>());

  bgIntegral_ = sumI_ - I1Time_ * numInfecs_;
  bgIntegral_ += (obsTime_ - I1Time_) * (realPopSize_ - numInfecs_);
  bgIntegral_ *= epsilon_;
}

inline
void
GpuLikelihood::CalcProduct()
{

  _calcProduct<<<integralBuffSize_,THREADSPERBLOCK>>>(numInfecs_,devDRowPtr_,devDColInd_,devDVal_,
      devEventTimes_,eventTimesPitch_,devSusceptibility_,devInfectivity_,epsilon_,gamma1_,gamma2_,delta_,devProduct_);

  thrust::device_ptr<float> prodPtr(devProduct_);
  prodPtr[I1Idx_] = 1.0;
  lp_ = thrust::transform_reduce(prodPtr, prodPtr+numInfecs_, Log<float>(), 0.0f,thrust::plus<float>());
}


inline
void
GpuLikelihood::CalcIntegral()
{
  _calcIntegral<<<integralBuffSize_,THREADSPERBLOCK>>>(numInfecs_,devDRowPtr_,devDColInd_,devDVal_,
      devEventTimes_,eventTimesPitch_,devSusceptibility_,devInfectivity_,gamma2_,delta_,obsTime_,devIntegral_);
  checkCudaError(hipGetLastError());

  thrust::device_ptr<float> integPtr(devIntegral_);
  integral_ = thrust::reduce(integPtr, integPtr + integralBuffSize_) * gamma1_;
}

void
GpuLikelihood::FullCalculate()
{

  timeval start, end;
  gettimeofday(&start, NULL);
  CalcInfectivityPow();
  CalcInfectivity();
  CalcSusceptibilityPow();
  CalcSusceptibility();

  CalcProduct();
  CalcIntegral();
  CalcBgIntegral();

  logLikelihood_ = lp_ - (integral_ + bgIntegral_);
  gettimeofday(&end, NULL);
  std::cerr << "Time (" << __PRETTY_FUNCTION__ << "): "
      << timeinseconds(start, end) << std::endl;
  std::cerr << "Likelihood (" << __PRETTY_FUNCTION__ << "): " << logLikelihood_ << std::endl;
}

void
GpuLikelihood::Calculate()
{
  timeval start, end;
  gettimeofday(&start, NULL);
  CalcInfectivity();
  CalcSusceptibility();
  CalcIntegral();
  CalcProduct();
  CalcBgIntegral();
  logLikelihood_ = lp_ - (integral_ + bgIntegral_);
  gettimeofday(&end, NULL);
  std::cerr << "Time (" << __PRETTY_FUNCTION__ << "): "
      << timeinseconds(start, end) << std::endl;
  std::cerr << "Likelihood (" << __PRETTY_FUNCTION__ << "): " << logLikelihood_ << std::endl;
}

void
GpuLikelihood::UpdateInfectionTime(const int idx, const float newTime)
{
  // Require to know number of cols per row -- probably store in host mem.
  int blocksPerGrid = (hostDRowPtr_[idx+1] - hostDRowPtr_[idx] + THREADSPERBLOCK - 1) / THREADSPERBLOCK;
  //_updateInfectionTime<<<blocksPerGrid, THREADSPERBLOCK>>>()
}

float
GpuLikelihood::LogLikelihood() const
{

  return logLikelihood_;
}

