#include "hip/hip_runtime.h"
/*
 * GpuLikelihood.cpp
 *
 *  Created on: Feb 13, 2012
 *      Author: stsiab
 */
#include <stdexcept>
#include <string>
#include <iostream>
#include <sstream>
#include <vector>
#include <utility>
#include <cmath>
#include <math_functions.h>
#include <hip/device_functions.h>
#include <sys/time.h>
#include <thrust/sort.h>
#include <thrust/count.h>
#include <thrust/find.h>

#include "GpuLikelihood.hpp"

// Constants
const float UNITY = 1.0;
const float ZERO = 0.0;

inline
double
timeinseconds(const timeval a, const timeval b)
{
  timeval result;
  timersub(&b, &a, &result);
  return result.tv_sec + result.tv_usec / 1000000.0;
}

class GpuRuntimeError : public std::exception
{
public:
  GpuRuntimeError(const std::string usrMsg, hipError_t cudaErr)
  {
    msg_ = "GPU Runtime Error: ";
    msg_ += usrMsg;
    msg_ += " (";
    msg_ += cudaErr;
    msg_ += ",";
    msg_ += hipGetErrorString(cudaErr);
    msg_ += ")";
  }
  ~GpuRuntimeError() throw ()
  {
  }
  ;
  const char*
  what() const throw ()
  {
    return msg_.c_str();
  }

private:
  std::string msg_;
};

#define checkCudaError(err)  __checkCudaError(err, __FILE__, __LINE__)

void
__checkCudaError(const hipError_t err, const char* file, const int line)
{
  if (err != hipSuccess)
    {
      std::stringstream s;
      s << file << "(" << line << ") : Cuda Runtime error ";
      throw GpuRuntimeError(s.str(), err);
    }
}

// CUDA kernels

template<typename T>
  struct Log
  {
    __host__ __device__
    T
    operator()(const T& val) const
    {
      return logf(val);
    }
  };

template<typename T>
struct LessThanZero
{
  __host__ __device__
  bool
  operator()(const T& val) const
  {
    return val < 0;
  }
};

template<typename T1, typename T2>
struct IndirectMin
{
  __host__ __device__
  IndirectMin(T2* ptr) : ptr_(ptr) {};

  __host__ __device__
  bool
  operator()(const T1 lhs, const T1 rhs) const
  {
    return ptr_[lhs] < ptr_[rhs];
  }
private:
  T2* ptr_;
};



__device__ float
_atomicAdd(float* address, float val)
{
  unsigned int* address_as_ui = (unsigned int*) address;
  unsigned int old = *address_as_ui, assumed;
  do
    {
      assumed = old;
      old = atomicCAS(address_as_ui, assumed,
          __float_as_int(val + __int_as_float(assumed)));
    }
  while (assumed != old);
  return __int_as_float(old);
}

__device__ void
_shmemReduce(float* buff)
{
  // Reduce buffer into output
  __syncthreads();


  for (unsigned int size = blockDim.x / 2; size > 32; size >>= 1)
    {
      if (threadIdx.x < size)
        buff[threadIdx.x] += buff[threadIdx.x + size];
      __syncthreads();
    }
  if (threadIdx.x < 32)
    {
      volatile float* vbuff = buff;
      vbuff[threadIdx.x] += vbuff[threadIdx.x + 32];
      vbuff[threadIdx.x] += vbuff[threadIdx.x + 16];
      vbuff[threadIdx.x] += vbuff[threadIdx.x + 8];
      vbuff[threadIdx.x] += vbuff[threadIdx.x + 4];
      vbuff[threadIdx.x] += vbuff[threadIdx.x + 2];
      vbuff[threadIdx.x] += vbuff[threadIdx.x + 1];
    }

  __syncthreads();
}

__global__ void
_sanitizeEventTimes(float* data, int pitch, const float time, const int size)
{
  // Ensures Ii <= Ni <= Ri for individual i
  int tid = threadIdx.x + blockDim.x * blockIdx.x;

  if (tid < size)
    {
      float R = data[tid + pitch * 2];
      float N = data[tid + pitch];
      float I = data[tid];

      R = fminf(R, time);
      N = fminf(N,    R);
      I = fminf(I,    N);

      data[tid + pitch * 2] = R;
      data[tid + pitch] = N;
      data[tid] = I;
    }
}

__global__ void
_calcIntegral(const unsigned int* infecIdx, const int infecSize, int* DRowPtr, int* DColInd, float* D,
    float* eventTimes, const int eventTimesPitch, const float* susceptibility,
    const float* infectivity, const float gamma2, const float delta,
    float* output)
{
  // Each warp calculates a row i of the sparse matrix

  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int row = tid / 32; // Global Warp id
  int lane = tid & (32 - 1); // Id within a warp

  __shared__
  float buff[THREADSPERBLOCK];

  buff[threadIdx.x] = 0.0f;

  if (row < infecSize)
    {
      int i = infecIdx[row];

      int begin = DRowPtr[i];
      int end = DRowPtr[i + 1];
      float Ii = eventTimes[i];
      float Ni = eventTimes[i + eventTimesPitch];
      float Ri = eventTimes[i + eventTimesPitch * 2];

      float threadSum = 0.0f;
      for (int jj = begin + lane; jj < end; jj += 32)
        {
          // Integrated infection pressure
          float Ij = eventTimes[DColInd[jj]];
          float betaij = fminf(Ni, Ij) - fminf(Ii, Ij);
          betaij += gamma2 * (fminf(Ri, Ij) - fminf(Ni, Ij));

          // Apply distance kernel and suscep
          betaij *= delta / (delta * delta + D[jj]);
          betaij *= susceptibility[DColInd[jj]];
          threadSum += betaij;
        }
      buff[threadIdx.x] = threadSum * infectivity[i];
    }

  // Reduce all warp sums and write to global memory.

  _shmemReduce(buff);

  if (threadIdx.x == 0)
    {
      output[blockIdx.x] = buff[0];
    }
}

__global__ void
_calcProduct(const unsigned int* infecIdx, const int infecSize, const int* DRowPtr, const int* DColInd,
    float* D, const float* eventTimes, const int eventTimesPitch,
    const float* susceptibility, const float* infectivity, const float epsilon,
    const float gamma1, const float gamma2, const float delta, float* prodCache)
{
  // Each warp calculates a row of the sparse matrix

  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int gwid = tid / 32; // Global Warp id
  int lane = tid & (32 - 1); // Id within a warp

  __shared__
  float threadProdCache[THREADSPERBLOCK];
  threadProdCache[threadIdx.x] = 0.0f;
  int row = gwid;

  if (row < infecSize)
    {
      int j = infecIdx[row];

      int begin = DRowPtr[j];
      int end = DRowPtr[j + 1];

      float Ij = eventTimes[j];

      for (int ii = begin + lane; ii < end/* and DColInd[ii] < infecSize*/;
          ii += 32)
        {
          int i = DColInd[ii];
          float Ii = eventTimes[i];
          float Ni = eventTimes[eventTimesPitch + i];
          float Ri = eventTimes[eventTimesPitch * 2 + i];

          if(Ii < Ni) {
              float idxOnj = 0.0f;
              if (Ii < Ij and Ij <= Ni)
                idxOnj += 1.0f;
              else if (Ni < Ij and Ij <= Ri)
                idxOnj += gamma2;
              threadProdCache[threadIdx.x] += idxOnj * infectivity[i] * delta / (delta*delta + D[ii]);
          }
        }
      __syncthreads();

      // Reduce semi-products into productCache
      volatile float* vThreadProdCache = threadProdCache;
      if (lane < 16)
        {
          vThreadProdCache[threadIdx.x] += vThreadProdCache[threadIdx.x + 16];
          vThreadProdCache[threadIdx.x] += vThreadProdCache[threadIdx.x + 8];
          vThreadProdCache[threadIdx.x] += vThreadProdCache[threadIdx.x + 4];
          vThreadProdCache[threadIdx.x] += vThreadProdCache[threadIdx.x + 2];
          vThreadProdCache[threadIdx.x] += vThreadProdCache[threadIdx.x + 1];
        }
      __syncthreads();

      // Write out to global memory
      if (lane == 0)
        prodCache[j] = threadProdCache[threadIdx.x] * susceptibility[j]
            * gamma1 + epsilon;
    }
}

__global__ void
calcSpecPow(const unsigned int size, const int nSpecies, float* specpow,
    const int specpowPitch, const float* animals, const int animalsPitch,
    const float* powers)
{
  int row = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < size)
    {
      for (unsigned int col = 0; col < nSpecies; ++col)
        {
          specpow[col * specpowPitch + row] = powf(
              animals[col * animalsPitch + row], powers[col]);
        }
    }
}

__global__ void
_updateInfectionTimeIntegral(const unsigned int idx, const unsigned int* infecIdx, const float newTime,
    int* DRowPtr, int* DColInd, float* D, float* eventTimes,
    const int eventTimesPitch, const float* susceptibility,
    const float* infectivity, const float gamma2, const float delta,
    float* output)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  extern __shared__
  float buff[];
  buff[threadIdx.x] = 0.0f;

  int i = infecIdx[idx];
  int begin = DRowPtr[i];
  int end = DRowPtr[i + 1];

  if (tid < end - begin)
    {
      int j = DColInd[begin + tid];

      float Ii = eventTimes[i];
      float Ni = eventTimes[i + eventTimesPitch];

      float Ij = eventTimes[j];
      float Nj = eventTimes[j + eventTimesPitch];
      float Rj = eventTimes[j + eventTimesPitch * 2];

      float jOnIdx = 0.0f;
      if(Ij < Nj)
        {
          // Recalculate pressure from j on idx
          jOnIdx = (fminf(Nj, newTime) - fminf(Ij, newTime))
                      + gamma2 * (fminf(Rj, newTime) - fminf(Nj, newTime)); // New pressure
          jOnIdx -= (fminf(Nj, Ii) - fminf(Ii, Ij))
                  + gamma2 * (fminf(Rj, Ii) - fminf(Nj, Ii)); // Old pressure
          // Apply infec and suscep
          jOnIdx *= susceptibility[i];
          jOnIdx *= infectivity[j];
     }

      // Recalculate pressure from idx on j
      float IdxOnj = fminf(Ni, Ij) - fminf(newTime, Ij);
      IdxOnj -= fminf(Ni, Ij) - fminf(Ii, Ij);
      IdxOnj *= susceptibility[j];
      IdxOnj *= infectivity[i];

      buff[threadIdx.x] = (IdxOnj + jOnIdx) * (delta / (delta * delta + D[begin + tid]));

      // Reduce buffer into output
      _shmemReduce(buff);

    }

  if (threadIdx.x == 0)
    {
      output[blockIdx.x] = buff[0];
    }
}

__global__ void
_updateInfectionTimeProduct(const unsigned int idx, const unsigned int* infecIdx, const float newTime,
    int* DRowPtr, int* DColInd, float* D,
    float* eventTimes, const int eventTimesPitch, const float* susceptibility,
    const float* infectivity, const float epsilon, const float gamma1,
    const float gamma2, const float delta, float* prodCache)
{
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  extern __shared__
  float buff[];
  buff[threadIdx.x] = 0.0f;

  int i = infecIdx[idx];
  int begin = DRowPtr[i];
  int end = DRowPtr[i + 1];

  if (tid < end - begin) // Massive amount of wasted time just here!
    {
      int j = DColInd[begin + tid];

      float Ij = eventTimes[j];
      float Nj = eventTimes[j + eventTimesPitch];

      if(Ij < Nj) {

          float Ii = eventTimes[i];
          float Ni = eventTimes[i + eventTimesPitch];

          float Rj = eventTimes[j + eventTimesPitch * 2];

          // Adjust product cache from idx on others
          float idxOnj = 0.0f;
          if (Ii < Ij and Ij <= Ni)
            idxOnj -= 1.0f;
          if (newTime < Ij and Ij <= Ni)
            idxOnj += 1.0f;

          idxOnj *= gamma1 * infectivity[i] * susceptibility[j] * delta
              / (delta * delta + D[begin + tid]);
          prodCache[j] += idxOnj;

          // Recalculate instantaneous pressure on idx
          float jOnIdx = 0.0f;
          if (Ij < newTime and newTime <= Nj)
            jOnIdx = 1.0f;
          else if (Nj < newTime and newTime <= Rj)
            jOnIdx = gamma2;

          jOnIdx *= susceptibility[i] * infectivity[j] * delta
                  / (delta * delta + D[begin + tid]);

          buff[threadIdx.x] = jOnIdx * gamma1;

          }

      _shmemReduce(buff);

      if (threadIdx.x == 0)
        _atomicAdd(prodCache + i, buff[0]); // Maybe better to create an external reduction buffer here.
      if (tid == 0)
        _atomicAdd(prodCache + i, epsilon);
    }
}



__global__ void
_addInfectionTimeIntegral(const unsigned int idx, const unsigned int* infecIdx, const float newTime,
    const int* DRowPtr, const int* DColInd, const float* D, const float* eventTimes,
    const int eventTimesPitch, const float* susceptibility,
    const float* infectivity, const float gamma2, const float delta,
    float* output)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  extern __shared__
  float buff[];
  buff[threadIdx.x] = 0.0f;

  int i = infecIdx[idx];
  int begin = DRowPtr[i];
  int end = DRowPtr[i + 1];

  if (tid < end - begin)
    {
      int j = DColInd[begin + tid];

      float Ii = eventTimes[i];
      float Ni = eventTimes[i + eventTimesPitch];
      float Ri = eventTimes[i + eventTimesPitch * 2];

      float Ij = eventTimes[j];
      float Nj = eventTimes[j + eventTimesPitch];
      float Rj = eventTimes[j + eventTimesPitch * 2];

      float jOnIdx = 0.0f;
      if(Ij < Nj)
        {
          // Calculate pressure from j on idx
          jOnIdx -= fminf(Nj, Ii) - fminf(Ij, Ii);
          jOnIdx -= gamma2 * (fminf(Rj, Ii) - fminf(Nj, Ii));
          jOnIdx += fminf(Nj, newTime) - fminf(Ij, newTime);
          jOnIdx += gamma2 * (fminf(Rj, newTime) - fminf(Nj, newTime));

          // Apply infec and suscep
          jOnIdx *= susceptibility[i];
          jOnIdx *= infectivity[j];
     }

      // Add pressure from idx on j
      float IdxOnj = fminf(Ni, Ij) - fminf(newTime, Ij);
      IdxOnj += gamma2 * (fminf(Ri, Ij) - fminf(Ni, Ij));
      IdxOnj *= susceptibility[j];
      IdxOnj *= infectivity[i];

      buff[threadIdx.x] = (IdxOnj + jOnIdx) * (delta / (delta * delta + D[begin + tid]));

      // Reduce buffer into output
      _shmemReduce(buff);
    }

  if (threadIdx.x == 0)
    {
      output[blockIdx.x] = buff[0];
    }
}



__global__ void
_delInfectionTimeIntegral(const unsigned int idx, const unsigned int* infecIdx, const float newTime,
    int* DRowPtr, int* DColInd, float* D, float* eventTimes,
    const int eventTimesPitch, const float* susceptibility,
    const float* infectivity, const float gamma2, const float delta,
    float* output)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  extern __shared__
  float buff[];
  buff[threadIdx.x] = 0.0f;

  int i = infecIdx[idx];
  int begin = DRowPtr[i];
  int end = DRowPtr[i + 1];

  if (tid < end - begin)
    {
      int j = DColInd[begin + tid];

      float Ii = eventTimes[i];
      float Ni = eventTimes[i + eventTimesPitch];
      float Ri = eventTimes[i + eventTimesPitch*2];

      float Ij = eventTimes[j];
      float Nj = eventTimes[j + eventTimesPitch];
      float Rj = eventTimes[j + eventTimesPitch * 2];

      float jOnIdx = 0.0f;
      if(Ij < Nj)
        {
          // Recalculate pressure from j on idx
          jOnIdx -= fminf(Nj, Ii) - fminf(Ii, Ij) + gamma2 * (fminf(Rj, Ii) - fminf(Nj, Ii)); // Old pressure
          jOnIdx += fminf(Nj, Ni) - fminf(Ij, Ni) + gamma2 * (fminf(Rj, Ni) - fminf(Nj, Ni)); // New pressure
          // Apply infec and suscep
          jOnIdx *= susceptibility[i];
          jOnIdx *= infectivity[j];
     }

      // Subtract pressure from idx on j
      float IdxOnj = 0.0f;
      IdxOnj -= fminf(Ni, Ij) - fminf(Ii, Ij);
      IdxOnj -= gamma2 * (fminf(Ri, Ij) - fminf(Ni, Ij));
      IdxOnj *= susceptibility[j];
      IdxOnj *= infectivity[i];

      buff[threadIdx.x] = (IdxOnj + jOnIdx) * (delta / (delta * delta + D[begin + tid]));

      // Reduce buffer into output
      _shmemReduce(buff);

    }

  if (threadIdx.x == 0)
    {
      output[blockIdx.x] = buff[0];
    }
}


__global__ void
_addInfectionTimeProduct(const unsigned int idx, const unsigned int* infecIdx, const float newTime,
    const int* DRowPtr, const int* DColInd, const float* D,
    const float* eventTimes, const int eventTimesPitch, const float* susceptibility,
    const float* infectivity, const float epsilon, const float gamma1,
    const float gamma2, const float delta, float* prodCache)
{
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  extern __shared__
  float buff[];
  buff[threadIdx.x] = 0.0f;

  int i = infecIdx[idx];
  int begin = DRowPtr[i];
  int end = DRowPtr[i + 1];

  if (tid < end - begin) // Massive amount of wasted time just here!
    {
      int j = DColInd[begin + tid];

      float Ij = eventTimes[j];
      float Nj = eventTimes[j + eventTimesPitch];

      if(Ij < Nj) { // Only look at infected individuals

          float Ni = eventTimes[i + eventTimesPitch    ];
          float Ri = eventTimes[i + eventTimesPitch * 2];
          float Rj = eventTimes[j + eventTimesPitch * 2];

          // Adjust product cache from idx on others
          float idxOnj = 0.0f;
          if (newTime < Ij and Ij <= Ni)
            idxOnj += 1.0f;
          else if (Ni < Ij and Ij <= Ri)
            idxOnj += gamma2;

          idxOnj *= gamma1 * infectivity[i] * susceptibility[j] * delta
              / (delta * delta + D[begin + tid]);
          prodCache[j] += idxOnj;

          // Calculate instantaneous pressure on idx
          float jOnIdx = 0.0f;
          if (Ij < newTime and newTime <= Nj)
            jOnIdx = 1.0f;
          else if (Nj < newTime and newTime <= Rj)
            jOnIdx = gamma2;

          jOnIdx *= gamma1 * infectivity[j] * susceptibility[i] * delta
                  / (delta * delta + D[begin + tid]);

          buff[threadIdx.x] = jOnIdx;

          }

      _shmemReduce(buff);

      if (threadIdx.x == 0)
        _atomicAdd(prodCache + i, buff[0]);
      if (tid == 0)
        _atomicAdd(prodCache + i, epsilon);
    }
}


__global__ void
_delInfectionTimeProduct(const unsigned int idx, const unsigned int* infecIdx, const float newTime,
    int* DRowPtr, int* DColInd, float* D,
    float* eventTimes, const int eventTimesPitch, const float* susceptibility,
    const float* infectivity, const float epsilon, const float gamma1,
    const float gamma2, const float delta, float* prodCache)
{
  int tid = threadIdx.x + blockDim.x * blockIdx.x;

  int i = infecIdx[idx];
  int begin = DRowPtr[i];
  int end = DRowPtr[i + 1];

  if (tid < end - begin) // Massive amount of wasted time just here!
    {
      int j = DColInd[begin + tid];

      float Ij = eventTimes[j];
      float Nj = eventTimes[j + eventTimesPitch];

      if(Ij < Nj) {

          float Ii = eventTimes[i];
          float Ni = eventTimes[i + eventTimesPitch];
          float Ri = eventTimes[i + eventTimesPitch*2];

          // Adjust product cache from idx on others
          float idxOnj = 0.0;
          if (Ii < Ij and Ij <= Ni)
            idxOnj -= 1.0;
          else if(Ni < Ij and Ij <= Ri)
            idxOnj -= gamma2;

          idxOnj *= gamma1 * infectivity[i] * susceptibility[j] * delta
              / (delta * delta + D[begin + tid]);
          prodCache[j] += idxOnj;
          }
    }
}





GpuLikelihood::GpuLikelihood(const size_t realPopSize, const size_t popSize,
    const size_t numInfecs, const size_t maxInfecs, const size_t nSpecies, const float obsTime,
    const size_t distanceNNZ) :
    realPopSize_(realPopSize), popSize_(popSize), numInfecs_(numInfecs), maxInfecs_(maxInfecs), numSpecies_(
        nSpecies), obsTime_(obsTime), I1Time_(0.0), I1Idx_(0), sumI_(0), bgIntegral_(
        0.0), covariateCopies_(0), devAnimals_(NULL), animalsPitch_(0), devAnimalsInfPow_(
        NULL), devAnimalsSuscPow_(NULL), devEventTimes_(NULL), devSusceptibility_(
        NULL), devInfectivity_(NULL), devDVal_(NULL), devDRowPtr_(NULL), devDColInd_(
        NULL), dnnz_(distanceNNZ), epsilon_(0.0f), gamma1_(0.0f), gamma2_(0.0f), devXi_(
        NULL), devPsi_(NULL), devZeta_(NULL), devPhi_(NULL), delta_(0.0f)
{

  // Allocate Animals_
  checkCudaError(
      hipMallocPitch(&devAnimals_, &animalsPitch_, popSize_ * sizeof(float), numSpecies_));
  animalsPitch_ /= sizeof(float);
  checkCudaError(
      hipMallocPitch(&devAnimalsSuscPow_, &animalsSuscPowPitch_, popSize_ * sizeof(float), numSpecies_));
  animalsSuscPowPitch_ /= sizeof(float);
  checkCudaError(
      hipMallocPitch(&devAnimalsInfPow_, &animalsInfPowPitch_, maxInfecs_ * sizeof(float), numSpecies_));
  animalsInfPowPitch_ /= sizeof(float);

  // Allocate Distance_ CRS matrix
  checkCudaError(hipMalloc(&devDVal_, dnnz_ * sizeof(float)));
  checkCudaError(hipMalloc(&devDRowPtr_, (popSize_ + 1) * sizeof(int)));
  checkCudaError(hipMalloc(&devDColInd_, dnnz_ * sizeof(float)));
  hostDRowPtr_ = new int[popSize_ + 1];

  // Set up reference counter to covariate data
  covariateCopies_ = new size_t;
  *covariateCopies_ = 1;

  // Allocate Event times - popSize_ * NUMEVENTS matrix
  checkCudaError(
      hipMallocPitch(&devEventTimes_, &eventTimesPitch_, popSize_ * sizeof(float), NUMEVENTS));
  eventTimesPitch_ /= sizeof(float);

  // Allocate intermediate infectivity and susceptibility
  checkCudaError(hipMalloc(&devSusceptibility_, popSize_ * sizeof(float)));
  checkCudaError(hipMalloc(&devInfectivity_, maxInfecs_ * sizeof(float)));

  // Allocate product cache
  devProduct_.resize(maxInfecs_);
  thrust::fill(devProduct_.begin(), devProduct_.end(), 1.0f);

  // Allocate integral array
  int numRequiredThreads = maxInfecs_ * 32; // One warp per infection
  integralBuffSize_ = (numRequiredThreads + THREADSPERBLOCK - 1)
      / THREADSPERBLOCK;
  devIntegral_.resize(integralBuffSize_);

  // Parameters
  checkCudaError(hipMalloc(&devXi_, numSpecies_ * sizeof(float)));
  checkCudaError(hipMalloc(&devPsi_, numSpecies_ * sizeof(float)));
  checkCudaError(hipMalloc(&devZeta_, numSpecies_ * sizeof(float)));
  checkCudaError(hipMalloc(&devPhi_, numSpecies_ * sizeof(float)));

  // BLAS handles
  blasStat_ = hipblasCreate(&cudaBLAS_);
  if (blasStat_ != HIPBLAS_STATUS_SUCCESS)
    throw std::runtime_error("CUBLAS init failed");

  sparseStat_ = hipsparseCreate(&cudaSparse_);
  if (sparseStat_ != HIPSPARSE_STATUS_SUCCESS)
    throw std::runtime_error("CUSPARSE init failed");

  sparseStat_ = hipsparseCreateMatDescr(&crsDescr_);
  if (sparseStat_ != HIPSPARSE_STATUS_SUCCESS)
    throw std::runtime_error("CUSPARSE matrix descriptor init failed");
  hipsparseSetMatType(crsDescr_, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(crsDescr_, HIPSPARSE_INDEX_BASE_ZERO);

}

// Copy constructor
GpuLikelihood::GpuLikelihood(const GpuLikelihood& other) :
    realPopSize_(other.realPopSize_), popSize_(other.popSize_), numInfecs_(
        other.numInfecs_), numSpecies_(other.numSpecies_), obsTime_(
        other.obsTime_), I1Time_(other.I1Time_), I1Idx_(other.I1Idx_), sumI_(
        other.sumI_), bgIntegral_(other.bgIntegral_), lp_(other.lp_), covariateCopies_(
        other.covariateCopies_), devAnimals_(other.devAnimals_), animalsPitch_(
        other.animalsPitch_), devDVal_(other.devDVal_), devDRowPtr_(
        other.devDRowPtr_), devDColInd_(other.devDColInd_), hostDRowPtr_(other.hostDRowPtr_),
        dnnz_(other.dnnz_), integralBuffSize_(
        other.integralBuffSize_), epsilon_(other.epsilon_), gamma1_(
        other.gamma1_), gamma2_(other.gamma2_), delta_(other.delta_)
{
  timeval start, end;
  gettimeofday(&start, NULL);
  // Allocate Animals_
  checkCudaError(
      hipMallocPitch(&devAnimalsInfPow_, &animalsInfPowPitch_, maxInfecs_ * sizeof(float), numSpecies_));
  animalsInfPowPitch_ /= sizeof(float);
  checkCudaError(
      hipMemcpy2D(devAnimalsInfPow_,animalsInfPowPitch_*sizeof(float),other.devAnimalsInfPow_,other.animalsInfPowPitch_*sizeof(float),maxInfecs_*sizeof(float),numSpecies_,hipMemcpyDeviceToDevice));

  checkCudaError(
      hipMallocPitch(&devAnimalsSuscPow_, &animalsSuscPowPitch_, popSize_ * sizeof(float), numSpecies_));
  animalsSuscPowPitch_ /= sizeof(float);
  checkCudaError(
      hipMemcpy2D(devAnimalsSuscPow_,animalsSuscPowPitch_*sizeof(float),other.devAnimalsSuscPow_,other.animalsSuscPowPitch_*sizeof(float),popSize_*sizeof(float),numSpecies_,hipMemcpyDeviceToDevice));

  // Allocate and copy event times - popSize_ * NUMEVENTS matrix
  checkCudaError(
      hipMallocPitch(&devEventTimes_, &eventTimesPitch_, popSize_ * sizeof(float), NUMEVENTS));
  eventTimesPitch_ /= sizeof(float);
  checkCudaError(
      hipMemcpy2D(devEventTimes_,eventTimesPitch_*sizeof(float),other.devEventTimes_,other.eventTimesPitch_*sizeof(float),popSize_*sizeof(float), NUMEVENTS, hipMemcpyDeviceToDevice));

  // Allocate and copy intermediate infectivity and susceptibility
  checkCudaError(hipMalloc(&devSusceptibility_, popSize_ * sizeof(float)));
  checkCudaError(
      hipMemcpy(devSusceptibility_, other.devSusceptibility_, popSize_ * sizeof(float),hipMemcpyDeviceToDevice));
  checkCudaError(hipMalloc(&devInfectivity_, maxInfecs_ * sizeof(float)));
  checkCudaError(
      hipMemcpy(devInfectivity_, other.devInfectivity_, maxInfecs_ * sizeof(float), hipMemcpyDeviceToDevice));

  // Infection index
  devInfecIdx_ = other.devInfecIdx_;
  hostInfecIdx_ = other.hostInfecIdx_;

  // Allocate and copy product vector
  devProduct_ = other.devProduct_;
  devIntegral_ = other.devIntegral_;

  // Parameters -- Allocate and Copy
  checkCudaError(hipMalloc(&devXi_, numSpecies_ * sizeof(float)));
  checkCudaError(
      hipMemcpy(devXi_, other.devXi_, numSpecies_ * sizeof(float), hipMemcpyDeviceToDevice));
  checkCudaError(hipMalloc(&devPsi_, numSpecies_ * sizeof(float)));
  checkCudaError(
      hipMemcpy(devPsi_, other.devPsi_, numSpecies_ * sizeof(float), hipMemcpyDeviceToDevice));
  checkCudaError(hipMalloc(&devZeta_, numSpecies_ * sizeof(float)));
  checkCudaError(
      hipMemcpy(devZeta_, other.devZeta_, numSpecies_ * sizeof(float), hipMemcpyDeviceToDevice));
  checkCudaError(hipMalloc(&devPhi_, numSpecies_ * sizeof(float)));
  checkCudaError(
      hipMemcpy(devPhi_, other.devPhi_, numSpecies_ * sizeof(float), hipMemcpyDeviceToDevice));

  // BLAS handles
  cudaBLAS_ = other.cudaBLAS_;
  cudaSparse_ = other.cudaSparse_;
  crsDescr_ = other.crsDescr_;

  ++*covariateCopies_; // Increment copies of covariate data

  gettimeofday(&end, NULL);
  std::cerr << "Time (" << __PRETTY_FUNCTION__ << "): "
      << timeinseconds(start, end) << std::endl;

}

// Assignment constructor
const GpuLikelihood&
GpuLikelihood::operator=(const GpuLikelihood& other)
{
  timeval start, end;
  gettimeofday(&start, NULL);
  // Copy animal powers
  checkCudaError(
      hipMemcpy2DAsync(devAnimalsInfPow_,animalsInfPowPitch_*sizeof(float),other.devAnimalsInfPow_,other.animalsInfPowPitch_*sizeof(float),maxInfecs_*sizeof(float),numSpecies_,hipMemcpyDeviceToDevice));
  checkCudaError(
      hipMemcpy2DAsync(devAnimalsSuscPow_,animalsSuscPowPitch_*sizeof(float),other.devAnimalsSuscPow_,other.animalsSuscPowPitch_*sizeof(float),popSize_*sizeof(float),numSpecies_,hipMemcpyDeviceToDevice));

  // copy event times
  checkCudaError(
      hipMemcpy2DAsync(devEventTimes_,eventTimesPitch_*sizeof(float),other.devEventTimes_,other.eventTimesPitch_*sizeof(float),popSize_*sizeof(float), NUMEVENTS, hipMemcpyDeviceToDevice));

  // copy intermediate infectivity and susceptibility
  checkCudaError(
      hipMemcpyAsync(devSusceptibility_, other.devSusceptibility_, popSize_ * sizeof(float),hipMemcpyDeviceToDevice));
  checkCudaError(
      hipMemcpyAsync(devInfectivity_, other.devInfectivity_, maxInfecs_ * sizeof(float), hipMemcpyDeviceToDevice));

  // Infection index
  devInfecIdx_ = other.devInfecIdx_;
  hostInfecIdx_ = other.hostInfecIdx_;

  // copy product vector
  devProduct_ = other.devProduct_;
  devIntegral_ = other.devIntegral_;

  // Device Parameters Copy
  checkCudaError(
      hipMemcpyAsync(devXi_, other.devXi_, numSpecies_ * sizeof(float), hipMemcpyDeviceToDevice));
  checkCudaError(
      hipMemcpyAsync(devPsi_, other.devPsi_, numSpecies_ * sizeof(float), hipMemcpyDeviceToDevice));
  checkCudaError(
      hipMemcpyAsync(devZeta_, other.devZeta_, numSpecies_ * sizeof(float), hipMemcpyDeviceToDevice));
  checkCudaError(
      hipMemcpyAsync(devPhi_, other.devPhi_, numSpecies_ * sizeof(float), hipMemcpyDeviceToDevice));

  // Host Parameters Copy
  epsilon_ = other.epsilon_;
  gamma1_ = other.gamma1_;
  gamma2_ = other.gamma2_;
  delta_ = other.delta_;

  // Likelihood components
  integral_ = other.integral_;
  bgIntegral_ = other.bgIntegral_;
  lp_ = other.lp_;

  gettimeofday(&end, NULL);
  std::cerr << "Time (" << __PRETTY_FUNCTION__ << "): "
      << timeinseconds(start, end) << std::endl;

  return *this;
}

GpuLikelihood::~GpuLikelihood()
{
  if (*covariateCopies_ == 1) // We're the last copy to be destroyed
    {
      hipFree(devAnimals_);
      hipFree(devDVal_);
      hipFree(devDRowPtr_);
      hipFree(devDColInd_);
      delete[] hostDRowPtr_;
      hipblasDestroy(cudaBLAS_);
      hipsparseDestroy(cudaSparse_);

      delete covariateCopies_;
    }

  if (devEventTimes_)
    hipFree(devEventTimes_);
  if (devAnimalsSuscPow_)
    hipFree(devAnimalsSuscPow_);
  if (devAnimalsInfPow_)
    hipFree(devAnimalsInfPow_);
  if (devSusceptibility_)
    hipFree(devSusceptibility_);
  if (devInfectivity_)
    hipFree(devInfectivity_);

  if (devXi_)
    hipFree(devXi_);
  if (devPsi_)
    hipFree(devPsi_);
  if (devZeta_)
    hipFree(devZeta_);
  if (devPhi_)
    hipFree(devPhi_);

}

void
GpuLikelihood::SetEvents(const float* data)
{
  // Get event times into GPU memory
  hipError_t rv = hipMemcpy2D(devEventTimes_,
      eventTimesPitch_ * sizeof(float), data, popSize_ * sizeof(float),
      popSize_ * sizeof(float), NUMEVENTS, hipMemcpyHostToDevice);
  if (rv != hipSuccess)
    throw GpuRuntimeError("Copying event times to device failed", rv);



  std::cerr << "Sanitizing events with obsTime: " << obsTime_ << std::endl;
  std::cerr << "Num infecs: " << numInfecs_ << std::endl;
  // Set any event times greater than obsTime to obsTime
  int blocksPerGrid = (popSize_ + THREADSPERBLOCK - 1) / THREADSPERBLOCK;
  _sanitizeEventTimes<<<blocksPerGrid, THREADSPERBLOCK>>>(devEventTimes_, eventTimesPitch_, obsTime_, popSize_);
  checkCudaError(hipGetLastError());

  thrust::device_ptr<float> p(devEventTimes_);
  hostInfecIdx_.clear();
  for(size_t i=0; i<numInfecs_; ++i)
    {
      hostInfecIdx_.push_back(i);
    }
  devInfecIdx_ = hostInfecIdx_;
  hipDeviceSynchronize();
  std::cerr << "\n\nHost InfecIdx_ size: " << hostInfecIdx_.size() <<  std::endl;

}

void
GpuLikelihood::SetSpecies(const float* data)
{
  // Loads species data assuming **COL MAJOR**
  hipError_t rv = hipMemcpy2D(devAnimals_, animalsPitch_ * sizeof(float),
      data, popSize_ * sizeof(float), popSize_ * sizeof(float), numSpecies_,
      hipMemcpyHostToDevice);
  if (rv != hipSuccess)
    throw GpuRuntimeError("Failed copying species data to device", rv);

  CalcInfectivity();
  CalcSusceptibility();
}

void
GpuLikelihood::SetDistance(const float* data, const int* rowptr,
    const int* colind)
{
  // Loads distance data into memory
  checkCudaError(
      hipMemcpy(devDVal_, data, dnnz_ * sizeof(float), hipMemcpyHostToDevice));
  checkCudaError(
      hipMemcpy(devDRowPtr_, rowptr, (popSize_ + 1) * sizeof(int), hipMemcpyHostToDevice));
  checkCudaError(
      hipMemcpy(devDColInd_, colind, dnnz_ * sizeof(int), hipMemcpyHostToDevice));
  checkCudaError(
      hipMemcpy(hostDRowPtr_, rowptr, (popSize_ + 1)*sizeof(int), hipMemcpyHostToHost));
}

void
GpuLikelihood::SetParameters(float* epsilon, float* gamma1, float* gamma2,
    float* xi, float* psi, float* zeta, float* phi, float* delta)
{
  epsilon_ = *epsilon;
  gamma1_ = *gamma1;
  gamma2_ = *gamma2;
  delta_ = *delta;

  checkCudaError(
      hipMemcpy(devXi_, xi, numSpecies_ * sizeof(float), hipMemcpyHostToDevice));

  checkCudaError(
      hipMemcpy(devPsi_, psi, numSpecies_ * sizeof(float), hipMemcpyHostToDevice));

  checkCudaError(
      hipMemcpy(devZeta_, zeta, numSpecies_ * sizeof(float), hipMemcpyHostToDevice));

  checkCudaError(
      hipMemcpy(devPhi_, phi, numSpecies_ * sizeof(float), hipMemcpyHostToDevice));
}

inline
void
GpuLikelihood::CalcInfectivityPow()
{
  int dimBlock(THREADSPERBLOCK);
  int dimGrid((maxInfecs_ + THREADSPERBLOCK - 1) / THREADSPERBLOCK);
calcSpecPow<<<dimGrid, dimBlock>>>(maxInfecs_,numSpecies_,devAnimalsInfPow_, animalsInfPowPitch_,devAnimals_,animalsPitch_,devPsi_);
                checkCudaError(hipGetLastError());
}

inline
void
GpuLikelihood::CalcInfectivity()
{

  // Now calculate infectivity
  blasStat_ = hipblasSgemv(cudaBLAS_, HIPBLAS_OP_N, maxInfecs_, numSpecies_,
      &UNITY, devAnimalsInfPow_, animalsInfPowPitch_, devXi_, 1, &ZERO,
      devInfectivity_, 1);
  if (blasStat_ != HIPBLAS_STATUS_SUCCESS)
    {
      std::cerr << "Error in infectivity: " << blasStat_ << std::endl;
    }

}

inline
void
GpuLikelihood::CalcSusceptibilityPow()
{
  int dimBlock(THREADSPERBLOCK);
  int dimGrid((popSize_ + THREADSPERBLOCK - 1) / THREADSPERBLOCK);
calcSpecPow<<<dimGrid, dimBlock>>>(popSize_,numSpecies_,devAnimalsSuscPow_,animalsSuscPowPitch_, devAnimals_,animalsPitch_,devPhi_);
                checkCudaError(hipGetLastError());
}

inline
void
GpuLikelihood::CalcSusceptibility()
{
  // Calculates susceptibility powers and sums over suscept.
  blasStat_ = hipblasSgemv(cudaBLAS_, HIPBLAS_OP_N, popSize_, numSpecies_, &UNITY,
      devAnimalsSuscPow_, animalsSuscPowPitch_, devZeta_, 1, &ZERO,
      devSusceptibility_, 1);
  if (blasStat_ != HIPBLAS_STATUS_SUCCESS)
    {
      std::cerr << "Error in susceptibility: " << blasStat_ << std::endl;
    }
}

inline
void
GpuLikelihood::UpdateI1()
{
  thrust::device_vector<unsigned int>::iterator myMin;
  myMin = thrust::min_element(devInfecIdx_.begin(), devInfecIdx_.end(), IndirectMin<unsigned int,float>(devEventTimes_));
  I1Idx_ = *myMin;

  thrust::device_ptr<float> v(devEventTimes_);
  I1Time_ = v[I1Idx_];
}
inline
void
GpuLikelihood::CalcBgIntegral()
{
  thrust::device_ptr<float> v(devEventTimes_);
  sumI_ = thrust::reduce(v, v + popSize_, (realPopSize_ - popSize_)*obsTime_, thrust::plus<float>());

  bgIntegral_ = sumI_ - (v[I1Idx_]*realPopSize_);
  bgIntegral_ *= epsilon_;
}

inline
void
GpuLikelihood::CalcProduct()
{

  thrust::device_vector<float> tmpProd(devProduct_.size());
  thrust::fill(tmpProd.begin(), tmpProd.end(), 1.0f);

  _calcProduct<<<integralBuffSize_,THREADSPERBLOCK>>>(devInfecIdx_.data().base(),devInfecIdx_.size(),devDRowPtr_,devDColInd_,devDVal_,
      devEventTimes_,eventTimesPitch_,devSusceptibility_,devInfectivity_,epsilon_,gamma1_,gamma2_,delta_,tmpProd.data().base());
  checkCudaError(hipGetLastError());

  std::cerr << "I1 = " << I1Idx_ << std::endl;
  tmpProd[I1Idx_] = 1.0f;

  // Check tmpProd is the same as devProduct
//  if(!thrust::equal(devProduct_.begin(), devProduct_.end(), tmpProd.begin()))
//    {
//      for(size_t i=0; i<tmpProd.size(); ++i)
//        {
//          std::cerr << i << "\t" << tmpProd[i] << "\t\t" << devProduct_[i];
//          if(tmpProd[i] != devProduct_[i]) std::cerr << "*****";
//          std::cerr << std::endl;
//        }
//    }

  devProduct_ = tmpProd;

  lp_ = thrust::transform_reduce(devProduct_.begin(), devProduct_.end(), Log<float>(),
      0.0f, thrust::plus<float>());
}

inline
void
GpuLikelihood::CalcIntegral()
{

  int numRequiredThreads = devInfecIdx_.size() * 32; // One warp per infection
  int integralBuffSize = (numRequiredThreads + THREADSPERBLOCK - 1)
       / THREADSPERBLOCK;


_calcIntegral<<<integralBuffSize_,THREADSPERBLOCK>>>(devInfecIdx_.data().base(),devInfecIdx_.size(),devDRowPtr_,devDColInd_,devDVal_,
      devEventTimes_,eventTimesPitch_,devSusceptibility_,devInfectivity_,gamma2_,delta_,devIntegral_.data().base());
        checkCudaError(hipGetLastError());

  integral_ = thrust::reduce(devIntegral_.begin(), devIntegral_.begin() + integralBuffSize) * gamma1_;
}

void
GpuLikelihood::FullCalculate()
{

  timeval start, end;
  gettimeofday(&start, NULL);
  CalcInfectivityPow();
  CalcInfectivity();
  CalcSusceptibilityPow();
  CalcSusceptibility();

  UpdateI1();
  CalcProduct();
  CalcIntegral();
  CalcBgIntegral();

  logLikelihood_ = lp_;// - (integral_ + bgIntegral_);
  gettimeofday(&end, NULL);
  std::cerr << "Time (" << __PRETTY_FUNCTION__ << "): "
      << timeinseconds(start, end) << std::endl;
  std::cerr << "Likelihood (" << __PRETTY_FUNCTION__ << "): " << logLikelihood_
      << std::endl;

}

void
GpuLikelihood::Calculate()
{
  timeval start, end;
  gettimeofday(&start, NULL);
  CalcInfectivity();
  CalcSusceptibility();

  UpdateI1();
  CalcIntegral();
  CalcProduct();
  CalcBgIntegral();
  logLikelihood_ = lp_;// - (integral_ + bgIntegral_);
  gettimeofday(&end, NULL);
  std::cerr << "Time (" << __PRETTY_FUNCTION__ << "): "
      << timeinseconds(start, end) << std::endl;
  std::cerr.precision(20);
  std::cerr << "Likelihood (" << __PRETTY_FUNCTION__ << "): " << logLikelihood_
      << std::endl;
}

void
GpuLikelihood::UpdateInfectionTime(const unsigned int idx, const float inTime)
{
  // Require to know number of cols per row -- probably store in host mem.
  // Also, may be optimal to use a much lower THREADSPERBLOCK than the app-wide setting.


  timeval start, end;
  gettimeofday(&start, NULL);

  thrust::device_ptr<float> eventTimesPtr(devEventTimes_);
  float newTime = *(eventTimesPtr+eventTimesPitch_+idx) - inTime;

  int blocksPerGrid = (hostDRowPtr_[idx + 1] - hostDRowPtr_[idx]
      + THREADSPERBLOCK - 1) / THREADSPERBLOCK + 1;
  _updateInfectionTimeIntegral<<<blocksPerGrid, THREADSPERBLOCK, THREADSPERBLOCK*sizeof(float)>>>(idx, devInfecIdx_.data().base(), newTime,
      devDRowPtr_, devDColInd_, devDVal_,
      devEventTimes_, eventTimesPitch_, devSusceptibility_,
      devInfectivity_, gamma2_, delta_, devIntegral_.data().base());
      checkCudaError(hipGetLastError());

  integral_ += thrust::reduce(devIntegral_.begin(), devIntegral_.begin() + blocksPerGrid) * gamma1_;


  // If a new I1 is created by moving a non-I1 infection time, zero out I1
  if (newTime < I1Time_ and idx != I1Idx_) devProduct_[I1Idx_] = epsilon_;

  devProduct_[idx] = 0.0f; // Zero out product entry for idx.
_updateInfectionTimeProduct<<<blocksPerGrid, THREADSPERBLOCK, THREADSPERBLOCK*sizeof(float)>>>(idx, devInfecIdx_.data().base(), newTime, devDRowPtr_,
      devDColInd_, devDVal_, devEventTimes_, eventTimesPitch_,
      devSusceptibility_, devInfectivity_, epsilon_, gamma1_, gamma2_,
      delta_, devProduct_.data().base());
      checkCudaError(hipGetLastError());

  // Make the change to the population
  eventTimesPtr[idx] = newTime;

  UpdateI1();
  CalcBgIntegral();

  devProduct_[I1Idx_] = 1.0f;
  lp_ = thrust::transform_reduce(devProduct_.begin(), devProduct_.end(), Log<float>(),
      0.0f, thrust::plus<float>());

  logLikelihood_ = lp_;// - (integral_ + bgIntegral_);

  gettimeofday(&end, NULL);
  std::cerr << "Time (" << __PRETTY_FUNCTION__ << "): "
      << timeinseconds(start, end) << std::endl;
  std::cerr.precision(20);
  std::cerr << "Likelihood (" << __PRETTY_FUNCTION__ << "): " << logLikelihood_
      << std::endl;
  std::cerr << "I1: " << I1Idx_ << " at " << I1Time_ << std::endl;
}


void
GpuLikelihood::AddInfectionTime(const unsigned int idx, const float inTime)
{
  // Require to know number of cols per row -- probably store in host mem.
  // Also, may be optimal to use a much lower THREADSPERBLOCK than the app-wide setting.


  timeval start, end;
  gettimeofday(&start, NULL);

  if(idx < numInfecs_ or idx >= maxInfecs_) throw std::range_error("Invalid idx in GpuLikelihood::AddInfectionTime");

  thrust::device_ptr<float> eventTimesPtr(devEventTimes_);
  float newTime = *(eventTimesPtr+eventTimesPitch_+idx) - inTime;

  // Ready the product cache to receive pressure
  devInfecIdx_.push_back(idx);
  devProduct_[idx] = 0.0f;

  unsigned int addIdx = devInfecIdx_.size()-1;

  int blocksPerGrid = (hostDRowPtr_[idx + 1] - hostDRowPtr_[idx]
      + THREADSPERBLOCK - 1) / THREADSPERBLOCK + 1;
  _addInfectionTimeIntegral<<<blocksPerGrid, THREADSPERBLOCK, THREADSPERBLOCK*sizeof(float)>>>(addIdx, devInfecIdx_.data().base(), newTime,
      devDRowPtr_, devDColInd_, devDVal_,
      devEventTimes_, eventTimesPitch_, devSusceptibility_,
      devInfectivity_, gamma2_, delta_, devIntegral_.data().base());
      checkCudaError(hipGetLastError());

  integral_ += thrust::reduce(devIntegral_.begin(), devIntegral_.begin() + blocksPerGrid) * gamma1_;


  // If a new I1 is created by moving a non-I1 infection time, set the old I1 to epsilon
  if (newTime < I1Time_) devProduct_[I1Idx_] = epsilon_;

  std::cerr << "Add index = " << addIdx << " (" << idx << ")" << std::endl;
_addInfectionTimeProduct<<<blocksPerGrid, THREADSPERBLOCK, THREADSPERBLOCK*sizeof(float)>>>(addIdx, thrust::raw_pointer_cast(&devInfecIdx_[0]), newTime, devDRowPtr_,
      devDColInd_, devDVal_, devEventTimes_, eventTimesPitch_,
      devSusceptibility_, devInfectivity_, epsilon_, gamma1_, gamma2_,
      delta_, thrust::raw_pointer_cast(&devProduct_[0]));
      checkCudaError(hipGetLastError());

  // Make the change to the population
  eventTimesPtr[idx] = newTime;

  UpdateI1();
  CalcBgIntegral();

  // Reduce product vector, correcting for I1
  devProduct_[I1Idx_] = 1.0f;
  lp_ = thrust::transform_reduce(devProduct_.begin(), devProduct_.end(), Log<float>(),
      0.0f, thrust::plus<float>());


  logLikelihood_ = lp_;// - (integral_ + bgIntegral_);



  gettimeofday(&end, NULL);
  std::cerr << "Time (" << __PRETTY_FUNCTION__ << "): "
      << timeinseconds(start, end) << std::endl;
  std::cerr.precision(20);
  std::cerr << "Likelihood (" << __PRETTY_FUNCTION__ << "): " << logLikelihood_
      << std::endl;
}


void
GpuLikelihood::DeleteInfectionTime(const unsigned int idx)
{
  // Require to know number of cols per row -- probably store in host mem.
  // Also, may be optimal to use a much lower THREADSPERBLOCK than the app-wide setting.


  timeval start, end;
  gettimeofday(&start, NULL);

  // Range check
  if(idx < numInfecs_ or idx >= devInfecIdx_.size()) throw std::range_error("Invalid idx in GpuLikelihood::DeleteInfectionTime");

  thrust::device_ptr<float> eventTimesPtr(devEventTimes_);
  unsigned int i = devInfecIdx_[idx];
  float notification = eventTimesPtr[i + eventTimesPitch_];
  devIntegral_.assign(devIntegral_.size(), 0.0f);
  std::cerr << "Deletin index " << i << std::endl;

  int blocksPerGrid = (hostDRowPtr_[i + 1] - hostDRowPtr_[i]
      + THREADSPERBLOCK - 1) / THREADSPERBLOCK + 1;
  _delInfectionTimeIntegral<<<blocksPerGrid, THREADSPERBLOCK, THREADSPERBLOCK*sizeof(float)>>>(idx, devInfecIdx_.data().base(), notification,
      devDRowPtr_, devDColInd_, devDVal_,
      devEventTimes_, eventTimesPitch_, devSusceptibility_,
      devInfectivity_, gamma2_, delta_, devIntegral_.data().base());
      checkCudaError(hipGetLastError());

  integral_ += thrust::reduce(devIntegral_.begin(), devIntegral_.begin() + blocksPerGrid) * gamma1_;

_delInfectionTimeProduct<<<blocksPerGrid, THREADSPERBLOCK, THREADSPERBLOCK*sizeof(float)>>>(idx, devInfecIdx_.data().base(), notification, devDRowPtr_,
      devDColInd_, devDVal_, devEventTimes_, eventTimesPitch_,
      devSusceptibility_, devInfectivity_, epsilon_, gamma1_, gamma2_,
      delta_, devProduct_.data().base());
      checkCudaError(hipGetLastError());

  // Make the change to the population
  eventTimesPtr[devInfecIdx_[idx]] = notification;

  devProduct_[i] = 1.0f;
  devInfecIdx_.erase(devInfecIdx_.begin() + idx);

  UpdateI1();
  CalcBgIntegral();

  // Reduce product vector, correcting for I1
  devProduct_[I1Idx_] = 1.0f;
  lp_ = thrust::transform_reduce(devProduct_.begin(), devProduct_.end(), Log<float>(),
      0.0f, thrust::plus<float>());

  logLikelihood_ = lp_;// - (integral_ + bgIntegral_);


  gettimeofday(&end, NULL);
  std::cerr << "Time (" << __PRETTY_FUNCTION__ << "): "
      << timeinseconds(start, end) << std::endl;
  std::cerr.precision(20);
  std::cerr << "Likelihood (" << __PRETTY_FUNCTION__ << "): " << logLikelihood_
      << std::endl;
}

float
GpuLikelihood::LogLikelihood() const
{

  return logLikelihood_;
}

float
GpuLikelihood::GetN(const int idx) const
{
  float rv;
  checkCudaError(hipMemcpy(devEventTimes_+idx+eventTimesPitch_,&rv,sizeof(float), hipMemcpyDeviceToHost));
  return rv;
}

void
GpuLikelihood::LazyAddInfecTime(const int idx, const float inTime)
{
  thrust::device_ptr<float> eventTimePtr(devEventTimes_);
  eventTimePtr[idx] = eventTimePtr[idx+eventTimesPitch_] - inTime;
  devInfecIdx_.push_back(idx);
  devProduct_.push_back(0.0f);
  hipDeviceSynchronize();
}

