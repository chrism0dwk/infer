#include "hip/hip_runtime.h"
/*
 * GpuLikelihood.cpp
 *
 *  Created on: Feb 13, 2012
 *      Author: stsiab
 */
#include <stdexcept>
#include <string>
#include <iostream>
#include <cmath>
#include <math_functions.h>

#include "GpuLikelihood.hpp"

// Constants
const float UNITY = 1.0;
const float ZERO = 0.0;

class GpuRuntimeError : public std::exception
{
public:
  GpuRuntimeError(const std::string usrMsg, hipError_t cudaErr)
  {
    msg_ = "GPU Runtime Error: ";
    msg_ += usrMsg;
    msg_ += " (";
    msg_ += cudaErr;
    msg_ += ",";
    msg_ += hipGetErrorString(cudaErr);
    msg_ += ")";
  }
  ~GpuRuntimeError() throw ()
  {
  }
  ;
  const char*
  what() const throw ()
  {
    return msg_.c_str();
  }

private:
  std::string msg_;
};

// CUDA kernels

template<typename T>
struct Min
  {
    __device__ __host__
    T
    operator()(const T& a, const T& b) const
    {
      return a < b ? a : b;
    }
    __device__ __host__
    T
    initval() const
    {
      return HUGE_VAL;
    }
  };

template<typename T>
struct Plus
  {
    __device__ __host__
    T
    operator()(const T& a, const T& b) const
    {
      return a + b;
    }
    __device__ __host__
    T
    initval() const
    {
      return 0;
    }
  };

__global__ void
calcDT(const float* D, const float* T, const int N, float* DT, const float delta)
{
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < N)
    DT[idx] = delta / (delta * delta + D[idx]) * T[idx];
}

__global__ void
calcT(const int infecSize, const int nnz, const int popSize, int* TRowPtr, int* TColInd,
    float* TVal, float* eventTimes, const int eventTimesPitch, const float gamma2, const float obsTime)
{
  // Each thread calculates a row i of the sparse matrix -- probably not efficient!

  int i = threadIdx.x + blockDim.x * blockIdx.x;

  if (i < infecSize)
    {
      int begin = TRowPtr[i];
      int end = TRowPtr[i + 1];

      float Ii = eventTimes[i]; // First column  -- argument for row-major here, I would have thought.
      float Ni = eventTimes[eventTimesPitch + i]; // Second column
      float Ri = eventTimes[eventTimesPitch * 2 + i]; // Third column

      for (int j = begin; j < end; ++j)
        {
          float Ij = eventTimes[TColInd[j]];
          float Nj = eventTimes[TColInd[j] + eventTimesPitch];

          float jMaxSuscep;
          jMaxSuscep = fminf(Nj, Ij);
          jMaxSuscep = fminf(jMaxSuscep, obsTime);
          float exposure = fminf(Ni, jMaxSuscep) - fminf(Ii, jMaxSuscep);
          exposure += gamma2 * (fminf(Ri, jMaxSuscep) - fminf(Ni, jMaxSuscep));
          TVal[j] = exposure;
        }
    }
}

__global__ void
calcSpecPow(const int size, const int nSpecies, float* specpow, const int specpowPitch,
    const float* animals, const int animalsPitch, const float* powers)
{
  int row = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < size)
    {
      for (unsigned int col=0; col<nSpecies; ++col)
        {
          specpow[col * specpowPitch + row] = powf(animals[col * animalsPitch + row],
              powers[col]);
        }
    }
}

template<unsigned int blockSize, typename Op>
  __global__ void
  reduction(const float* buffer, float* rb, int size)
  {
    extern __shared__ float threadBuff[];

    Op op; // Operation functor to use

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockSize * 2) + tid;
    unsigned int gridSize = blockSize * 2 * gridDim.x;
    threadBuff[tid] = op.initval(); // Initialize with default init value

    while (i < size)
      {
        threadBuff[tid] = buffer[i];
        if (i + blockSize < size) threadBuff[tid] = op(threadBuff[tid], buffer[i + blockSize]);
        i += gridSize;
      }
    __syncthreads();

    if (blockSize >= 512)
      {
        if (tid < 256)
          {
            threadBuff[tid] = op(threadBuff[tid], threadBuff[tid + 256]);
          }
        __syncthreads();
      }
    if (blockSize >= 256)
      {
        if (tid < 128)
          {
            threadBuff[tid] = op(threadBuff[tid], threadBuff[tid + 128]);
          }
        __syncthreads();
      }
    if (blockSize >= 128)
      {
        if (tid < 64)
          {
            threadBuff[tid] = op(threadBuff[tid], threadBuff[tid + 64]);
          }
        __syncthreads();
      }

    if (tid < 32)
      {
        if (blockSize >= 64)
          threadBuff[tid] = op(threadBuff[tid], threadBuff[tid + 32]);
        if (blockSize >= 32)
          threadBuff[tid] = op(threadBuff[tid], threadBuff[tid + 16]);
        if (blockSize >= 16)
          threadBuff[tid] = op(threadBuff[tid], threadBuff[tid + 8]);
        if (blockSize >= 8)
          threadBuff[tid] = op(threadBuff[tid], threadBuff[tid + 4]);
        if (blockSize >= 4)
          threadBuff[tid] = op(threadBuff[tid], threadBuff[tid + 2]);
        if (blockSize >= 2)
          threadBuff[tid] = op(threadBuff[tid], threadBuff[tid + 1]);
      }

    if (tid == 0)
      rb[blockIdx.x] = threadBuff[0];
  }


__global__ void
sequence(float* buff, int size)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < size) buff[idx] = (float)idx;
}

template<class Op>
  __host__ float
  gpuReduction(const float* deviceBuffer, const int size)
  {
    int numBlocks = (size + THREADSPERBLOCK - 1) / THREADSPERBLOCK / 2;
    float* buff = new float[numBlocks];
    float* devRB;

    hipError_t rv = hipMalloc(&devRB, numBlocks * sizeof(float));
    if (rv != hipSuccess)
      throw GpuRuntimeError("Cannot allocate memory for reduction buffer", rv);

    reduction<THREADSPERBLOCK, Op><<<numBlocks, THREADSPERBLOCK, THREADSPERBLOCK*sizeof(float)>>>(deviceBuffer, devRB, size);

    rv = hipMemcpy(buff, devRB, numBlocks * sizeof(float),
        hipMemcpyDeviceToHost);
    if (rv != hipSuccess)
      throw GpuRuntimeError("Could not retrieve reduction buffer from device",
          rv);

    hipFree(devRB);

    // Final reduction on host
    float result = buff[0];
    Op op;
    for (size_t i = 1; i < numBlocks; ++i)
      result = op(buff[i],result);

    delete[] buff;

    return result;
  }

GpuLikelihood::GpuLikelihood(const size_t realPopSize, const size_t popSize,
    const size_t numInfecs, const size_t nSpecies, const float obsTime,
    const size_t distanceNNZ) :
    realPopSize_(realPopSize), popSize_(popSize), numInfecs_(numInfecs), numSpecies_(
        nSpecies), obsTime_(obsTime), I1Time_(0.0), bgIntegral_(0.0), devAnimals_(
        NULL), animalsPitch_(0), devAnimalsInfPow_(NULL), devAnimalsSuscPow_(NULL), devEventTimes_(
        NULL), devSusceptibility_(NULL), devInfectivity_(NULL), devDVal_(NULL), devDRowPtr_(
        NULL), devDColInd_(NULL), dnnz_(distanceNNZ), devTVal_(NULL), devDTVal_(
        NULL), devEVal_(NULL), devEColPtr_(NULL), devERowInd_(NULL), devTmp_(
        NULL), epsilon_(0.0f), gamma1_(0.0f), gamma2_(0.0f), devXi_(NULL), devPsi_(
        NULL), devZeta_(NULL), devPhi_(NULL), delta_(0.0f)
{

  int rv;

  std::cerr << "Pop size: " << popSize_ << std::endl;
  std::cerr << "Num infected: " << numInfecs_ << std::endl;
  std::cerr << "Num species: " << numSpecies_ << std::endl;

  rv = hipDeviceReset();
  if(rv != hipSuccess) throw GpuRuntimeError("Failed to reset device: ",(hipError_t)rv);

  // Allocate Event times - popSize_ * NUMEVENTS matrix
  rv = hipMallocPitch(&devEventTimes_, &eventTimesPitch_, popSize_ * sizeof(float), NUMEVENTS);
  eventTimesPitch_ /= sizeof(float);
  if (rv != hipSuccess)
    throw GpuRuntimeError("Cannot allocate memory for event times",
        (hipError_t) rv);

  // Allocate Animals_
  rv = hipMallocPitch(&devAnimals_, &animalsPitch_, popSize_ * sizeof(float), numSpecies_);
  animalsPitch_ /= sizeof(float);
  if (rv != hipSuccess)
    throw GpuRuntimeError("Cannot allocate memory for animals",
        (hipError_t) rv);
  rv = hipMallocPitch(&devAnimalsSuscPow_, &animalsSuscPowPitch_, popSize_ * sizeof(float), numSpecies_);
  animalsSuscPowPitch_ /= sizeof(float);
  if (rv != hipSuccess)
    throw GpuRuntimeError("Cannot allocate memory for animals susc pow",
        (hipError_t) rv);
  rv = hipMallocPitch(&devAnimalsInfPow_, &animalsInfPowPitch_, numInfecs_ * sizeof(float), numSpecies_);
  animalsInfPowPitch_ /= sizeof(float);
  if (rv != hipSuccess)
    throw GpuRuntimeError("Cannot allocate memory for animals inf pow",
        (hipError_t) rv);

  // Allocate Distance_ CRS matrix
  rv = hipMalloc(&devDVal_, dnnz_ * sizeof(float));
  rv |= hipMalloc(&devDRowPtr_, (popSize_ + 1) * sizeof(int));
  rv |= hipMalloc(&devDColInd_, dnnz_ * sizeof(float));
  if (rv != hipSuccess)
    throw GpuRuntimeError("Cannot allocate distance matrix", (hipError_t) rv);

  // Allocate intermediate T and DT
  rv = hipMalloc(&devTVal_, dnnz_ * sizeof(float));
  rv |= hipMalloc(&devDTVal_, dnnz_ * sizeof(float));
  if (rv != hipSuccess)
    throw GpuRuntimeError("Cannot allocate temporary structures",
        (hipError_t) rv);

  // Allocate intermediate infectivity and susceptibility
  rv = hipMalloc(&devSusceptibility_, popSize_ * sizeof(float));
  rv |= hipMalloc(&devInfectivity_, numInfecs_ * sizeof(float));
  if (rv != hipSuccess)
    throw GpuRuntimeError("Cannot allocate temporary structures",
        (hipError_t) rv);

  // Allocate temporary vector
  rv = hipMalloc(&devTmp_, numInfecs_ * sizeof(float));
  if (rv != hipSuccess)
    throw GpuRuntimeError("Cannot allocate temporary structures",
        (hipError_t) rv);

  // Parameters
  rv = hipMalloc(&devXi_, numSpecies_ * sizeof(float));
  rv |= hipMalloc(&devPsi_, numSpecies_ * sizeof(float));
  rv |= hipMalloc(&devZeta_, numSpecies_ * sizeof(float));
  rv |= hipMalloc(&devPhi_, numSpecies_ * sizeof(float));
  if (rv != hipSuccess)
    throw GpuRuntimeError("Cannot allocate device parameters",
        (hipError_t) rv);

  // BLAS handles
  blasStat_ = hipblasCreate(&cudaBLAS_);
  if (blasStat_ != HIPBLAS_STATUS_SUCCESS)
    throw std::runtime_error("CUBLAS init failed");

  sparseStat_ = hipsparseCreate(&cudaSparse_);
  if (sparseStat_ != HIPSPARSE_STATUS_SUCCESS)
    throw std::runtime_error("CUSPARSE init failed");

  sparseStat_ = hipsparseCreateMatDescr(&crsDescr_);
  if (sparseStat_ != HIPSPARSE_STATUS_SUCCESS)
    throw std::runtime_error("CUSPARSE matrix descriptor init failed");
  hipsparseSetMatType(crsDescr_, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(crsDescr_, HIPSPARSE_INDEX_BASE_ZERO);

}

GpuLikelihood::~GpuLikelihood()
{
  if (devEventTimes_ != NULL)
    hipFree(devEventTimes_);
  if (devAnimals_ != NULL)
    hipFree(devAnimals_);
  if (devAnimalsSuscPow_ != NULL)
    hipFree(devAnimalsSuscPow_);
  if (devAnimalsInfPow_ != NULL)
    hipFree(devAnimalsInfPow_);
  if (devDVal_ != NULL)
    hipFree(devDVal_);
  if (devDRowPtr_ != NULL)
    hipFree(devDRowPtr_);
  if (devDColInd_ != NULL)
    hipFree(devDColInd_);
  if (devTVal_ != NULL)
    hipFree(devTVal_);
  if (devDTVal_ != NULL)
    hipFree(devDTVal_);
  if (devSusceptibility_ != NULL)
    hipFree(devSusceptibility_);
  if (devInfectivity_ != NULL)
    hipFree(devInfectivity_);
  if (devTmp_ != NULL)
    hipFree(devTmp_);

  if (devXi_)
    hipFree(devXi_);
  if (devPsi_)
    hipFree(devPsi_);
  if (devZeta_)
    hipFree(devZeta_);
  if (devPhi_)
    hipFree(devPhi_);

  hipblasDestroy(cudaBLAS_);
  hipsparseDestroy(cudaSparse_);
}

void
GpuLikelihood::SetEvents(const float* data)
{
  // Get event times into GPU memory
  hipError_t rv = hipMemcpy2D(devEventTimes_, eventTimesPitch_*sizeof(float), data, popSize_*sizeof(float), popSize_*sizeof(float), NUMEVENTS, hipMemcpyHostToDevice);
  if (rv != hipSuccess)
    throw GpuRuntimeError("Copying event times to device failed", rv);
}

void
GpuLikelihood::SetSpecies(const float* data)
{
  // Loads species data assuming **COL MAJOR**
  hipError_t rv = hipMemcpy2D(devAnimals_, animalsPitch_*sizeof(float), data, popSize_*sizeof(float), popSize_*sizeof(float), numSpecies_, hipMemcpyHostToDevice);
  if(rv != hipSuccess) throw GpuRuntimeError("Failed copying species data to device",rv);

  CalcInfectivity();
  CalcSusceptibility();
}

void
GpuLikelihood::SetDistance(const float* data, const int* rowptr,
    const int* colind)
{
  // Loads distance data into memory
  int rv = hipMemcpy(devDVal_, data, dnnz_ * sizeof(float),
      hipMemcpyHostToDevice);
  rv |= hipMemcpy(devDRowPtr_, rowptr, (popSize_ + 1) * sizeof(int),
      hipMemcpyHostToDevice);
  rv |= hipMemcpy(devDColInd_, colind, dnnz_ * sizeof(int),
      hipMemcpyHostToDevice);
  if (rv != hipSuccess)
    throw GpuRuntimeError("Copy of distance matrix to device failed",
        (hipError_t) rv);
}

void
GpuLikelihood::SetParameters(float* epsilon, float* gamma1, float* gamma2,
    float* xi, float* psi, float* zeta, float* phi, float* delta)
{
  epsilon_ = *epsilon;
  gamma1_ = *gamma1;
  gamma2_ = *gamma2;
  delta_ = *delta;

  hipError_t rv = hipMemcpy(devXi_, xi, numSpecies_ * sizeof(float),
      hipMemcpyHostToDevice);
  if (rv != hipSuccess)
    throw GpuRuntimeError("Error copying xi to GPU", rv);

  rv = hipMemcpy(devPsi_, psi, numSpecies_ * sizeof(float),
      hipMemcpyHostToDevice);
  if (rv != hipSuccess)
    throw GpuRuntimeError("Error copying psi to GPU", rv);

  rv = hipMemcpy(devZeta_, zeta, numSpecies_ * sizeof(float),
      hipMemcpyHostToDevice);
  if (rv != hipSuccess)
    throw GpuRuntimeError("Error copying zeta to GPU", rv);

  rv = hipMemcpy(devPhi_, phi, numSpecies_ * sizeof(float),
      hipMemcpyHostToDevice);
  if (rv != hipSuccess)
    throw GpuRuntimeError("Error copying psi to GPU", rv);
}

void
GpuLikelihood::CalcEvents()
{
  // Calculates the T_ matrix -- sparse matrix operation!
  hipGetLastError();
  size_t blocksPerGrid = (dnnz_ + THREADSPERBLOCK - 1) / THREADSPERBLOCK;
  std::cerr << "Calculating Events with block size "
      << THREADSPERBLOCK << " and blocks per grid " << blocksPerGrid
      << std::endl;
  calcT<<<blocksPerGrid,THREADSPERBLOCK>>>(numInfecs_, dnnz_, popSize_, devDRowPtr_, devDColInd_, devTVal_, devEventTimes_, eventTimesPitch_, gamma2_, obsTime_);

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    throw GpuRuntimeError("Error calculating events", err);

//  float* vals = new float[dnnz_];
//  int* rowptr = new int[popSize_];
//  int* colind = new int[dnnz_];
//
//  hipMemcpy(vals, devTVal_, dnnz_ * sizeof(float), hipMemcpyDeviceToHost);
//  hipMemcpy(rowptr, devDRowPtr_, popSize_ * sizeof(int),
//      hipMemcpyDeviceToHost);
//  hipMemcpy(colind, devDColInd_, dnnz_ * sizeof(int), hipMemcpyDeviceToHost);
//
//  std::cerr << "T_: ";
//  for (size_t i = 0; i < dnnz_; ++i)
//    std::cerr << vals[i] << " ";
//  std::cerr << std::endl;
//  std::cerr << "TColInd: ";
//  for (size_t i = 0; i < dnnz_; ++i)
//    std::cerr << colind[i] << " ";
//  std::cerr << std::endl;
//  std::cerr << "TRowPtr: ";
//  for (size_t i = 0; i < popSize_; ++i)
//    std::cerr << rowptr[i] << " ";
//  std::cerr << std::endl;
//
//  delete[] vals;

}

void
GpuLikelihood::CalcInfectivity()
{
  // Calculates infectivity powers and sums over infectivity
  hipGetLastError(); // Reset error status
  // First do devAnimalsInfPow_
  int dimBlock(THREADSPERBLOCK);
  int dimGrid((numInfecs_ + THREADSPERBLOCK - 1) / THREADSPERBLOCK);
  calcSpecPow<<<dimGrid, dimBlock>>>(numInfecs_,numSpecies_,devAnimalsInfPow_, animalsInfPowPitch_,devAnimals_,animalsPitch_,devPsi_);

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    {
      throw GpuRuntimeError("Launch of infectivity power kernel failed", err);
    }

  // Now calculate infectivity
  blasStat_ = hipblasSgemv(cudaBLAS_, HIPBLAS_OP_N, numInfecs_, numSpecies_,
      &UNITY, devAnimalsInfPow_, animalsInfPowPitch_, devXi_, 1, &ZERO, devInfectivity_,
      1);
  if (blasStat_ != HIPBLAS_STATUS_SUCCESS)
    {
      std::cerr << "Error in infectivity: " << blasStat_ << std::endl;
    }

  float res;
  hipblasSasum(cudaBLAS_, numInfecs_, devInfectivity_, 1, &res);
  std::cerr << "Sum devInfectivity_ (GPU) = " << res << std::endl;
}

void
GpuLikelihood::CalcSusceptibility()
{
  // Calculates susceptibility powers and sums over suscept.
  hipGetLastError();
  int dimBlock(THREADSPERBLOCK);
  int dimGrid((popSize_ + THREADSPERBLOCK - 1) / THREADSPERBLOCK);
  calcSpecPow<<<dimGrid, dimBlock>>>(popSize_,numSpecies_,devAnimalsSuscPow_,animalsSuscPowPitch_, devAnimals_,animalsPitch_,devPhi_);

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    {
      throw GpuRuntimeError("Launch of susceptibility power kernel failed",
          err);
    }

  blasStat_ = hipblasSgemv(cudaBLAS_, HIPBLAS_OP_N, popSize_, numSpecies_, &UNITY,
      devAnimalsSuscPow_, animalsSuscPowPitch_, devZeta_, 1, &ZERO, devSusceptibility_, 1);
  if (blasStat_ != HIPBLAS_STATUS_SUCCESS)
    {
      std::cerr << "Error in susceptibility: " << blasStat_ << std::endl;
    }

  float res;
  hipblasSasum(cudaBLAS_, popSize_, devSusceptibility_, 1, &res);
  std::cerr << "Sum devSusceptibility_ (GPU) = " << res << std::endl;

}

inline
void
GpuLikelihood::CalcDistance()
{

  // Apply distance kernel to D_, place result in DT_.
  hipGetLastError();
  size_t blocksPerGrid = (dnnz_ + THREADSPERBLOCK - 1) / THREADSPERBLOCK;
  calcDT<<<blocksPerGrid,THREADSPERBLOCK>>>(devDVal_,devTVal_,dnnz_,devDTVal_, delta_);

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    throw GpuRuntimeError("Distance calculation failed", err);

  float res;
  hipblasSasum(cudaBLAS_, dnnz_, devTVal_, 1, &res);
  std::cerr << "Sum devTVal_ (GPU) = " << res << std::endl;

}

void
GpuLikelihood::CalcBgIntegral()
{
  // Get I1Time

  float I1 = gpuReduction<Min<float> >(devEventTimes_, numInfecs_);
  float sumI = gpuReduction<Plus<float> >(devEventTimes_, numInfecs_);

  bgIntegral_ = sumI - I1 * numInfecs_;
  bgIntegral_ += (obsTime_ - I1) * (realPopSize_ - numInfecs_);
  bgIntegral_ *= epsilon_;

  std::cerr << "Cuda sumI = " << sumI << std::endl;
  std::cerr << "Cuda I1 = " << I1 << std::endl;
}

void
GpuLikelihood::Calculate()
{

  CalcEvents();
  CalcInfectivity();
  CalcSusceptibility();
  CalcDistance();
  CalcBgIntegral();

  // DT * Susceptibility
  sparseStat_ = hipsparseScsrmv(cudaSparse_, HIPSPARSE_OPERATION_NON_TRANSPOSE,
      numInfecs_, popSize_, UNITY, crsDescr_, devDTVal_, devDRowPtr_,
      devDColInd_, devSusceptibility_, ZERO, devTmp_);
  if (sparseStat_ != HIPSPARSE_STATUS_SUCCESS)
    {
      std::cerr << "Error in hipsparseScsrmv() " << sparseStat_ << std::endl;
    }
  hipDeviceSynchronize();

  // infectivity * devTmp
  blasStat_ = hipblasSdot(cudaBLAS_, numInfecs_, devInfectivity_, 1, devTmp_, 1,
      &logLikelihood_); // May have an issue with 1-based indexing here!
  if (blasStat_ != HIPBLAS_STATUS_SUCCESS)
    {
      std::cerr << "Error in hipblasSdot() " << blasStat_ << std::endl;
    }
  hipDeviceSynchronize();

  logLikelihood_ *= gamma1_;
  logLikelihood_ += bgIntegral_;
  logLikelihood_ *= -1;
}

void
GpuLikelihood::UpdateDistance()
{

  // Now calculate infectivity
  blasStat_ = hipblasSgemv(cudaBLAS_, HIPBLAS_OP_N, numInfecs_, numSpecies_,
      &UNITY, devAnimalsInfPow_, animalsInfPowPitch_, devXi_, 1, &ZERO, devInfectivity_,
      1);
  if (blasStat_ != HIPBLAS_STATUS_SUCCESS)
    {
      std::cerr << "Error in infectivity: " << blasStat_ << std::endl;
    }

  // Calculate Susceptibility
  blasStat_ = hipblasSgemv(cudaBLAS_, HIPBLAS_OP_N, popSize_, numSpecies_, &UNITY,
      devAnimalsSuscPow_, animalsSuscPowPitch_, devZeta_, 1, &ZERO, devSusceptibility_, 1);
  if (blasStat_ != HIPBLAS_STATUS_SUCCESS)
    {
      std::cerr << "Error in susceptibility: " << blasStat_ << std::endl;
    }

  CalcDistance();

  // DT * Susceptibility
  sparseStat_ = hipsparseScsrmv(cudaSparse_, HIPSPARSE_OPERATION_NON_TRANSPOSE,
      numInfecs_, popSize_, UNITY, crsDescr_, devDTVal_, devDRowPtr_,
      devDColInd_, devSusceptibility_, ZERO, devTmp_);
  if (sparseStat_ != HIPSPARSE_STATUS_SUCCESS)
    {
      std::cerr << "Error in hipsparseScsrmv() " << sparseStat_ << std::endl;
    }

  // infectivity * devTmp
  blasStat_ = hipblasSdot(cudaBLAS_, numInfecs_, devInfectivity_, 1, devTmp_, 1,
      &logLikelihood_); // May have an issue with 1-based indexing here!
  if (blasStat_ != HIPBLAS_STATUS_SUCCESS)
    {
      std::cerr << "Error in hipblasSdot() " << blasStat_ << std::endl;
    }
  hipDeviceSynchronize();

  // Background integral
  CalcBgIntegral();

  logLikelihood_ *= gamma1_;
  logLikelihood_ += bgIntegral_;
  logLikelihood_ *= -1;
}

float
GpuLikelihood::LogLikelihood() const
{

  return logLikelihood_;
}

