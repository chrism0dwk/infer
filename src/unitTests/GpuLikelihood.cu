#include "hip/hip_runtime.h"
/*
 * GpuLikelihood.cpp
 *
 *  Created on: Feb 13, 2012
 *      Author: stsiab
 */
#include <stdexcept>
#include <string>
#include <iostream>
#include <cmath>
#include <math_functions.h>

#include "GpuLikelihood.hpp"

// Constants
const float UNITY = 1.0;
const float ZERO = 0.0;

class GpuRuntimeError : public std::exception
{
public:
  GpuRuntimeError(const std::string usrMsg, hipError_t cudaErr)
  {
    msg_ = "GPU Runtime Error: ";
    msg_ += usrMsg;
    msg_ += " (";
    msg_ += cudaErr;
    msg_ += ",";
    msg_ += hipGetErrorString(cudaErr);
    msg_ += ")";
  }
  ~GpuRuntimeError() throw ()
  {
  }
  ;
  const char*
  what() const throw ()
  {
    return msg_.c_str();
  }

private:
  std::string msg_;
};

// CUDA kernels

__global__ void
calcDT(float* D, float* T, int N, float* DT, float delta)
{
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < N)
    DT[idx] = delta / (delta * delta + D[idx]) * T[idx];
}

__global__ void
calcT(int infecSize, int nnz, int popSize, int* TRowPtr, int* TColInd, float* TVal,
    float* eventTimes, float gamma2, float obsTime)
{
  // Each thread calculates a row i of the sparse matrix -- probably not efficient!

  int i = threadIdx.x + blockDim.x * blockIdx.x;

  if (i < infecSize)
    {
      int begin = TRowPtr[i];
      int end = TRowPtr[i + 1];

      float Ii = eventTimes[i]; // First column  -- argument for row-major here, I would have thought.
      float Ni = eventTimes[popSize + i]; // Second column
      float Ri = eventTimes[popSize * 2 + i]; // Third column

      for (int j = begin; j < end; ++j)
        {
          float Ij = eventTimes[TColInd[j]];
          float Nj = eventTimes[TColInd[j]+popSize];

          float jMaxSuscep;
          jMaxSuscep = fminf(Nj,Ij);
          jMaxSuscep = fminf(jMaxSuscep, obsTime);
          float exposure = fminf(Ni, jMaxSuscep) - fminf(Ii, jMaxSuscep);
          exposure += gamma2 * (fminf(Ri, jMaxSuscep) - fminf(Ni, jMaxSuscep));
          TVal[j] = exposure;
        }
    }
}

__global__ void
calcSpecPow(const int size, const int nSpecies, float* specpow, const float* animals, const int stride,
    const float* powers)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if ((row < size) and (col < nSpecies))
    {
      specpow[col * size + row] = powf(animals[col * stride + row], powers[col]);
    }
}

GpuLikelihood::GpuLikelihood(const size_t popSize, const size_t numInfecs,
    const size_t nSpecies, const float obsTime, const size_t distanceNNZ) :
    popSize_(popSize), numInfecs_(numInfecs), numSpecies_(nSpecies), obsTime_(obsTime), devAnimals_(
        NULL), devAnimalsInfPow_(NULL), devAnimalsSuscPow_(NULL), devEventTimes_(
        NULL), devSusceptibility_(NULL), devInfectivity_(NULL), devDVal_(NULL), devDRowPtr_(
        NULL), devDColInd_(NULL), dnnz_(distanceNNZ), devTVal_(NULL), devDTVal_(
        NULL), devEVal_(NULL), devEColPtr_(NULL), devERowInd_(NULL), devTmp_(
        NULL), epsilon_(NULL), gamma1_(NULL), gamma2_(NULL), devXi_(NULL), devPsi_(
        NULL), devZeta_(NULL), devPhi_(NULL), delta_(NULL)
{

  int rv;

  std::cerr << "Pop size: " << popSize_ << std::endl;
  std::cerr << "Num infected: " << numInfecs_ << std::endl;
  std::cerr << "Num species: " << numSpecies_ << std::endl;

  hipDeviceSynchronize();

  // Allocate Event times - popSize_ * NUMEVENTS matrix
  rv = hipMalloc(&devEventTimes_, popSize_ * NUMEVENTS * sizeof(float));
  if (rv != hipSuccess)
    throw GpuRuntimeError("Cannot allocate memory for event times",
        (hipError_t) rv);

  // Allocate Animals_
  rv = hipMalloc(&devAnimals_, popSize_ * numSpecies_ * sizeof(float));
  if (rv != hipSuccess)
    throw GpuRuntimeError("Cannot allocate memory for animals",
        (hipError_t) rv);
  rv = hipMalloc(&devAnimalsSuscPow_, popSize_ * numSpecies_ * sizeof(float));
  if (rv != hipSuccess)
    throw GpuRuntimeError("Cannot allocate memory for animals susc pow",
        (hipError_t) rv);
  rv = hipMalloc(&devAnimalsInfPow_, numInfecs_ * numSpecies_ * sizeof(float));
  if (rv != hipSuccess)
    throw GpuRuntimeError("Cannot allocate memory for animals inf pow",
        (hipError_t) rv);

  // Allocate Distance_ CRS matrix
  rv = hipMalloc(&devDVal_, dnnz_ * sizeof(float));
  rv |= hipMalloc(&devDRowPtr_, (popSize_ + 1) * sizeof(int));
  rv |= hipMalloc(&devDColInd_, dnnz_ * sizeof(float));
  if (rv != hipSuccess)
    throw GpuRuntimeError("Cannot allocate distance matrix", (hipError_t) rv);

  // Allocate intermediate T and DT
  rv = hipMalloc(&devTVal_, dnnz_ * sizeof(float));
  rv |= hipMalloc(&devDTVal_, dnnz_ * sizeof(float));
  if (rv != hipSuccess)
    throw GpuRuntimeError("Cannot allocate temporary structures",
        (hipError_t) rv);

  // Allocate intermediate infectivity and susceptibility
  rv = hipMalloc(&devSusceptibility_, popSize_ * sizeof(float));
  rv |= hipMalloc(&devInfectivity_, numInfecs_ * sizeof(float));
  if (rv != hipSuccess)
    throw GpuRuntimeError("Cannot allocate temporary structures",
        (hipError_t) rv);

  // Allocate temporary vector
  rv = hipMalloc(&devTmp_, numInfecs_ * sizeof(float));
  if (rv != hipSuccess)
    throw GpuRuntimeError("Cannot allocate temporary structures",
        (hipError_t) rv);

  // Parameters
  rv = hipMalloc(&devXi_, numSpecies_ * sizeof(float));
  rv |= hipMalloc(&devPsi_, numSpecies_ * sizeof(float));
  rv |= hipMalloc(&devZeta_, numSpecies_ * sizeof(float));
  rv |= hipMalloc(&devPhi_, numSpecies_ * sizeof(float));
  if (rv != hipSuccess)
    throw GpuRuntimeError("Cannot allocate device parameters",
        (hipError_t) rv);

  // BLAS handles
  blasStat_ = hipblasCreate(&cudaBLAS_);
  if (blasStat_ != HIPBLAS_STATUS_SUCCESS)
    throw std::runtime_error("CUBLAS init failed");

  sparseStat_ = hipsparseCreate(&cudaSparse_);
  if (sparseStat_ != HIPSPARSE_STATUS_SUCCESS)
    throw std::runtime_error("CUSPARSE init failed");

  sparseStat_ = hipsparseCreateMatDescr(&crsDescr_);
  if (sparseStat_ != HIPSPARSE_STATUS_SUCCESS)
    throw std::runtime_error("CUSPARSE matrix descriptor init failed");
  hipsparseSetMatType(crsDescr_, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(crsDescr_, HIPSPARSE_INDEX_BASE_ZERO);

}

GpuLikelihood::~GpuLikelihood()
{
  if (devEventTimes_ != NULL)
    hipFree(devEventTimes_);
  if (devAnimals_ != NULL)
    hipFree(devAnimals_);
  if (devAnimalsSuscPow_ != NULL)
    hipFree(devAnimalsSuscPow_);
  if (devAnimalsInfPow_ != NULL)
    hipFree(devAnimalsInfPow_);
  if (devDVal_ != NULL)
    hipFree(devDVal_);
  if (devDRowPtr_ != NULL)
    hipFree(devDRowPtr_);
  if (devDColInd_ != NULL)
    hipFree(devDColInd_);
  if (devTVal_ != NULL)
    hipFree(devTVal_);
  if (devDTVal_ != NULL)
    hipFree(devDTVal_);
  if (devSusceptibility_ != NULL)
    hipFree(devSusceptibility_);
  if (devInfectivity_ != NULL)
    hipFree(devInfectivity_);
  if (devTmp_ != NULL)
    hipFree(devTmp_);

  if (devXi_)
    hipFree(devXi_);
  if (devPsi_)
    hipFree(devPsi_);
  if (devZeta_)
    hipFree(devZeta_);
  if (devPhi_)
    hipFree(devPhi_);

  hipblasDestroy(cudaBLAS_);
  hipsparseDestroy(cudaSparse_);
}

void
GpuLikelihood::SetEvents(const float* data)
{
  // Get event times into GPU memory
  hipError_t rv = hipMemcpy(devEventTimes_, data,
      popSize_ * NUMEVENTS * sizeof(float), hipMemcpyHostToDevice);
  if (rv != hipSuccess)
    throw GpuRuntimeError("Copying event times to device failed", rv);
}

void
GpuLikelihood::SetSpecies(const float* data)
{
  // Loads species data assuming **COL MAJOR**
  hipError_t rv = hipMemcpy(devAnimals_, data,
      popSize_ * numSpecies_ * sizeof(float), hipMemcpyHostToDevice);
  if (rv != hipSuccess)
    throw GpuRuntimeError("Copy species matrix to device failed", rv);

  rv = hipMemcpy(devAnimalsSuscPow_, data,
      popSize_ * numSpecies_ * sizeof(float), hipMemcpyHostToDevice);
  if (rv != hipSuccess)
    throw GpuRuntimeError("Copy species matrix to device failed", rv);

}

void
GpuLikelihood::SetDistance(const float* data, const int* rowptr,
    const int* colind)
{
  // Loads distance data into memory
  int rv = hipMemcpy(devDVal_, data, dnnz_ * sizeof(float),
      hipMemcpyHostToDevice);
  rv |= hipMemcpy(devDRowPtr_, rowptr, (popSize_ + 1) * sizeof(int),
      hipMemcpyHostToDevice);
  rv |= hipMemcpy(devDColInd_, colind, dnnz_ * sizeof(int),
      hipMemcpyHostToDevice);
  if (rv != hipSuccess)
    throw GpuRuntimeError("Copy of distance matrix to device failed",
        (hipError_t) rv);
}

void
GpuLikelihood::SetParameters(float* epsilon, float* gamma1, float* gamma2,
    float* xi, float* psi, float* zeta, float* phi, float* delta)
{
  epsilon_ = *epsilon;
  gamma1_ = *gamma1;
  gamma2_ = *gamma2;
  delta_ = *delta;

  hipError_t rv = hipMemcpy(devXi_, xi, numSpecies_ * sizeof(float),
      hipMemcpyHostToDevice);
  if (rv != hipSuccess)
    throw GpuRuntimeError("Error copying xi to GPU", rv);

  rv = hipMemcpy(devPsi_, psi, numSpecies_ * sizeof(float),
      hipMemcpyHostToDevice);
  if (rv != hipSuccess)
    throw GpuRuntimeError("Error copying psi to GPU", rv);

  rv = hipMemcpy(devZeta_, zeta, numSpecies_ * sizeof(float),
      hipMemcpyHostToDevice);
  if (rv != hipSuccess)
    throw GpuRuntimeError("Error copying zeta to GPU", rv);

  rv = hipMemcpy(devPhi_, phi, numSpecies_ * sizeof(float),
      hipMemcpyHostToDevice);
  if (rv != hipSuccess)
    throw GpuRuntimeError("Error copying psi to GPU", rv);
}

void
GpuLikelihood::CalcEvents()
{
  // Calculates the T_ matrix -- sparse matrix operation!
  hipGetLastError();
  size_t blocksPerGrid = (dnnz_ + THREADSPERBLOCK - 1) / THREADSPERBLOCK;
  std::cerr << "Calculating Events with block size " << THREADSPERBLOCK << " and blocks per grid " << blocksPerGrid << std::endl;
  calcT<<<blocksPerGrid,THREADSPERBLOCK>>>(numInfecs_, dnnz_, popSize_, devDRowPtr_, devDColInd_, devTVal_, devEventTimes_, gamma2_, obsTime_);

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    throw GpuRuntimeError("Error calculating events", err);

//  float* vals = new float[dnnz_];
//  int* rowptr = new int[popSize_];
//  int* colind = new int[dnnz_];
//
//  hipMemcpy(vals, devTVal_, dnnz_ * sizeof(float), hipMemcpyDeviceToHost);
//  hipMemcpy(rowptr, devDRowPtr_, popSize_ * sizeof(int),
//      hipMemcpyDeviceToHost);
//  hipMemcpy(colind, devDColInd_, dnnz_ * sizeof(int), hipMemcpyDeviceToHost);
//
//  std::cerr << "T_: ";
//  for (size_t i = 0; i < dnnz_; ++i)
//    std::cerr << vals[i] << " ";
//  std::cerr << std::endl;
//  std::cerr << "TColInd: ";
//  for (size_t i = 0; i < dnnz_; ++i)
//    std::cerr << colind[i] << " ";
//  std::cerr << std::endl;
//  std::cerr << "TRowPtr: ";
//  for (size_t i = 0; i < popSize_; ++i)
//    std::cerr << rowptr[i] << " ";
//  std::cerr << std::endl;
//
//  delete[] vals;

}

void
GpuLikelihood::CalcInfectivity()
{
  // Calculates infectivity powers and sums over infectivity
  hipGetLastError(); // Reset error status
  // First do devAnimalsInfPow_
  dim3 dimBlock(3, THREADSPERBLOCK);
  dim3 dimGrid(1, (numInfecs_ + THREADSPERBLOCK - 1) / THREADSPERBLOCK);
  calcSpecPow<<<dimGrid, dimBlock>>>(numInfecs_,numSpecies_,devAnimalsInfPow_,devAnimals_,popSize_,devPsi_);

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    {
      throw GpuRuntimeError("Launch of infectivity power kernel failed", err);
    }


  // Now calculate infectivity
  blasStat_ = hipblasSgemv(cudaBLAS_, HIPBLAS_OP_N, numInfecs_, numSpecies_,
      &UNITY, devAnimalsInfPow_, numInfecs_, devXi_, 1, &ZERO, devInfectivity_,
      1);
  if (blasStat_ != HIPBLAS_STATUS_SUCCESS)
    {
      std::cerr << "Error in infectivity: " << blasStat_ << std::endl;
    }

  float res;
  hipblasSasum(cudaBLAS_, numInfecs_, devInfectivity_, 1, &res);
  std::cerr << "Sum devInfectivity_ (GPU) = " << res << std::endl;
}

void
GpuLikelihood::CalcSusceptibility()
{
  // Calculates susceptibility powers and sums over suscept.
  hipGetLastError();
  dim3 dimBlock(3, THREADSPERBLOCK);
  dim3 dimGrid(1, (popSize_ + THREADSPERBLOCK - 1) / THREADSPERBLOCK);
  calcSpecPow<<<dimGrid, dimBlock>>>(popSize_,numSpecies_,devAnimalsSuscPow_,devAnimals_,popSize_,devPhi_);

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    {
      throw GpuRuntimeError("Launch of susceptibility power kernel failed",
          err);
    }


  blasStat_ = hipblasSgemv(cudaBLAS_, HIPBLAS_OP_N, popSize_, numSpecies_, &UNITY,
      devAnimalsSuscPow_, popSize_, devZeta_, 1, &ZERO, devSusceptibility_, 1);
  if (blasStat_ != HIPBLAS_STATUS_SUCCESS)
    {
      std::cerr << "Error in susceptibility: " << blasStat_ << std::endl;
    }

  float res;
  hipblasSasum(cudaBLAS_, popSize_, devSusceptibility_, 1, &res);
  std::cerr << "Sum devSusceptibility_ (GPU) = " << res << std::endl;

}

inline
void
GpuLikelihood::CalcDistance()
{

  // Apply distance kernel to D_, place result in DT_.
  hipGetLastError();
  size_t blocksPerGrid = (dnnz_ + THREADSPERBLOCK - 1) / THREADSPERBLOCK;
  calcDT<<<blocksPerGrid,THREADSPERBLOCK>>>(devDVal_,devTVal_,dnnz_,devDTVal_, delta_);

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    throw GpuRuntimeError("Distance calculation failed", err);

  float res;
  hipblasSasum(cudaBLAS_, dnnz_, devTVal_, 1, &res);
  std::cerr << "Sum devTVal_ (GPU) = " << res << std::endl;



}

void
GpuLikelihood::Calculate()
{

  CalcEvents();
  CalcInfectivity();
  CalcSusceptibility();
  CalcDistance();

  // DT * Susceptibility
  sparseStat_ = hipsparseScsrmv(cudaSparse_, HIPSPARSE_OPERATION_NON_TRANSPOSE,
      numInfecs_, popSize_, dnnz_, &UNITY, crsDescr_, devDTVal_, devDRowPtr_,
      devDColInd_, devSusceptibility_, &ZERO, devTmp_);
  if (sparseStat_ != HIPSPARSE_STATUS_SUCCESS)
    {
      std::cerr << "Error in hipsparseScsrmv() " << sparseStat_ << std::endl;
    }
  hipDeviceSynchronize();

  // infectivity * devTmp
  blasStat_ = hipblasSdot(cudaBLAS_, numInfecs_, devInfectivity_, 1, devTmp_, 1,
      &logLikelihood_); // May have an issue with 1-based indexing here!
  if (blasStat_ != HIPBLAS_STATUS_SUCCESS)
    {
      std::cerr << "Error in hipblasSdot() " << blasStat_ << std::endl;
    }
  hipDeviceSynchronize();
}

void
GpuLikelihood::UpdateDistance()
{
  CalcDistance();

  // DT * Susceptibility
  sparseStat_ = hipsparseScsrmv(cudaSparse_, HIPSPARSE_OPERATION_NON_TRANSPOSE,
      numInfecs_, popSize_, dnnz_, &UNITY, crsDescr_, devDTVal_, devDRowPtr_,
      devDColInd_, devSusceptibility_, &ZERO, devTmp_);
  if (sparseStat_ != HIPSPARSE_STATUS_SUCCESS)
    {
      std::cerr << "Error in hipsparseScsrmv() " << sparseStat_ << std::endl;
    }

  // infectivity * devTmp
  blasStat_ = hipblasSdot(cudaBLAS_, numInfecs_, devInfectivity_, 1, devTmp_, 1,
      &logLikelihood_); // May have an issue with 1-based indexing here!
  if (blasStat_ != HIPBLAS_STATUS_SUCCESS)
    {
      std::cerr << "Error in hipblasSdot() " << blasStat_ << std::endl;
    }
  hipDeviceSynchronize();

  logLikelihood_ *= -gamma1_;
}

float
GpuLikelihood::LogLikelihood() const
{

  return logLikelihood_;
}
