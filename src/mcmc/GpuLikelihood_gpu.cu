#include "hip/hip_runtime.h"
/*
 * Gpulikelihood.cpp
 *
 *  Created on: Feb 13, 2012
 *      Author: stsiab
 */
#include <stdexcept>
#include <string>
#include <cstddef>
#include <iostream>
#include <sstream>
#include <vector>
#include <utility>
#include <cmath>
#include <cassert>
#include <math_functions.h>
#include <hip/device_functions.h>
#include <sys/time.h>
#include <thrust/sort.h>
#include <thrust/count.h>
#include <thrust/find.h>
#include <gsl/gsl_cdf.h>

#ifndef __HIPCC__
#define __HIPCC__
#endif

//#define ALPHA 0.3

#include "GpuLikelihood.hpp"


#define T2P(thrustvec) thrust::raw_pointer_cast(&(thrustvec)[0])

namespace EpiRisk
{
  class GpuRuntimeError : public std::exception
  {
  public:
    GpuRuntimeError(const std::string usrMsg, hipError_t cudaErr)
    {
      msg_ = "GPU Runtime Error: ";
      msg_ += usrMsg;
      msg_ += " (";
      msg_ += cudaErr;
      msg_ += ",";
      msg_ += hipGetErrorString(cudaErr);
      msg_ += ")";
    }
    ~GpuRuntimeError() throw ()
    {
    }
    ;
    const char*
    what() const throw ()
    {
      return msg_.c_str();
    }

  private:
    std::string msg_;
  };

#define checkCudaError(err) __checkCudaError(err, __FILE__, __LINE__)

  void
  __checkCudaError(const hipError_t err, const char* file, const int line)
  {
    if (err != hipSuccess)
      {
        std::stringstream s;
        s << file << "(" << line << ") : Cuda Runtime error ";
        throw GpuRuntimeError(s.str(), err);
      }
  }

  // CUDA kernels

  __device__
  float cache[5];

  template<typename T>
  struct Log
  {
    __host__ __device__
    T
    operator()(const T& val) const
    {
      return logf(val);
    }
  };

  template<typename T>
  struct LessThanZero
  {
    __host__ __device__
    bool
    operator()(const T& val) const
    {
      return val < 0;
    }
  };


  template<typename T>
  struct IndirectMin
  {
    __host__ __device__
    IndirectMin(T* ptr) :
      ptr_(ptr)
    {
    }
    ;

    __host__ __device__
    bool
    operator()(const InfecIdx_t& lhs, const InfecIdx_t& rhs) const
    {
      return ptr_[lhs.ptr] < ptr_[rhs.ptr];
    }
  private:
    T* ptr_;
  };

  __device__ float
  _h(const float t, float nu, float alpha)
  {
    // Returns a logistic 'h' function
    //return 1.0f / (1.0f + expf(-nu*(t-alpha)));
    //return exp(nu*t) / ( alpha + exp(nu*t));
    //return nu*nu*t*exp(-nu*t);
    
    return t < alpha ? 0.0f : 1.0f;
  }

  __device__ float
  _H(const float t, float nu, float alpha)
  {
    // Returns the integral of the 'h' function over [0,t]

    //float integral = 1.0f / nu * logf( (1.0f + expf(nu*(t - alpha))) / (1.0f + expf(-nu*alpha)));

    //float integral = 1.0f / nu * logf( (alpha + expf(nu*t)) / (1.0f + alpha));
    //float integral = -nu * t * exp(-nu * t) - exp(-nu * t) + 1;
    
    float integral = t - alpha;
    return fmaxf(0.0f, integral);
  }

  __device__ float
  _K(const float dsq, const float delta, const float omega)
  {
    return delta / powf(delta*delta + dsq, omega);
  }

  __device__ float
  _atomicAdd(float* address, float val)
  {
    unsigned int* address_as_ui = (unsigned int*) address;
    unsigned int old = *address_as_ui, assumed;
    do
      {
        assumed = old;
        old = atomicCAS(address_as_ui, assumed,
			__float_as_int(val + __int_as_float(assumed)));
      }
    while (assumed != old);
    return __int_as_float(old);
  }

  template<typename T>
  __device__
  void
  _shmemReduce(T* buff)
  {
    // Reduce buffer into output
    __syncthreads();

    for (unsigned int size = blockDim.x / 2; size > 32; size >>= 1)
      {
	if (threadIdx.x < size)
	  buff[threadIdx.x] += buff[threadIdx.x + size];
	__syncthreads();
      }
    if (threadIdx.x < 32)
      {
	volatile T* vbuff = buff;
	vbuff[threadIdx.x] += vbuff[threadIdx.x + 32];
	vbuff[threadIdx.x] += vbuff[threadIdx.x + 16];
	vbuff[threadIdx.x] += vbuff[threadIdx.x + 8];
	vbuff[threadIdx.x] += vbuff[threadIdx.x + 4];
	vbuff[threadIdx.x] += vbuff[threadIdx.x + 2];
	vbuff[threadIdx.x] += vbuff[threadIdx.x + 1];
      }

    __syncthreads();
  }

  __global__
  void
  _computeDistance(const float2* coords, int* output, size_t outputPitch,
		   const float distance, const int n)
  {
    // Launch this kernel with block dim [THREADSPERBLOCK,1]

    // Get global row/col
    int col = threadIdx.x + blockDim.x * blockIdx.x;
    int row = threadIdx.x + blockIdx.y * blockDim.x;
    float dsq = distance * distance;
    __shared__
      float2 ybuff[THREADSPERBLOCK];
    __shared__
      int nnzbuff[THREADSPERBLOCK];

    nnzbuff[threadIdx.x] = 0.0f;

    if (row < n)
      {
        ybuff[threadIdx.x] = coords[row];
      }
    __syncthreads();


    if (col < n)
      {
        float2 x = coords[col];
        int rowlimit = min(blockDim.x, n - blockIdx.y * blockDim.x);

        for (int myrow = 0; myrow < rowlimit; myrow++)
          {
            float2 y = ybuff[myrow];
            float dx = x.x - y.x;
            float dy = x.y - y.y;
            float d = dx * dx + dy * dy; //hypotf(dx, dy);
            nnzbuff[threadIdx.x] += d <= dsq and d > 0.0f;
          }
      }

    _shmemReduce<int>(nnzbuff);
    int* rowptr = (int*) ((char*) output + blockIdx.y * outputPitch);
    rowptr[blockIdx.x] = nnzbuff[0];

  }

  __global__
  void
  _computeDrow(const float2* coords, float* devDrow, unsigned int* devIsValid,
	       const int n, const int row, const float distance)
  {
    int col = threadIdx.x + blockDim.x * blockIdx.x;

    if (col < n)
      {
        float dsq = distance * distance;
        float2 y = coords[row];
        float2 x = coords[col];
        float dx = x.x - y.x;
        float dy = x.y - y.y;
        float d = dx * dx + dy * dy; //hypotf(dx,dy);// Require squared distance here!
        devDrow[col] = d;
        devIsValid[col] = (0.0f < d) and (d <= dsq) ? 1 : 0;
      }

  }

  __global__
  void
  _fillIndex(int* index, const size_t n)
  {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n)
      index[i] = i;
  }

  size_t
  distanceNumValid(const float2* devCoords, const size_t n, const float dLimit)
  {
    int* devNNZ;
    size_t devNNZPitch;
    int* hostNNZ;

    // Get Grid dimensions
    dim3 numThreads(THREADSPERBLOCK, 1);
    size_t numBlocks_xy = (n + THREADSPERBLOCK - 1) / THREADSPERBLOCK;
    dim3 numBlocks(numBlocks_xy, numBlocks_xy);

    checkCudaError(
		   hipMallocPitch(&devNNZ, &devNNZPitch, numBlocks_xy*sizeof(int), numBlocks_xy));
    checkCudaError(
		   hipMemset2DAsync(devNNZ, devNNZPitch, 0, numBlocks_xy*sizeof(int), numBlocks_xy));

    hostNNZ = new int[numBlocks_xy * numBlocks_xy];

    // Now launch the calculating kernel
    _computeDistance<<<numBlocks,numThreads>>>(devCoords, devNNZ, devNNZPitch, dLimit, n);
    hipDeviceSynchronize();
    checkCudaError(hipGetLastError());

    // Copy memory back to host
    checkCudaError(
		   hipMemcpy2D(hostNNZ, numBlocks_xy*sizeof(int), devNNZ, devNNZPitch, numBlocks_xy*sizeof(int), numBlocks_xy, hipMemcpyDeviceToHost));

    // Reduction on the host
    int i;
    size_t nnz = 0;
#pragma omp parallel for reduction(+:nnz)
    for (i = 0; i < numBlocks_xy * numBlocks_xy; ++i)
      {
        nnz += hostNNZ[i];
      }

    // Destroy memory
    checkCudaError(hipFree(devNNZ));
    delete[] hostNNZ;

    return nnz;
  }

  CsrMatrix*
  makeSparseDistance(const float2* coords, const size_t n, const float dLimit)
  {
    // Constructs a sparse matrix

    float2* devCoords;
    checkCudaError(hipMalloc(&devCoords, n*sizeof(float2)));
    checkCudaError(
		   hipMemcpy(devCoords, coords, n*sizeof(float2), hipMemcpyHostToDevice));

    // Get number of valid (ie 0 < d <= dLimit) entries
    size_t nnz = distanceNumValid(devCoords, n, dLimit);
    // Allocate the sparse matrix -- may bomb out!
    CsrMatrix* csrMatrix = new CsrMatrix;
    csrMatrix->rowPtr = NULL;
    csrMatrix->colInd = NULL;
    csrMatrix->val = NULL;
    csrMatrix->nnz = nnz;
    csrMatrix->m = n;
    csrMatrix->n = n;

    float* devDrow = NULL;
    int* devColNums = NULL;
    unsigned int* devIsValid = NULL;

    try
      {
        hipMalloc(&csrMatrix->rowPtr, (n + 1) * sizeof(size_t));
        hipMalloc(&csrMatrix->colInd, nnz * sizeof(int));
        hipMalloc(&csrMatrix->val, nnz * sizeof(int));
        hipMalloc(&devDrow, n * sizeof(float));
        hipMalloc(&devColNums, n * sizeof(int));
        hipMalloc(&devIsValid, n * sizeof(unsigned int));
      }
    catch (runtime_error& e)
      {
        if (csrMatrix->rowPtr)
          hipFree(csrMatrix->rowPtr);
        if (csrMatrix->colInd)
          hipFree(csrMatrix->colInd);
        if (csrMatrix->val)
          hipFree(csrMatrix->val);
        if (devDrow)
          hipFree(devDrow);
        if (devColNums)
          hipFree(devColNums);
        if (devIsValid)
          hipFree(devIsValid);
        throw e;
      }

    // For each row of the distance matrix
    // 1) Calculate it, outputting valid flags, and indices in output array
    // 2) Compact col and val into respective arrays
    // 3) Enter the rowptr

    // CUDPP bits
    CUDPPHandle theCudpp;
    CUDPPResult result = cudppCreate(&theCudpp);
    if (result != CUDPP_SUCCESS)
      {
        throw runtime_error("Could not create the CUDPP instance");
      }

    // Compact plan
    CUDPPConfiguration compactFloatConfig;
    compactFloatConfig.algorithm = CUDPP_COMPACT;
    compactFloatConfig.datatype = CUDPP_FLOAT;
    compactFloatConfig.options = CUDPP_OPTION_FORWARD;
    CUDPPConfiguration compactIntConfig;
    compactIntConfig.algorithm = CUDPP_COMPACT;
    compactIntConfig.datatype = CUDPP_INT;
    compactIntConfig.options = CUDPP_OPTION_FORWARD;
    CUDPPHandle compactFloatPlan;
    result = cudppPlan(theCudpp, &compactFloatPlan, compactFloatConfig, n, 1,
		       0);
    if (result != CUDPP_SUCCESS)
      cerr << "Help!  Could not create float plan!" << endl;
    CUDPPHandle compactIntPlan;
    result = cudppPlan(theCudpp, &compactIntPlan, compactIntConfig, n, 1, 0);
    if (result != CUDPP_SUCCESS)
      cerr << "Help! Could not create int plan!" << endl;
    size_t *numValid, *devNumValid;
    checkCudaError(
		   hipHostAlloc(&numValid, sizeof(size_t), hipHostMallocMapped));
    checkCudaError(hipHostGetDevicePointer(&devNumValid, numValid, 0));
    int* hostRowptr = new int[n + 1];
    hostRowptr[0] = 0;

    int numBlocks = (n + THREADSPERBLOCK - 1) / THREADSPERBLOCK;
    _fillIndex<<<numBlocks, THREADSPERBLOCK>>>(devColNums, n);

    for (int row = 0; row < n; ++row)
      {
        // Compute distances, record valid entries
	_computeDrow<<<numBlocks, THREADSPERBLOCK>>>(devCoords, devDrow, devIsValid, n, row, dLimit);
	checkCudaError(hipGetLastError());

        // Compact into col
        cudppCompact(compactFloatPlan, csrMatrix->val + hostRowptr[row],
		     devNumValid, devDrow, devIsValid, n);
        cudppCompact(compactIntPlan, csrMatrix->colInd + hostRowptr[row],
		     devNumValid, devColNums, devIsValid, n);
        hipDeviceSynchronize();

        // Update rowptr
        hostRowptr[row + 1] = hostRowptr[row] + (int) *numValid;
      }

    checkCudaError(
		   hipMemcpy(csrMatrix->rowPtr, hostRowptr, (n+1)*sizeof(int), hipMemcpyHostToDevice));

    // Clean up
    hipFree(devDrow);
    hipFree(devIsValid);
    hipFree(devColNums);
    hipHostFree(numValid);
    delete[] hostRowptr;

    cudppDestroyPlan(compactFloatPlan);
    cudppDestroyPlan(compactIntPlan);
    cudppDestroy(theCudpp);

    hipFree(devCoords);

    hipDeviceSynchronize();
    return csrMatrix;
  }

  void
  destroyCsrMatrix(CsrMatrix* csrMatrix)
  {
    checkCudaError(hipFree(csrMatrix->val));
    checkCudaError(hipFree(csrMatrix->colInd));
    checkCudaError(hipFree(csrMatrix->rowPtr));

    delete csrMatrix;
  }


  int
  checkDistMatrixSymmetry(const CsrMatrix* csrMatrix)
  {

    int row;
    int nonsymmetric = 0;
#pragma omp parallel for shared(csrMatrix) private(row) reduction(+:nonsymmetric)
    for(row=0; row<csrMatrix->n; ++row)
      {
        int rowptr = csrMatrix->rowPtr[row];
        int cRowLen = csrMatrix->rowPtr[row+1] - rowptr;

        for(size_t colidx=0; colidx<cRowLen; ++colidx)
          {
            int colnum = csrMatrix->colInd[rowptr + colidx];
            float rtoc = csrMatrix->val[rowptr + colidx];
            float ctor;
            bool rv = getDistMatrixElement(colnum,row,csrMatrix,&ctor);

	    if(rtoc != ctor or !rv) {
#pragma omp critical
	      {
		cerr << "Non-symmetry: (" << row << "," << colnum << ") = " << rtoc
		     << " but (" << colnum << "," << row << ") = " << ctor;
		cerr << endl;
	      }
	      nonsymmetric += 1;
	    }
          }
      }

    return nonsymmetric;
  }


  __global__ void
  _sanitizeEventTimes(float* data, int pitch, const float time, const int size)
  {
    // Ensures Ii <= Ni <= Ri for individual i
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    if (tid < size)
      {
        float R = data[tid + pitch * 2];
        float N = data[tid + pitch];
        float I = data[tid];

        R = fminf(R, time);
        N = fminf(N, R);
        I = fminf(I, N);

        data[tid + pitch * 2] = R;
        data[tid + pitch] = N;
        data[tid] = I;
      }
  }

  __global__ void
  _calcIntegral(const InfecIdx_t* infecIdx, const int infecSize,
		const CsrMatrix distance, float* eventTimes, const int eventTimesPitch,
		const float* susceptibility, const float* infectivity, const float gamma2,
		const float delta, const float omega, const float nu, const float alpha, float* output)
  {
    // Each warp calculates a row i of the sparse matrix

    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int row = tid / 32; // Global Warp id
    int lane = tid & (32 - 1); // Id within a warp

    __shared__
      float buff[THREADSPERBLOCK];

    buff[threadIdx.x] = 0.0f;

    if (row < infecSize)
      {
        int i = infecIdx[row].ptr;

        int begin = distance.rowPtr[i];
        int end = distance.rowPtr[i + 1];
        float Ii = eventTimes[i];
        float Ni = eventTimes[i + eventTimesPitch];
        float Ri = eventTimes[i + eventTimesPitch * 2];

        float threadSum = 0.0f;
        for (int jj = begin + lane; jj < end; jj += 32)
          {
            // Integrated infection pressure
            float Ij = eventTimes[distance.colInd[jj]];
            float betaij = _H(fminf(Ni, Ij) - fminf(Ii, Ij), nu, alpha);
            betaij += gamma2
	      * (_H(fminf(Ri, Ij) - Ii, nu, alpha)
		 - _H(fminf(Ni, Ij) - Ii, nu, alpha));

            // Apply distance kernel and suscep
            betaij *= _K(distance.val[jj], delta, omega);
            betaij *= susceptibility[distance.colInd[jj]];
            threadSum += betaij;
          }
        buff[threadIdx.x] = threadSum * infectivity[i];
      }

    // Reduce all warp sums and write to global memory.

    _shmemReduce(buff);

    if (threadIdx.x == 0)
      {
        output[blockIdx.x] = buff[0];
      }
  }

  __global__ void
  _bgIntegral(float* output, const float* eventTimes, const int popSize,
	      const float epsilon1, const float epsilon2, const float movtBan,
	      const float I1Time)
  {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    __shared__
      float buff[THREADSPERBLOCK];
    buff[threadIdx.x] = 0.0f;

    if (tid < popSize)
      {
        float I = eventTimes[tid];
        buff[threadIdx.x] = epsilon1 * max((min(I, movtBan) - I1Time),0.0f);
        buff[threadIdx.x] += epsilon1 * epsilon2 * max(I - max(movtBan,I1Time), 0.0f);
      }

    _shmemReduce(buff);

    if(threadIdx.x == 0) output[blockIdx.x] = buff[0];
  }

  __global__ void
  _calcProduct(const InfecIdx_t* infecIdx, const int infecSize,
	       const CsrMatrix distance, const float* eventTimes,
	       const int eventTimesPitch, const float* susceptibility,
	       const float* infectivity, const float epsilon1, const float epsilon2,
	       const float gamma1, const float gamma2, const float delta, const float omega, const float nu,
	       const float alpha, const float movtBan, float* prodCache)
  {
    // Each warp calculates a row of the sparse matrix

    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int gwid = tid / 32; // Global Warp id
    int lane = tid & (32 - 1); // Id within a warp

    __shared__
      float threadProdCache[THREADSPERBLOCK];
    threadProdCache[threadIdx.x] = 0.0f;
    int row = gwid;

    if (row < infecSize)
      {
        int j = infecIdx[row].ptr;

        int begin = distance.rowPtr[j];
        int end = distance.rowPtr[j + 1];

        float Ij = eventTimes[j];

        for (int ii = begin + lane; ii < end; ii += 32)
          {
            int i = distance.colInd[ii];
            float Ii = eventTimes[i];
            float Ni = eventTimes[eventTimesPitch + i];
            float Ri = eventTimes[eventTimesPitch * 2 + i];

            if (Ii < Ni)
              {
                float idxOnj = 0.0f;
                if (Ii < Ij and Ij <= Ni)
                  idxOnj += _h(Ij - Ii, nu, alpha);
                else if (Ni < Ij and Ij <= Ri)
                  idxOnj += gamma2 * _h(Ij - Ii, nu, alpha);
                threadProdCache[threadIdx.x] += idxOnj * infectivity[i] * _K(distance.val[ii],delta,omega);
              }
          }
        __syncthreads();

        // Reduce semi-products into productCache
        volatile float* vThreadProdCache = threadProdCache;
        if (lane < 16)
          {
            vThreadProdCache[threadIdx.x] += vThreadProdCache[threadIdx.x + 16];
            vThreadProdCache[threadIdx.x] += vThreadProdCache[threadIdx.x + 8];
            vThreadProdCache[threadIdx.x] += vThreadProdCache[threadIdx.x + 4];
            vThreadProdCache[threadIdx.x] += vThreadProdCache[threadIdx.x + 2];
            vThreadProdCache[threadIdx.x] += vThreadProdCache[threadIdx.x + 1];
          }
        __syncthreads();

        // Write out to global memory
        if (lane == 0)
          {
            float epsilon = Ij < movtBan ? epsilon1 : (epsilon1*epsilon2);
            prodCache[j] = threadProdCache[threadIdx.x] * susceptibility[j]
	      * gamma1 + epsilon;
          }
      }
  }

  __global__ void
  _calcSpecPow(const unsigned int size, const int nSpecies, float* specpow,
	       const int specpowPitch, const float* animals, const int animalsPitch,
	       const float* powers)
  {
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < size)
      {
        for (unsigned int col = 0; col < nSpecies; ++col)
          {
            specpow[col * specpowPitch + row] = powf(
						     animals[col * animalsPitch + row], powers[col]);
          }
      }
  }




  __global__
  void
  _knownInfectionsLikelihood(const InfecIdx_t* infecIdx,
			     const unsigned int knownInfecs, const float* eventTimes,
			     const int eventTimesPitch, const float a, const float b,
			     float* reductionBuff)
  {
    extern
      __shared__ float buff[];

    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    buff[threadIdx.x] = 0.0f;

    if (tid < knownInfecs)
      {
        int i = infecIdx[tid].ptr;
        float Ii = eventTimes[i];
        float Ni = eventTimes[eventTimesPitch + i];
        float d = Ni - Ii;
        buff[threadIdx.x] = logf(powf(b, a) * powf(d, a - 1) * expf(-d * b));
      }

    _shmemReduce(buff);

    if (threadIdx.x == 0)
      reductionBuff[blockIdx.x] = buff[0];
  }

  __global__
  void
  _knownInfectionsLikelihoodPNC(const InfecIdx_t* infecIdx,
				const unsigned int knownInfecs, const float* eventTimes,
				const int eventTimesPitch, const float a, const float oldGamma,
				const float newGamma, const float* rns, const float prob,
				float* reductionBuff)
  {
    extern
      __shared__ float buff[];

    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    buff[threadIdx.x] = 0.0f;

    if (tid < knownInfecs)
      {
	int i = infecIdx[tid].ptr;
	float Ii = eventTimes[i];
	float Ni = eventTimes[eventTimesPitch + i];
	float d = Ni - Ii;
        buff[threadIdx.x] = (powf(newGamma, prob) / powf(oldGamma, prob - 1.0f) - newGamma) * d
	  + a * (1 - prob) * log( newGamma / oldGamma );
      }

    _shmemReduce(buff);

    if (threadIdx.x == 0)
      reductionBuff[blockIdx.x] = buff[0];
  }

  __global__
  void
  _nonCentreInfecTimes(const InfecIdx_t* index, const int size,
		       float* eventTimes, int eventTimesPitch, const float factor,
		       const float* toCentre, const float prop)
  {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < size)
      {
	unsigned int i = index[tid].ptr;
	float notification = eventTimes[i + eventTimesPitch];
	float infection = eventTimes[i];
	eventTimes[i] = notification - (notification - infection) * powf(factor,prop);
      }
  }

  __global__
  void
  _collectInfectiousPeriods(const InfecIdx_t* index, const int size,
			    const float* eventTimes, const int eventTimesPitch, float* output)
  {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < size)
      {
        int i = index[tid].ptr;
        float infecPeriod = eventTimes[eventTimesPitch + i] - eventTimes[i];
        output[tid] = infecPeriod;
      }
  }

  __global__
  void
  _logTransform(const float* input, const int size, float* output)
  {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < size)
      output[tid] = logf(input[tid]);
  }

  __global__
  void
  _indirectedSum(const InfecIdx_t* index, const int size, const float* data,
		 float* output)
  {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    extern
      __shared__ float buff[];
    buff[threadIdx.x] = 0.0f;

    if (tid < size)
      {
        buff[threadIdx.x] = data[index[tid].ptr];
        _shmemReduce(buff);
      }
    if (threadIdx.x == 0)
      output[blockIdx.x] = buff[0];
  }

  float
  indirectedSum(const InfecIdx_t* index, const int size, const float* data)
  {
    int numBlocks = (size + THREADSPERBLOCK - 1) / THREADSPERBLOCK;
    thrust::device_vector<float> output(numBlocks);

    _indirectedSum<<<numBlocks, THREADSPERBLOCK, THREADSPERBLOCK*sizeof(float)>>>(index, size, data, thrust::raw_pointer_cast(&output[0]));
    checkCudaError(hipGetLastError());
    return thrust::reduce(output.begin(), output.end());

  }

  template<int STRIDE>
  __global__
  void
  _reducePVectorStage1(float* input, const int size, const int I1Idx,
		       float* output)
  {
    int dataBlk = blockIdx.x*blockDim.x*STRIDE;
    
    extern
      __shared__ float buff[];
    buff[threadIdx.x] = 0.0f;

    if(I1Idx / (blockDim.x*STRIDE) == blockIdx.x and threadIdx.x==0)
	input[I1Idx] = 1.0f;

    for(int ii=0; ii<STRIDE; ++ii) {
      int i = dataBlk+blockDim.x*ii+threadIdx.x;
      if(i < size) buff[threadIdx.x] += logf(input[i]);
    }

    _shmemReduce(buff);
    if (threadIdx.x == 0)
      output[blockIdx.x] = buff[0];
  }

  void
  GpuLikelihood::ReduceProductVector()
  {
    // Reduces the device-side product vector into the device-side components struct

    int blocksPerGrid = (devProduct_->size() + THREADSPERBLOCK * 4 - 1)
      / (THREADSPERBLOCK*4);

    _reducePVectorStage1<4><<<blocksPerGrid, THREADSPERBLOCK, THREADSPERBLOCK * sizeof(float)>>>
      (thrust::raw_pointer_cast(&(*devProduct_)[0]),
       devProduct_->size(),
       I1Idx_,
       thrust::raw_pointer_cast(&(*devWorkspace_)[0]));
    checkCudaError(hipGetLastError());

    if(blocksPerGrid > 1) {
      CUDPPResult res = cudppReduce(addReduce_,
				    (float*) ((char*) devComponents_
					      + offsetof(LikelihoodComponents,logProduct)),
				    T2P(*devWorkspace_), blocksPerGrid);
      if (res != CUDPP_SUCCESS)
	throw std::runtime_error(
				 "cudppReduce failed in GpuLikelihood::ReduceProductVector()");
    }
    else
      checkCudaError(hipMemcpy(&(devComponents_->logProduct), 
				T2P(*devWorkspace_), 
				sizeof(float), hipMemcpyDeviceToDevice));

  }

  GpuLikelihood::GpuLikelihood(PopDataImporter& population,
			       EpiDataImporter& epidemic, 
			       const size_t nSpecies, const float obsTime,
			       const float dLimit, const bool occultsOnlyDC,
			       const int gpuId) :
    Likelihood(population, epidemic, nSpecies, obsTime, occultsOnlyDC),
    I1Time_(0.0), I1Idx_(0), covariateCopies_(0)
  {

    // Get GPU details
    int deviceId;
    if(gpuId > -1) checkCudaError(hipSetDevice(gpuId));
    checkCudaError(hipGetDevice(&deviceId));
    hipDeviceProp_t deviceProp;
    checkCudaError(hipGetDeviceProperties(&deviceProp, deviceId));
    int runtimeVersion;
    checkCudaError(hipRuntimeGetVersion(&runtimeVersion));

    // Print device properties
    std::cerr << "Initialised CUDA environment. CUDA runtime version " << runtimeVersion << "\n";
    std::cerr << "Using GPU: " << deviceProp.name << ", id " << deviceId << ", located at PCI bus ID " << deviceProp.pciBusID << "\n";
    std::cerr << "Free memory: " << deviceProp.totalGlobalMem << "\n";

#ifndef NDEBUG
    std::cout << "Using GPGPU: " << deviceProp.name << ", id " << deviceId
	      << ", located at PCI bus ID " << deviceProp.pciBusID << "\n";
#endif

    checkCudaError(hipSetDeviceFlags(hipDeviceMapHost));

    // Calculate distance matrix
    CalcDistanceMatrix(dLimit);

    // Allocate infec indicies
    hostInfecIdx_ = new thrust::host_vector<InfecIdx_t>;
    devInfecIdx_ = new thrust::device_vector<InfecIdx_t>;
    hostSuscOccults_ = new thrust::host_vector<InfecIdx_t>;
    for(size_t i = numKnownInfecs_; i < maxInfecs_; ++i)
      hostSuscOccults_->push_back(i);

    // Set up on GPU
    SetSpecies();
    SetEvents();

    // Set up reference counter to covariate data
    covariateCopies_ = new size_t;
    *covariateCopies_ = 1;

    // Allocate product cache
    devProduct_ = new thrust::device_vector<float>;
    devProduct_->resize(maxInfecs_);
    thrust::fill(devProduct_->begin(), devProduct_->end(), 1.0f);
    
    // Allocate integral array
    int numRequiredThreads = maxInfecs_ * 32; // One warp per infection
    integralBuffSize_ = (numRequiredThreads + THREADSPERBLOCK - 1)
      / THREADSPERBLOCK;
    devWorkspace_ = new thrust::device_vector<float>;
    devWorkspace_->resize(maxInfecs_);
    checkCudaError(hipMalloc(&devScratch_, SCRATCHSIZE*sizeof(FP_t)));

    // Components
    checkCudaError(
		   hipHostAlloc((void**)&hostComponents_, sizeof(LikelihoodComponents), hipHostMallocMapped));

    hostComponents_->bgIntegral = 0.0f;
    hostComponents_->integral = 0.0f;
    hostComponents_->sumI = 0.0f;
    hostComponents_->logProduct = 0.0f;

    checkCudaError(
		   hipHostGetDevicePointer(&devComponents_, hostComponents_, 0));

    // Parameters
    checkCudaError(hipMalloc(&devXi_, numSpecies_ * sizeof(float)));
    checkCudaError(hipMalloc(&devPsi_, numSpecies_ * sizeof(float)));
    checkCudaError(hipMalloc(&devZeta_, numSpecies_ * sizeof(float)));
    checkCudaError(hipMalloc(&devPhi_, numSpecies_ * sizeof(float)));

    // BLAS handles
    blasStat_ = hipblasCreate(&cudaBLAS_);
    if (blasStat_ != HIPBLAS_STATUS_SUCCESS)
      throw std::runtime_error("CUBLAS init failed");

    sparseStat_ = hipsparseCreate(&cudaSparse_);
    if (sparseStat_ != HIPSPARSE_STATUS_SUCCESS)
      throw std::runtime_error("CUSPARSE init failed");

    sparseStat_ = hipsparseCreateMatDescr(&crsDescr_);
    if (sparseStat_ != HIPSPARSE_STATUS_SUCCESS)
      throw std::runtime_error("CUSPARSE matrix descriptor init failed");
    hipsparseSetMatType(crsDescr_, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(crsDescr_, HIPSPARSE_INDEX_BASE_ZERO);

    // RNG for non-centering
    hiprandStatus_t hiprandStatus = hiprandCreateGenerator(&cuRand_,
							HIPRAND_RNG_PSEUDO_DEFAULT);
    if (hiprandStatus != HIPRAND_STATUS_SUCCESS)
      {
        throw std::runtime_error("CURAND init failed");
      }
    hiprandStatus = hiprandSetPseudoRandomGeneratorSeed(cuRand_, 0ULL);
    if (hiprandStatus != HIPRAND_STATUS_SUCCESS)
      {
        throw std::runtime_error("Setting CURAND seed failed");
      }

    // CUDPP for faster reductions than Thrust (hopefully!)
    addReduceCfg_.op = CUDPP_ADD;
    addReduceCfg_.algorithm = CUDPP_REDUCE;
    addReduceCfg_.datatype = CUDPP_FLOAT;
    addReduceCfg_.options = CUDPP_OPTION_FORWARD;

    cudppCreate(&cudpp_);
    CUDPPResult res = cudppPlan(cudpp_, &addReduce_, addReduceCfg_, popSize_, 1,
				0);
    if (res != CUDPP_SUCCESS)
      {
        std::stringstream msg;
        msg << "CUDPP initialization failed with error " << res;
        throw std::runtime_error(msg.str().c_str());
      }

    // CUDPP for faster min reducitons
    minReduceCfg_.op = CUDPP_MIN;
    minReduceCfg_.algorithm = CUDPP_REDUCE;
    minReduceCfg_.datatype = CUDPP_FLOAT;
    minReduceCfg_.options = CUDPP_OPTION_FORWARD;

#ifndef NDEBUG
    cerr << "ObsTime: " << obsTime_ << endl;
#endif

  }

  // Copy constructor
  GpuLikelihood::GpuLikelihood(const GpuLikelihood& other) :
    Likelihood(other), I1Time_(other.I1Time_), I1Idx_(other.I1Idx_),
    covariateCopies_(other.covariateCopies_), 
    devAnimals_(other.devAnimals_), 
    animalsPitch_(other.animalsPitch_), 
    devD_(other.devD_), 
    hostDRowPtr_(other.hostDRowPtr_), 
    dnnz_(other.dnnz_), 
    integralBuffSize_(other.integralBuffSize_), 
    cuRand_(other.cuRand_)
  {
    timeval start, end;
    gettimeofday(&start, NULL);
    // Allocate Animals_
    checkCudaError(
		   hipMallocPitch(&devAnimalsInfPow_, &animalsInfPowPitch_, maxInfecs_ * sizeof(float), numSpecies_));
    animalsInfPowPitch_ /= sizeof(float);
    checkCudaError(
		   hipMemcpy2D(devAnimalsInfPow_,animalsInfPowPitch_*sizeof(float),other.devAnimalsInfPow_,other.animalsInfPowPitch_*sizeof(float),maxInfecs_*sizeof(float),numSpecies_,hipMemcpyDeviceToDevice));

    checkCudaError(
		   hipMallocPitch(&devAnimalsSuscPow_, &animalsSuscPowPitch_, popSize_ * sizeof(float), numSpecies_));
    animalsSuscPowPitch_ /= sizeof(float);
    checkCudaError(
		   hipMemcpy2D(devAnimalsSuscPow_,animalsSuscPowPitch_*sizeof(float),other.devAnimalsSuscPow_,other.animalsSuscPowPitch_*sizeof(float),popSize_*sizeof(float),numSpecies_,hipMemcpyDeviceToDevice));

    // Allocate and copy event times - popSize_ * NUMEVENTS matrix
    checkCudaError(
		   hipMallocPitch(&devEventTimes_, &eventTimesPitch_, popSize_ * sizeof(float), NUMEVENTS));
    eventTimesPitch_ /= sizeof(float);
    checkCudaError(
		   hipMemcpy2D(devEventTimes_,eventTimesPitch_*sizeof(float),other.devEventTimes_,other.eventTimesPitch_*sizeof(float),popSize_*sizeof(float), NUMEVENTS, hipMemcpyDeviceToDevice));

    // Allocate and copy intermediate infectivity and susceptibility
    checkCudaError(hipMalloc(&devSusceptibility_, popSize_ * sizeof(float)));
    checkCudaError(
		   hipMemcpy(devSusceptibility_, other.devSusceptibility_, popSize_ * sizeof(float),hipMemcpyDeviceToDevice));
    checkCudaError(hipMalloc(&devInfectivity_, maxInfecs_ * sizeof(float)));
    checkCudaError(
		   hipMemcpy(devInfectivity_, other.devInfectivity_, maxInfecs_ * sizeof(float), hipMemcpyDeviceToDevice));

    // Infection index
    devInfecIdx_ = new thrust::device_vector<InfecIdx_t>(*other.devInfecIdx_);
    hostInfecIdx_ = new thrust::host_vector<InfecIdx_t>(*other.hostInfecIdx_);

    hostSuscOccults_ = new thrust::host_vector<InfecIdx_t>(*other.hostSuscOccults_);

    // Allocate and copy likelihood components;
    devProduct_ = new thrust::device_vector<float>(*(other.devProduct_));
    devWorkspace_ = new thrust::device_vector<float>;
    devWorkspace_->resize(other.devWorkspace_->size());
    checkCudaError(hipMalloc(&devScratch_, SCRATCHSIZE*sizeof(FP_t)));
    logLikelihood_ = other.logLikelihood_;

    checkCudaError(
		   hipHostAlloc(&hostComponents_, sizeof(LikelihoodComponents), hipHostMallocMapped));

    *hostComponents_ = *(other.hostComponents_);

    checkCudaError(
		   hipHostGetDevicePointer(&devComponents_, hostComponents_, 0));

    // Parameters -- device side
    checkCudaError(hipMalloc(&devXi_, numSpecies_ * sizeof(float)));
    checkCudaError(hipMalloc(&devPsi_, numSpecies_ * sizeof(float)));
    checkCudaError(hipMalloc(&devZeta_, numSpecies_ * sizeof(float)));
    checkCudaError(hipMalloc(&devPhi_, numSpecies_ * sizeof(float)));
    RefreshParameters();

    // BLAS handles
    blasStat_ = other.blasStat_;
    cudaBLAS_ = other.cudaBLAS_;
    sparseStat_ = other.sparseStat_;
    cudaSparse_ = other.cudaSparse_;
    crsDescr_ = other.crsDescr_;

    ++*covariateCopies_; // Increment copies of covariate data

    gettimeofday(&end, NULL);
    std::cerr << "Time (" << __PRETTY_FUNCTION__ << "): "
	      << timeinseconds(start, end) << std::endl;

    // CUDAPP bits
    // CUDPP for faster reductions than Thrust (hopefully!)
    addReduceCfg_ = other.addReduceCfg_;
    cudpp_ = other.cudpp_;
    addReduce_ = other.addReduce_;

    hipDeviceSynchronize();
    


  }

  GpuLikelihood*
  GpuLikelihood::clone() const
  {
    return new GpuLikelihood(*this);
  }

  // Assignment constructor
  const Likelihood&
  GpuLikelihood::assign(const Likelihood& rhs)
  {
    //  timeval start, end;
    //  gettimeofday(&start, NULL);
    const GpuLikelihood& other = static_cast<const GpuLikelihood&>(rhs);

    // Copy animal powers
    checkCudaError(
		   hipMemcpy2DAsync(devAnimalsInfPow_,animalsInfPowPitch_*sizeof(float),other.devAnimalsInfPow_,other.animalsInfPowPitch_*sizeof(float),maxInfecs_*sizeof(float),numSpecies_,hipMemcpyDeviceToDevice));
    checkCudaError(
		   hipMemcpy2DAsync(devAnimalsSuscPow_,animalsSuscPowPitch_*sizeof(float),other.devAnimalsSuscPow_,other.animalsSuscPowPitch_*sizeof(float),popSize_*sizeof(float),numSpecies_,hipMemcpyDeviceToDevice));

    // copy event times
    checkCudaError(
		   hipMemcpy2DAsync(devEventTimes_,eventTimesPitch_*sizeof(float),other.devEventTimes_,other.eventTimesPitch_*sizeof(float),popSize_*sizeof(float), NUMEVENTS, hipMemcpyDeviceToDevice));

    // copy intermediate infectivity and susceptibility
    checkCudaError(
		   hipMemcpyAsync(devSusceptibility_, other.devSusceptibility_, popSize_ * sizeof(float),hipMemcpyDeviceToDevice));
    checkCudaError(
		   hipMemcpyAsync(devInfectivity_, other.devInfectivity_, maxInfecs_ * sizeof(float), hipMemcpyDeviceToDevice));

    // Infection index
    *devInfecIdx_ = *(other.devInfecIdx_);
    *hostInfecIdx_ = *(other.hostInfecIdx_);

    // Internals
    I1Idx_ = other.I1Idx_;
    I1Time_ = other.I1Time_;
    *hostSuscOccults_ = *other.hostSuscOccults_;

    // Likelihood components
    // copy product vector
    *devProduct_ = *other.devProduct_;

    *hostComponents_ = *other.hostComponents_;
    logLikelihood_ = other.logLikelihood_;

    //  gettimeofday(&end, NULL);
    //  std::cerr << "Time (" << __PRETTY_FUNCTION__ << "): "
    //      << timeinseconds(start, end) << std::endl;

    RefreshParameters();

    hipDeviceSynchronize();
    return *this;
  }

  void
  GpuLikelihood::InfecCopy(const Likelihood& rhs)
  {
    const GpuLikelihood& other = static_cast<const GpuLikelihood&>(rhs);

    // copy event times
    checkCudaError(
		   hipMemcpy2DAsync(devEventTimes_,eventTimesPitch_*sizeof(float),other.devEventTimes_,other.eventTimesPitch_*sizeof(float),maxInfecs_*sizeof(float), NUMEVENTS, hipMemcpyDeviceToDevice));

    // Infection index
    *devInfecIdx_ = *other.devInfecIdx_;
    *hostInfecIdx_ = *other.hostInfecIdx_;

    // Internals
    I1Idx_ = other.I1Idx_;
    I1Time_ = other.I1Time_;
    *hostSuscOccults_ = *other.hostSuscOccults_;

    // copy product vector
    *devProduct_ = *other.devProduct_;

    // Likelihood components
    *hostComponents_ = *other.hostComponents_;
    logLikelihood_ = other.logLikelihood_;

    hipDeviceSynchronize();

  }

  GpuLikelihood::~GpuLikelihood()
  {

    // Destroy non-shared members first
    if (devEventTimes_)
      hipFree(devEventTimes_);
    if (devAnimalsSuscPow_)
      hipFree(devAnimalsSuscPow_);
    if (devAnimalsInfPow_)
      hipFree(devAnimalsInfPow_);
    if (devSusceptibility_)
      hipFree(devSusceptibility_);
    if (devInfectivity_)
      hipFree(devInfectivity_);

    if (devXi_)
      hipFree(devXi_);
    if (devPsi_)
      hipFree(devPsi_);
    if (devZeta_)
      hipFree(devZeta_);
    if (devPhi_)
      hipFree(devPhi_);

    if (hostComponents_)
      hipHostFree(hostComponents_);

    // Destroy Thrust bits
    if(hostInfecIdx_) delete hostInfecIdx_;
    if(devInfecIdx_) delete devInfecIdx_;
    if(hostSuscOccults_) delete hostSuscOccults_;
    if(devProduct_) delete devProduct_;
    if(devWorkspace_) delete devWorkspace_;

    // Choose whether to destroy shared members
    if (*covariateCopies_ == 1) // We're the last copy to be destroyed
      {
        hipFree(devAnimals_);
        destroyCsrMatrix(devD_);
        delete[] hostDRowPtr_;
        hipblasDestroy(cudaBLAS_);
        hipsparseDestroy(cudaSparse_);
        hiprandDestroyGenerator(cuRand_);
        delete covariateCopies_;

        cudppDestroyPlan(addReduce_);
        cudppDestroy(cudpp_);

	if(crsDescr_)
	  hipsparseDestroyMatDescr(crsDescr_);
	
	hipDeviceSynchronize();
	checkCudaError(hipDeviceReset());
      }
    else
      {
        --(*covariateCopies_);
      }


  }

  void
  GpuLikelihood::CalcDistanceMatrix(const float dLimit)
  {
    float2* coords = new float2[popSize_];
    Population::iterator it = population_.begin();
    for (size_t i = 0; i < popSize_; ++i)
      {
        coords[i] = make_float2((float) it->x, (float) it->y);
        it++;
      }

    devD_ = makeSparseDistance(coords, popSize_, dLimit);
    dnnz_ = devD_->nnz;

    cerr << "About to allocate hostDRowPtr" << endl;
    hostDRowPtr_ = new int[popSize_ + 1];
    cerr << "Allocated hostDRowPtr_ " << endl;
    checkCudaError(
		   hipMemcpy(hostDRowPtr_, devD_->rowPtr, (popSize_+1)*sizeof(int), hipMemcpyDeviceToHost));

    delete[] coords;
  }

  void
  GpuLikelihood::SetEvents()
  {

    // Set up Species and events
    float* eventsMatrix = new float[popSize_ * NUMEVENTS];
    Population::iterator it = population_.begin();
    for (size_t i = 0; i < popSize_; ++i)
      {
        eventsMatrix[i] = it->I;
        eventsMatrix[i + popSize_] = it->N;
        eventsMatrix[i + popSize_ * 2] = it->R;
        ++it;
      }

    // Allocate Event times - popSize_ * NUMEVENTS matrix
    checkCudaError(
		   hipMallocPitch(&devEventTimes_, &eventTimesPitch_, popSize_ * sizeof(float), NUMEVENTS));
    eventTimesPitch_ /= sizeof(float);

    // Get event times into GPU memory
    hipError_t rv = hipMemcpy2D(devEventTimes_,
				  eventTimesPitch_ * sizeof(float), eventsMatrix,
				  popSize_ * sizeof(float), popSize_ * sizeof(float), NUMEVENTS,
				  hipMemcpyHostToDevice);
    if (rv != hipSuccess)

      throw GpuRuntimeError("Copying event times to device failed", rv);

    // Set any event times greater than obsTime to obsTime
    int blocksPerGrid = (popSize_ + THREADSPERBLOCK - 1) / THREADSPERBLOCK;
    _sanitizeEventTimes<<<blocksPerGrid, THREADSPERBLOCK>>>(devEventTimes_, eventTimesPitch_, obsTime_, popSize_);
    checkCudaError(hipGetLastError());

    thrust::device_ptr<float> p(devEventTimes_);
    hostInfecIdx_->clear();
    for (size_t i = 0; i < numKnownInfecs_; ++i)
      {
        hostInfecIdx_->push_back(i);
      }
    *devInfecIdx_ = *hostInfecIdx_;

    delete[] eventsMatrix;
  }

  void
  GpuLikelihood::SetSpecies()
  {

    // Set up Species and events
    float* speciesMatrix = new float[popSize_ * numSpecies_];
    Population::const_iterator it = population_.begin();
    for (size_t i = 0; i < population_.size(); ++i)
      {
	speciesMatrix[i] = it->cattle;
	if(numSpecies_ > 1)
	  speciesMatrix[i + population_.size()] = it->pigs;
	if(numSpecies_ > 2)
	  speciesMatrix[i + population_.size() * 2] = it->sheep;
        ++it;
      }

    // Allocate Animals_
    checkCudaError(
		   hipMallocPitch(&devAnimals_, &animalsPitch_, popSize_ * sizeof(float), numSpecies_));
    animalsPitch_ /= sizeof(float);
    checkCudaError(
		   hipMallocPitch(&devAnimalsSuscPow_, &animalsSuscPowPitch_, popSize_ * sizeof(float), numSpecies_));
    animalsSuscPowPitch_ /= sizeof(float);
    checkCudaError(
		   hipMallocPitch(&devAnimalsInfPow_, &animalsInfPowPitch_, maxInfecs_ * sizeof(float), numSpecies_));
    animalsInfPowPitch_ /= sizeof(float);

    // Allocate intermediate infectivity and susceptibility
    checkCudaError(hipMalloc(&devSusceptibility_, popSize_ * sizeof(float)));
    checkCudaError(hipMalloc(&devInfectivity_, maxInfecs_ * sizeof(float)));

    hipError_t rv = hipMemcpy2D(devAnimals_, animalsPitch_ * sizeof(float),
				  speciesMatrix, popSize_ * sizeof(float), popSize_ * sizeof(float),
				  numSpecies_, hipMemcpyHostToDevice);
    if (rv != hipSuccess)
      throw GpuRuntimeError("Failed copying species data to device", rv);

    delete[] speciesMatrix;

  }

  void
  GpuLikelihood::SetDistance(const float* data, const int* rowptr,
			     const int* colind)
  {

    checkCudaError(hipMalloc(&devD_->val, dnnz_ * sizeof(float)));
    checkCudaError(hipMalloc(&devD_->rowPtr, (maxInfecs_ + 1) * sizeof(int)));
    checkCudaError(hipMalloc(&devD_->colInd, dnnz_ * sizeof(float)));

    hostDRowPtr_ = new int[maxInfecs_ + 1];

    checkCudaError(
		   hipMemcpy(devD_->val, data, dnnz_ * sizeof(float), hipMemcpyHostToDevice));
    checkCudaError(
		   hipMemcpy(devD_->rowPtr, rowptr, (maxInfecs_ + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaError(
		   hipMemcpy(devD_->colInd, colind, dnnz_ * sizeof(int), hipMemcpyHostToDevice));
    checkCudaError(
		   hipMemcpy(hostDRowPtr_, rowptr, (maxInfecs_ + 1)*sizeof(int), hipMemcpyHostToHost));
  }

  void
  GpuLikelihood::RefreshParameters()
  {

    float* tmp = new float[numSpecies_];

    for (size_t i = 0; i < numSpecies_; ++i)
      tmp[i] = xi_[i];
    checkCudaError(
		   hipMemcpy(devXi_, tmp, numSpecies_ * sizeof(float), hipMemcpyHostToDevice));

    for (size_t i = 0; i < numSpecies_; ++i)
      tmp[i] = psi_[i];
    checkCudaError(
		   hipMemcpy(devPsi_, tmp, numSpecies_ * sizeof(float), hipMemcpyHostToDevice));

    for (size_t i = 0; i < numSpecies_; ++i)
      tmp[i] = zeta_[i];
    checkCudaError(
		   hipMemcpy(devZeta_, tmp, numSpecies_ * sizeof(float), hipMemcpyHostToDevice));

    for (size_t i = 0; i < numSpecies_; ++i)
      tmp[i] = phi_[i];
    checkCudaError(
		   hipMemcpy(devPhi_, tmp, numSpecies_ * sizeof(float), hipMemcpyHostToDevice));

    delete[] tmp;
  }


  inline
  void
  GpuLikelihood::wrapDataParms(Data* data, ParmVals* p)
  {
    data->eventTimes = devEventTimes_;
    data->eventTimesPitch = eventTimesPitch_;
    data->infectivity = devInfectivity_;
    data->susceptibility = devSusceptibility_;
    data->movtBan = movtBan_;
    data->D = *devD_;
    
    p->epsilon1 = *epsilon1_;
    p->epsilon2 = *epsilon2_;
    p->gamma1 = *gamma1_;
    p->gamma2 = *gamma2_;
    p->delta = *delta_;
    p->omega = *omega_;
    p->nu = *nu_;
    p->alpha = *alpha_;
  }

  inline
  void
  GpuLikelihood::CalcInfectivityPow()
  {
    int dimBlock(THREADSPERBLOCK);
    int dimGrid((maxInfecs_ + THREADSPERBLOCK - 1) / THREADSPERBLOCK);
    _calcSpecPow<<<dimGrid, dimBlock>>>(maxInfecs_,numSpecies_,devAnimalsInfPow_, animalsInfPowPitch_,devAnimals_,animalsPitch_,devPsi_);
    checkCudaError(hipGetLastError());
				 
  }

  inline
  void
  GpuLikelihood::CalcInfectivity()
  {

    // Now calculate infectivity
    blasStat_ = hipblasSgemv(cudaBLAS_, HIPBLAS_OP_N, maxInfecs_, numSpecies_,
			    &UNITY, devAnimalsInfPow_, animalsInfPowPitch_, devXi_, 1, &ZERO,
			    devInfectivity_, 1);
    if (blasStat_ != HIPBLAS_STATUS_SUCCESS)
      {
        std::cerr << "Error in infectivity: " << blasStat_ << std::endl;
      }

  }

  inline
  void
  GpuLikelihood::CalcSusceptibilityPow()
  {
    int dimBlock(THREADSPERBLOCK);
    int dimGrid((popSize_ + THREADSPERBLOCK - 1) / THREADSPERBLOCK);
    _calcSpecPow<<<dimGrid, dimBlock>>>(popSize_,numSpecies_,devAnimalsSuscPow_,animalsSuscPowPitch_, devAnimals_,animalsPitch_,devPhi_);
    checkCudaError(hipGetLastError());

					   
  }

  inline
  void
  GpuLikelihood::CalcSusceptibility()
  {
    // Calculates susceptibility powers and sums over suscept.
    blasStat_ = hipblasSgemv(cudaBLAS_, HIPBLAS_OP_N, popSize_, numSpecies_,
			    &UNITY, devAnimalsSuscPow_, animalsSuscPowPitch_, devZeta_, 1, &ZERO,
			    devSusceptibility_, 1);
    if (blasStat_ != HIPBLAS_STATUS_SUCCESS)
      {
        std::cerr << "Error in susceptibility: " << blasStat_ << std::endl;
      }
  }

  inline
  void
  GpuLikelihood::UpdateI1()
  {
    thrust::device_vector<InfecIdx_t>::iterator myMin;
    myMin = thrust::min_element(devInfecIdx_->begin(), devInfecIdx_->end(),
				IndirectMin<float>(devEventTimes_));
    InfecIdx_t tmp = *myMin;
    I1Idx_ = tmp.ptr;

    thrust::device_ptr<float> v(devEventTimes_);
    I1Time_ = v[I1Idx_];
    hipDeviceSynchronize();
  }
  inline
  void
  GpuLikelihood::CalcBgIntegral()
  {
    int numBlocks = (popSize_ + THREADSPERBLOCK - 1) / THREADSPERBLOCK;
    _bgIntegral<<<numBlocks, THREADSPERBLOCK>>>(thrust::raw_pointer_cast(&(*devWorkspace_)[0]), devEventTimes_, popSize_, *epsilon1_, *epsilon2_, movtBan_, I1Time_);

    if(numBlocks > 1) {
      CUDPPResult res = cudppReduce(addReduce_, &devComponents_->bgIntegral,
				    thrust::raw_pointer_cast(&(*devWorkspace_)[0]), numBlocks);
      if(res != CUDPP_SUCCESS) throw logic_error("CUDPP failed");
    }
    else
      checkCudaError(hipMemcpy(&devComponents_->bgIntegral, thrust::raw_pointer_cast(&(*devWorkspace_)[0]), sizeof(float), hipMemcpyDeviceToDevice));

#ifndef NDEBUG
    hipDeviceSynchronize();
    if(hostComponents_->bgIntegral <= 0.0f) cerr << "bgIntegral = " << hostComponents_->bgIntegral << endl;
    assert(hostComponents_->bgIntegral >= 0.0f);
#endif
  }

  inline
  void
  GpuLikelihood::CalcProduct()
  {

    _calcProduct<<<integralBuffSize_,THREADSPERBLOCK>>>(thrust::raw_pointer_cast(&(*devInfecIdx_)[0]),devInfecIdx_->size(),*devD_,
							devEventTimes_,eventTimesPitch_,devSusceptibility_,devInfectivity_,*epsilon1_, *epsilon2_, *gamma1_,*gamma2_,*delta_,*omega_,*nu_, *alpha_, movtBan_, thrust::raw_pointer_cast(&(*devProduct_)[0]));
    checkCudaError(hipGetLastError());

    ReduceProductVector();
  }

  inline
  void
  GpuLikelihood::CalcIntegral()
  {
    int numRequiredThreads = devInfecIdx_->size() * 32; // One warp per infection
    int integralBuffSize = (numRequiredThreads + THREADSPERBLOCK - 1)
      / THREADSPERBLOCK;

    _calcIntegral<<<integralBuffSize,THREADSPERBLOCK>>>(thrust::raw_pointer_cast(&(*devInfecIdx_)[0]),devInfecIdx_->size(),*devD_,
							devEventTimes_,eventTimesPitch_,devSusceptibility_,devInfectivity_,*gamma2_,*delta_,*omega_,*nu_, *alpha_, thrust::raw_pointer_cast(&(*devWorkspace_)[0]));
    checkCudaError(hipGetLastError());

    hipDeviceSynchronize();
    if(integralBuffSize > 1) {
      CUDPPResult res = cudppReduce(addReduce_, &devComponents_->integral,
				    thrust::raw_pointer_cast(&(*devWorkspace_)[0]), integralBuffSize);
      if (res != CUDPP_SUCCESS)
        throw std::runtime_error(
				 "cudppReduce failed in GpuLikelihood::CalcIntegral()");
    }
    else checkCudaError(hipMemcpy(&devComponents_->integral, thrust::raw_pointer_cast(&(*devWorkspace_)[0]), sizeof(float), hipMemcpyDeviceToDevice));
  }

  void
  GpuLikelihood::FullCalculate()
  {

#ifdef GPUTIMING
    timeval start, end;
    gettimeofday(&start, NULL);
#endif

    RefreshParameters();
    CalcInfectivityPow();
    CalcInfectivity();
    CalcSusceptibilityPow();
    CalcSusceptibility();

    UpdateI1();
    CalcIntegral();
    CalcProduct();
    CalcBgIntegral();

    hipDeviceSynchronize();
    hostComponents_->integral *= *gamma1_;
    logLikelihood_ = hostComponents_->logProduct
      - (hostComponents_->integral + hostComponents_->bgIntegral);


#ifdef GPUTIMING
    gettimeofday(&end, NULL);
    std::cerr << "Time (" << __PRETTY_FUNCTION__ << "): "
	      << timeinseconds(start, end) << std::endl;
    std::cerr << "Likelihood (" << __PRETTY_FUNCTION__ << "): " << logLikelihood_
	      << std::endl;
#endif

#ifndef NDEBUG
    cerr << __FUNCTION__ << " (likelihood)\n";
    PrintLikelihoodComponents();
    PrintParameters();
    PrintEventTimes();
    cerr << endl;
#endif
  }

  void
  GpuLikelihood::Calculate()
  {
#ifdef GPUTIMING
    timeval start, end;
    gettimeofday(&start, NULL);
#endif

    RefreshParameters();
    CalcInfectivity();
    CalcSusceptibility();

    UpdateI1();
    CalcIntegral();
    CalcProduct();
    CalcBgIntegral();

    hipDeviceSynchronize();
    hostComponents_->integral *= *gamma1_;

    logLikelihood_ = hostComponents_->logProduct
      - (hostComponents_->integral + hostComponents_->bgIntegral);

#ifdef GPUTIMING
    gettimeofday(&end, NULL);
    std::cerr << "Time (" << __PRETTY_FUNCTION__ << "): "
	      << timeinseconds(start, end) << std::endl;
#endif

#ifndef NDEBUG
    cerr << __FUNCTION__ << " (likelihood)\n";
    PrintLikelihoodComponents();
    PrintParameters();
    PrintEventTimes();
    cerr << endl;
#endif
  }

  float
  GpuLikelihood::InfectionPart()
  {
    int blocksPerGrid = (GetNumKnownInfecs() + THREADSPERBLOCK - 1)
      / THREADSPERBLOCK;

    _knownInfectionsLikelihood<<<blocksPerGrid, THREADSPERBLOCK, THREADSPERBLOCK*sizeof(float)>>>(thrust::raw_pointer_cast(&(*devInfecIdx_)[0]),
												  GetNumKnownInfecs(), devEventTimes_, eventTimesPitch_, *a_, *b_, thrust::raw_pointer_cast(&(*devWorkspace_)[0]));
    checkCudaError(hipGetLastError());

    float loglikelihood = 0.0f;

    for (size_t i = GetNumKnownInfecs(); i < this->GetNumInfecs(); ++i)
      {
        float Ii, Ni;
        checkCudaError(
		       hipMemcpy(&Ii, devEventTimes_+(*hostInfecIdx_)[i].ptr, sizeof(float), hipMemcpyDeviceToHost));
        checkCudaError(
		       hipMemcpy(&Ni, devEventTimes_+eventTimesPitch_+(*hostInfecIdx_)[i].ptr, sizeof(float), hipMemcpyDeviceToHost));
        loglikelihood += log(
			     gsl_cdf_gamma_Q(Ni - Ii, (float) *a_, 1.0 / (float) *b_));
      }

    loglikelihood += thrust::reduce(devWorkspace_->begin(),
				    devWorkspace_->begin() + blocksPerGrid);

    return loglikelihood;
  }

  /*************************
   * Update Infection Time *
   *************************/
  __global__ void
  _updateInfectionTimePrepare(const int i, 
			      const float inTime,
			      const int I1Idx,
			      const GpuLikelihood::Data data,
			      const GpuLikelihood::ParmVals p,
			      float* devOldTime, 
			      float* prodVector,
			      GpuLikelihood::LikelihoodComponents* devComponents_)
  {
    // Calculate new time
    float newTime = data.eventTimes[i+data.eventTimesPitch] - inTime;
    float oldTime = data.eventTimes[i];
    
    // Work out if we're creating a new I1
    if(i==I1Idx or newTime < data.eventTimes[I1Idx]) {
      prodVector[I1Idx] = newTime < data.movtBan ? p.epsilon1 : (p.epsilon1 * p.epsilon2);
    }
    else { // Update background pressure IF NOT CHANGING I1.  Else we delegate
      //  to the UpdateI1() function.
      devComponents_->bgIntegral += p.epsilon1 * (min(data.movtBan,newTime) 
						  - min(data.movtBan,oldTime));
      devComponents_->bgIntegral += p.epsilon1 * p.epsilon2 * (max(data.movtBan,newTime) 
							       - max(data.movtBan,oldTime));
    }
    
    // Cache the old infection time -- used in subsequent calculations
    *devOldTime = oldTime;  // Save old time
    
    // Insert new time
    data.eventTimes[i] = data.eventTimes[i+data.eventTimesPitch] - inTime;
    
    // Zero product vector for movee
    prodVector[i] = 0.0f;
  }
  
  
  __global__ void
  _updateInfectionTimeIntegral(const unsigned int idx,
			       const InfecIdx_t* infecIdx, 
			       const GpuLikelihood::Data data,
			       const GpuLikelihood::ParmVals p,
			       const float* oldTime,
			       float* output)
  {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__
      float buff[];
    buff[threadIdx.x] = 0.0f;

    int i = infecIdx[idx].ptr;
    int begin = data.D.rowPtr[i];
    int end = data.D.rowPtr[i + 1];

    if (tid < end - begin)
      {
        int j = data.D.colInd[begin + tid];

        float Ii = *oldTime;
	float newTime = data.eventTimes[i];
        float Ni = data.eventTimes[i + data.eventTimesPitch];
        float Ri = data.eventTimes[i + data.eventTimesPitch * 2];

        float Ij = data.eventTimes[j];
        float Nj = data.eventTimes[j + data.eventTimesPitch];
        float Rj = data.eventTimes[j + data.eventTimesPitch * 2];

        float jOnIdx = 0.0f;
        if (Ij < Nj)
          {
            // Recalculate pressure from j on idx
            jOnIdx = _H(fminf(Nj, newTime) - fminf(Ij, newTime), p.nu, p.alpha)
	      + p.gamma2
	      * (_H(fminf(Rj, newTime) - Ij, p.nu, p.alpha)
		 - _H(fminf(Nj, newTime) - Ij, p.nu, p.alpha)); // New pressure
            jOnIdx -= _H(fminf(Nj, Ii) - fminf(Ii, Ij), p.nu, p.alpha)
	      + p.gamma2
	      * (_H(fminf(Rj, Ii) - Ij, p.nu, p.alpha)
		 - _H(fminf(Nj, Ii) - Ij, p.nu, p.alpha)); // Old pressure
	    // Apply infec and suscep
            jOnIdx *= data.susceptibility[i];
            jOnIdx *= data.infectivity[j];
          }

        // Recalculate pressure from idx on j
        float IdxOnj = _H(fminf(Ni, Ij) - fminf(newTime, Ij), p.nu, p.alpha);
        IdxOnj -= _H(fminf(Ni, Ij) - fminf(Ii, Ij), p.nu, p.alpha);
        IdxOnj += p.gamma2
	  * (_H(fminf(Ri, Ij) - newTime, p.nu, p.alpha)
	     - _H(fminf(Ni, Ij) - newTime, p.nu, p.alpha));
        IdxOnj -= p.gamma2
	  * (_H(fminf(Ri, Ij) - Ii, p.nu, p.alpha)
	     - _H(fminf(Ni, Ij) - Ii, p.nu, p.alpha));
        IdxOnj *= data.susceptibility[j];
        IdxOnj *= data.infectivity[i];

        buff[threadIdx.x] = (IdxOnj + jOnIdx)
	  * _K(data.D.val[begin+tid], p.delta,p.omega);

        // Reduce buffer into output
        _shmemReduce(buff);

      }

    if (threadIdx.x == 0)
      {
        output[blockIdx.x] = buff[0];
      }
  }

  __global__ void
  _updateInfectionTimeProduct(const unsigned int idx,
			      const InfecIdx_t* infecIdx,
			      const GpuLikelihood::Data data,
			      const GpuLikelihood::ParmVals p,
			      const int I1Idx,
			      const float* oldTime,
			      float* prodCache)
  {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    extern __shared__
      float buff[];
    buff[threadIdx.x] = 0.0f;

    int i = infecIdx[idx].ptr;

    int begin = data.D.rowPtr[i];
    int end = data.D.rowPtr[i + 1];
    float newTime = data.eventTimes[i];

    if (tid < end - begin) // Massive amount of wasted time just here!
      {
        int j = data.D.colInd[begin + tid];

        float Ij = data.eventTimes[j];
        float Nj = data.eventTimes[j + data.eventTimesPitch];

        if (Ij < Nj)
          {
            float Ii = *oldTime;
            float Ni = data.eventTimes[i + data.eventTimesPitch];
            float Ri = data.eventTimes[i + data.eventTimesPitch * 2];
            float Rj = data.eventTimes[j + data.eventTimesPitch * 2];

            // Adjust product cache from idx on others
            float idxOnj = 0.0f;
            if (Ii < Ij and Ij <= Ni)
              idxOnj -= _h(Ij - Ii, p.nu, p.alpha);
            else if (Ni < Ij and Ij <= Ri)
              {
                idxOnj -= p.gamma2 * _h(Ij - Ii, p.nu, p.alpha);
                idxOnj += p.gamma2 * _h(Ij - newTime, p.nu, p.alpha);
              }
            if (newTime < Ij and Ij <= Ni)
              idxOnj += _h(Ij - newTime, p.nu, p.alpha);

            idxOnj *= p.gamma1 * data.infectivity[i] * data.susceptibility[j]
	      * _K(data.D.val[begin+tid], p.delta, p.omega);
            prodCache[j] += idxOnj;

            // Recalculate instantaneous pressure on idx
            float jOnIdx = 0.0f;
            if (Ij < newTime and newTime <= Nj)
              jOnIdx = _h(newTime - Ij, p.nu, p.alpha);
            else if (Nj < newTime and newTime <= Rj)
              jOnIdx = p.gamma2 * _h(newTime - Ij, p.nu, p.alpha);

            jOnIdx *= data.susceptibility[i] * data.infectivity[j]
	      * _K(data.D.val[begin+tid],p.delta, p.omega);
            buff[threadIdx.x] = jOnIdx * p.gamma1;

          }

        _shmemReduce(buff);

        if (threadIdx.x == 0)
          _atomicAdd(prodCache + i, buff[0]); // Maybe better to create an external reduction buffer here.
        if (tid == 0) {
          float epsilon = newTime < data.movtBan ? p.epsilon1 : (p.epsilon1 * p.epsilon2);
          _atomicAdd(prodCache + i, epsilon);
        }
      }
  }

  void
  GpuLikelihood::UpdateInfectionTime(const unsigned int idx, const float inTime)
  {
    // Require to know number of cols per row -- probably store in host mem.
    // Also, may be optimal to use a much lower THREADSPERBLOCK than the app-wide setting.

#ifdef GPUTIMING
    timeval start, end;
    gettimeofday(&start, NULL);
#endif

#ifndef NDEBUG
    float oldTime = eventTimesPtr[i];
    cerr << "Moving idx " << idx << " from " <<  oldTime << " to " << newTime << endl;
#endif

    // Pack data and parameters (Delegate to constructor!)
    Data data;
    ParmVals p;
    wrapDataParms(&data, &p);

    if (idx >= hostInfecIdx_->size())
      throw std::range_error(
			     "Invalid idx in GpuLikelihood::UpdateInfectionTime");

    // Save likelihood components
    float savedIntegral = hostComponents_->integral;

    int i = (*hostInfecIdx_)[idx].ptr;

    float newTime = population_[i].N - inTime; // Relies on hostPopulation.N *NOT* being changed!

    bool haveNewI1 = false;
    if (newTime < I1Time_ or i == I1Idx_)
      {
        haveNewI1 = true;
      }

    _updateInfectionTimePrepare<<<1,1>>>(i, inTime,I1Idx_, data, p,
					 devScratch_,
					 thrust::raw_pointer_cast(&(*devProduct_)[0]),
					 devComponents_);

    int blocksPerGrid = (hostDRowPtr_[i + 1] - hostDRowPtr_[i] + THREADSPERBLOCK
			 - 1) / THREADSPERBLOCK + 1;



    // Integrated infection pressure
    _updateInfectionTimeIntegral<<<blocksPerGrid, THREADSPERBLOCK, THREADSPERBLOCK*sizeof(float)>>>
      (idx, thrust::raw_pointer_cast(&(*devInfecIdx_)[0]), data, p, devScratch_,
       thrust::raw_pointer_cast(&(*devWorkspace_)[0]));
    checkCudaError(hipGetLastError());
    if(blocksPerGrid > 1) {
      CUDPPResult res = cudppReduce(addReduce_, &devComponents_->integral,
				    thrust::raw_pointer_cast(&(*devWorkspace_)[0]),
				    blocksPerGrid);
      if (res != CUDPP_SUCCESS)
	throw std::runtime_error(
				 "cudppReduce failed in GpuLikelihood::UpdateInfectionTime()");
    }
    else 
      {
	checkCudaError(hipMemcpy(&devComponents_->integral, 
				  thrust::raw_pointer_cast(&(*devWorkspace_)[0]), 
				  sizeof(float), hipMemcpyDeviceToDevice));
      }
    
    _updateInfectionTimeProduct<<<blocksPerGrid, THREADSPERBLOCK, THREADSPERBLOCK*sizeof(float)>>>
      (idx, T2P(*devInfecIdx_),data, p, I1Idx_, devScratch_, T2P(*devProduct_));
    checkCudaError(hipGetLastError());

    if (haveNewI1)
      {
        UpdateI1();
        CalcBgIntegral();
#ifndef NDEBUG
        std::cerr << "New I1" << std::endl;
#endif
      }

    ReduceProductVector();

    // Collect results and update likelihood
    hipDeviceSynchronize();
    hostComponents_->integral = savedIntegral
      + hostComponents_->integral * *gamma1_;

    logLikelihood_ = hostComponents_->logProduct
      - (hostComponents_->integral + hostComponents_->bgIntegral);

#ifdef GPUTIMING
    gettimeofday(&end, NULL);
    std::cerr << "Time (" << __PRETTY_FUNCTION__ << "): "
	      << timeinseconds(start, end) << std::endl;
    std::cerr.precision(20);
    std::cerr << "Likelihood (" << __PRETTY_FUNCTION__ << "): " << logLikelihood_
	      << std::endl;
    std::cerr << "I1: " << I1Idx_ << " at " << I1Time_ << std::endl;
#endif

#ifndef NDEBUG
    cerr << __FUNCTION__ << " (likelihood)\n";
    PrintLikelihoodComponents();
    PrintParameters();
    PrintEventTimes();
    cerr << endl;
#endif
  }


  /**********************
   * Add infection Time *
   **********************/

    __global__ void
  _addInfectionTimePrepare(const int i, 
			   const float inTime,
			   const int I1Idx,
			   const GpuLikelihood::Data data,
			   const GpuLikelihood::ParmVals p,
			   float* devOldTime, 
			   float* prodVector,
			   GpuLikelihood::LikelihoodComponents* devComponents_)
  {
    // Calculate new time
    float newTime = data.eventTimes[i+data.eventTimesPitch] - inTime;
    float oldTime = data.eventTimes[i];
    
    // Work out if we're creating a new I1
    if(i==I1Idx or newTime < data.eventTimes[I1Idx]) {
      prodVector[I1Idx] = newTime < data.movtBan ? p.epsilon1 : (p.epsilon1 * p.epsilon2);
    }
    else { // Update background pressure IF NOT CHANGING I1.  Else we delegate
      //  to the UpdateI1() function.
      devComponents_->bgIntegral += p.epsilon1 * (min(data.movtBan,newTime) 
						  - min(data.movtBan,oldTime));
      devComponents_->bgIntegral += p.epsilon1 * p.epsilon2 * (max(data.movtBan,newTime) 
							       - max(data.movtBan,oldTime));
    }
    
    // Cache the old infection time -- used in subsequent calculations
    *devOldTime = oldTime;  // Save old time
    
    // Insert new time
    data.eventTimes[i] = data.eventTimes[i+data.eventTimesPitch] - inTime;
    
    // Zero product vector for movee
    prodVector[i] = 0.0f;
  }


  __global__ void
  _addInfectionTimeIntegral(const unsigned int idx,
			    const InfecIdx_t* infecIdx,
			    const GpuLikelihood::Data data,
			    const GpuLikelihood::ParmVals p,
			    const float* oldTime,
			    float* output)
  {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__
      float buff[];
    buff[threadIdx.x] = 0.0f;

    int i = infecIdx[idx].ptr;
    int begin = data.D.rowPtr[i];
    int end = data.D.rowPtr[i + 1];

    if (tid < end - begin)
      {
        int j = data.D.colInd[begin + tid];

        float Ii = *oldTime;
	float newTime = data.eventTimes[i];
        float Ni = data.eventTimes[i + data.eventTimesPitch];
        float Ri = data.eventTimes[i + data.eventTimesPitch * 2];

        float Ij = data.eventTimes[j];
        float Nj = data.eventTimes[j + data.eventTimesPitch];
        float Rj = data.eventTimes[j + data.eventTimesPitch * 2];

        float jOnIdx = 0.0f;
        if (Ij < Nj)
          {
            // Calculate pressure from j on idx
            jOnIdx -= _H(fminf(Nj, Ii) - fminf(Ij, Ii), p.nu, p.alpha);
            jOnIdx -= p.gamma2
	      * (_H(fminf(Rj, Ii) - Ij, p.nu, p.alpha)
		 - _H(fminf(Nj, Ii) - Ij, p.nu, p.alpha));
            jOnIdx += _H(fminf(Nj, newTime) - fminf(Ij, newTime), p.nu, p.alpha);
            jOnIdx += p.gamma2
	      * (_H(fminf(Rj, newTime) - Ij, p.nu, p.alpha)
		 - _H(fminf(Nj, newTime) - Ij, p.nu, p.alpha));

            // Apply infec and suscep
            jOnIdx *= data.susceptibility[i];
            jOnIdx *= data.infectivity[j];
          }

        // Add pressure from idx on j
        float IdxOnj = _H(fminf(Ni, Ij) - fminf(newTime, Ij), p.nu, p.alpha);
        IdxOnj += p.gamma2
	  * (_H(fminf(Ri, Ij) - newTime, p.nu, p.alpha)
	     - _H(fminf(Ni, Ij) - newTime, p.nu, p.alpha));
        IdxOnj *= data.susceptibility[j];
        IdxOnj *= data.infectivity[i];

        buff[threadIdx.x] = (IdxOnj + jOnIdx)
	  * _K(data.D.val[begin+tid], p.delta, p.omega);

        // Reduce buffer into output
        _shmemReduce(buff);
      }

    if (threadIdx.x == 0)
      {
        output[blockIdx.x] = buff[0];
      }
  }

    __global__ void
  _addInfectionTimeProduct(const unsigned int idx,
			   const InfecIdx_t* infecIdx,
			   const GpuLikelihood::Data data,
			   const GpuLikelihood::ParmVals p,
			   const int I1Idx,
			   float* prodCache)
  {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    extern __shared__
      float buff[];
    buff[threadIdx.x] = 0.0f;

    int i = infecIdx[idx].ptr;
    float newTime = data.eventTimes[i];
    
    int begin = data.D.rowPtr[i];
    int end = data.D.rowPtr[i + 1];

    if (tid < end - begin) // Massive amount of wasted time just here!
      {
        int j = data.D.colInd[begin + tid];

        float Ij = data.eventTimes[j];
        float Nj = data.eventTimes[j + data.eventTimesPitch];

        if (Ij < Nj)
          { // Only look at infected individuals
            float Ni = data.eventTimes[i + data.eventTimesPitch];
            float Ri = data.eventTimes[i + data.eventTimesPitch * 2];
            float Rj = data.eventTimes[j + data.eventTimesPitch * 2];

            // Adjust product cache from idx on others
            float idxOnj = 0.0f;
            if (newTime < Ij and Ij <= Ni)
              idxOnj += _h(Ij - newTime, p.nu, p.alpha);
            else if (Ni < Ij and Ij <= Ri)
              idxOnj += p.gamma2 * _h(Ij - newTime, p.nu, p.alpha);

            idxOnj *= p.gamma1 * data.infectivity[i] * data.susceptibility[j]
			  * _K(data.D.val[begin+tid], p.delta, p.omega);
            prodCache[j] += idxOnj;

            // Calculate instantaneous pressure on idx
            float jOnIdx = 0.0f;
            if (Ij < newTime and newTime <= Nj)
              jOnIdx = _h(newTime - Ij, p.nu, p.alpha);
            else if (Nj < newTime and newTime <= Rj)
              jOnIdx = p.gamma2 * _h(newTime - Ij, p.nu, p.alpha);

            jOnIdx *= p.gamma1 * data.infectivity[j] * data.susceptibility[i]
			  * _K(data.D.val[begin+tid], p.delta, p.omega);

            buff[threadIdx.x] = jOnIdx;

          }

        _shmemReduce(buff);

        if (threadIdx.x == 0)
          _atomicAdd(prodCache + i, buff[0]);
        if (tid == 0) { // Add background pressure, or turn to 1.0f for I1
          float epsilon = newTime < data.movtBan ? p.epsilon1 : (p.epsilon1*p.epsilon2);
          _atomicAdd(prodCache + i, epsilon);
        }
      }
  }
  
  void
  GpuLikelihood::AddInfectionTime(const unsigned int idx, const float inTime)
  {
    // idx is the position in the hostSuscOccults vector (ie the idx'th occult)
    // inTime is the proposed Ni - Ii time

#ifdef GPUTIMING
    timeval start, end;
    gettimeofday(&start, NULL);
#endif

    Data data;
    ParmVals p;
    wrapDataParms(&data, &p);

    if (idx >= hostSuscOccults_->size())
      throw std::range_error("Invalid idx in GpuLikelihood::AddInfectionTime");

    // Save likelihood components
    float savedIntegral = hostComponents_->integral;

    unsigned int i = (*hostSuscOccults_)[idx].ptr;

    float Ni = population_[i].N;
    float newTime = Ni - inTime;

    _updateInfectionTimePrepare<<<1,1>>>(i, inTime, I1Idx_, data, p, devScratch_,
					 T2P(*devProduct_),
					 devComponents_);
    
    // Update the indices
    devInfecIdx_->push_back(i);
    hostInfecIdx_->push_back(i);
    hostSuscOccults_->erase(hostSuscOccults_->begin() + idx);

    bool haveNewI1 = false;
    if(newTime < I1Time_) {
      haveNewI1 = true;
    }

    unsigned int addIdx = devInfecIdx_->size() - 1;

    int blocksPerGrid = (hostDRowPtr_[i + 1] - hostDRowPtr_[i] + THREADSPERBLOCK
			 - 1) / THREADSPERBLOCK + 1;
    _addInfectionTimeIntegral<<<blocksPerGrid, THREADSPERBLOCK, THREADSPERBLOCK*sizeof(float)>>>
      (addIdx, T2P(*devInfecIdx_), data, p, devScratch_, T2P(*devWorkspace_));
    checkCudaError(hipGetLastError());
    if(blocksPerGrid > 1) {
      CUDPPResult res = cudppReduce(addReduce_, &devComponents_->integral,
				    T2P(*devWorkspace_),
				    blocksPerGrid);
      if (res != CUDPP_SUCCESS)
	throw std::runtime_error(
				 "cudppReduce failed in GpuLikelihood::UpdateInfectionTime()");
    }
    else {
      checkCudaError(hipMemcpy(&devComponents_->integral,
				thrust::raw_pointer_cast(&(*devWorkspace_)[0]),
				sizeof(float), hipMemcpyDeviceToDevice));
    }
    
    _addInfectionTimeProduct<<<blocksPerGrid, THREADSPERBLOCK, THREADSPERBLOCK*sizeof(float)>>>
      (addIdx, T2P(*devInfecIdx_), data, p, I1Idx_, T2P(*devProduct_));
    checkCudaError(hipGetLastError());

    if (haveNewI1)
      {
        UpdateI1();
        CalcBgIntegral();
#ifndef NDEBUG
        std::cerr << "New I1" << std::endl;
#endif
      }

    ReduceProductVector();

    // Collect results and update likelihood
    hipDeviceSynchronize();
    hostComponents_->integral = savedIntegral
      + hostComponents_->integral * *gamma1_;

    logLikelihood_ = hostComponents_->logProduct
      - (hostComponents_->integral + hostComponents_->bgIntegral);

#ifdef GPUTIMING
    gettimeofday(&end, NULL);
    std::cerr << "Time (" << __PRETTY_FUNCTION__ << "): "
	      << timeinseconds(start, end) << std::endl;
#endif

#ifndef NDEBUG
    cerr << __FUNCTION__ << " (likelihood)\n";
    PrintLikelihoodComponents();
    PrintParameters();
    PrintEventTimes();
    cerr << endl;
#endif
  }



  /*************************
   * Delete infection time *
   *************************/

    __global__ void
  _delInfectionTimePrepare(const int i, 
			   const int I1Idx,
			   const GpuLikelihood::Data data,
			   const GpuLikelihood::ParmVals p,
			   float* devOldTime, 
			   float* prodVector,
			   GpuLikelihood::LikelihoodComponents* devComponents_)
  {
    // Calculate new time
    float oldTime = data.eventTimes[i];
    float newTime = data.eventTimes[i+data.eventTimesPitch];
    
    // Work out if we're creating a new I1
    if(i!=I1Idx and newTime > data.eventTimes[I1Idx]) {
      // Update background pressure IF NOT CHANGING I1.  Else we delegate
      //  to the UpdateI1() function.
      devComponents_->bgIntegral += p.epsilon1 * (min(data.movtBan,newTime) 
						  - min(data.movtBan,oldTime));
      devComponents_->bgIntegral += p.epsilon1 * p.epsilon2 * (max(data.movtBan,newTime) 
							       - max(data.movtBan,oldTime));
    }
    
    // Cache the old infection time -- used in subsequent calculations
    *devOldTime = oldTime;  // Save old time
    
    // Insert new time
    data.eventTimes[i] = newTime;
    
    // Zero product vector for movee
    prodVector[i] = 1.0f;
  }

  
  __global__ void
  _delInfectionTimeIntegral(const unsigned int idx, const InfecIdx_t* infecIdx,
			    GpuLikelihood::Data data,
			    GpuLikelihood::ParmVals p,
			    const float* oldTime,
			    float* output)
  {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__
      float buff[];
    buff[threadIdx.x] = 0.0f;

    int i = infecIdx[idx].ptr;
    int begin = data.D.rowPtr[i];
    int end = data.D.rowPtr[i + 1];

    if (tid < end - begin)
      {
        int j = data.D.colInd[begin + tid];

        float Ii = *oldTime;
        float Ni = data.eventTimes[i + data.eventTimesPitch];
        float Ri = data.eventTimes[i + data.eventTimesPitch * 2];

        float Ij = data.eventTimes[j];
        float Nj = data.eventTimes[j + data.eventTimesPitch];
        float Rj = data.eventTimes[j + data.eventTimesPitch * 2];

        float jOnIdx = 0.0f;
        if (Ij < Nj)
          {
            // Recalculate pressure from j on idx
            jOnIdx -= _H(fminf(Nj, Ii) - fminf(Ii, Ij), p.nu, p.alpha)
	      + p.gamma2
	      * (_H(fminf(Rj, Ii) - Ij, p.nu, p.alpha)
		 - _H(fminf(Nj, Ii) - Ij, p.nu, p.alpha)); // Old pressure
            jOnIdx += _H(fminf(Nj, Ni) - fminf(Ij, Ni), p.nu, p.alpha)
	      + p.gamma2
	      * (_H(fminf(Rj, Ni) - Ij, p.nu, p.alpha)
		 - _H(fminf(Nj, Ni) - Ij, p.nu, p.alpha)); // New pressure
	    // Apply infec and suscep
            jOnIdx *= data.susceptibility[i];
            jOnIdx *= data.infectivity[j];
          }

        // Subtract pressure from idx on j
        float IdxOnj = 0.0f;
        IdxOnj -= _H(fminf(Ni, Ij) - fminf(Ii, Ij), p.nu, p.alpha);
        IdxOnj -= p.gamma2
	  * (_H(fminf(Ri, Ij) - Ii, p.nu, p.alpha)
	     - _H(fminf(Ni, Ij) - Ii, p.nu, p.alpha));
        IdxOnj *= data.susceptibility[j];
        IdxOnj *= data.infectivity[i];

        buff[threadIdx.x] = (IdxOnj + jOnIdx)
	  * _K(data.D.val[begin+tid], p.delta, p.omega);

        // Reduce buffer into output
        _shmemReduce(buff);

      }

    if (threadIdx.x == 0)
      {
        output[blockIdx.x] = buff[0];
      }
  }



  __global__ void
  _delInfectionTimeProduct(const unsigned int idx, const InfecIdx_t* infecIdx,
			   GpuLikelihood::Data data,
			   GpuLikelihood::ParmVals p,
			   const float* oldTime,
			   float* prodCache)
  {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    int i = infecIdx[idx].ptr;

    int begin = data.D.rowPtr[i];
    int end = data.D.rowPtr[i + 1];

    if (tid < end - begin) // Massive amount of wasted time just here!
      {
        int j = data.D.colInd[begin + tid];

        float Ij = data.eventTimes[j];
        float Nj = data.eventTimes[j + data.eventTimesPitch];

        if (Ij < Nj)
          {

            float Ii = *oldTime;
            float Ni = data.eventTimes[i + data.eventTimesPitch];
            float Ri = data.eventTimes[i + data.eventTimesPitch * 2];

            // Adjust product cache from idx on others
            float idxOnj = 0.0;
            if (Ii < Ij and Ij <= Ni)
              idxOnj -= _h(Ij - Ii, p.nu, p.alpha);
            else if (Ni < Ij and Ij <= Ri)
              idxOnj -= p.gamma2 * _h(Ij - Ii, p.nu, p.alpha);

            idxOnj *= p.gamma1 * data.infectivity[i] * data.susceptibility[j]
			  * _K(data.D.val[begin+tid],p.delta,p.omega);
            prodCache[j] += idxOnj;
          }
      }
  }


  void
  GpuLikelihood::DeleteInfectionTime(const unsigned int idx)
  {
    // Delete the idx'th occult ( = idx+numKnownInfecs_ infective)

#ifdef GPUTIMING
    timeval start, end;
    gettimeofday(&start, NULL);
#endif

    Data data;
    ParmVals p;
    wrapDataParms(&data, &p);

    // Range check
    if (idx >= devInfecIdx_->size() - numKnownInfecs_)
      throw std::range_error(
			     "Invalid idx in GpuLikelihood::DeleteInfectionTime");

    // Save likelihood components
    float savedIntegral = hostComponents_->integral;

    // Identify occult to delete
    unsigned int ii = idx + numKnownInfecs_;
    unsigned int i = (*hostInfecIdx_)[ii].ptr;

    _delInfectionTimePrepare<<<1,1>>>(i, I1Idx_, data, p, devScratch_,
				      T2P(*devProduct_), devComponents_);
    
    //thrust::device_ptr<float> eventTimesPtr(devEventTimes_);

    float notification = population_[i].N;
    //float oldI = eventTimesPtr[i];

    int blocksPerGrid = (hostDRowPtr_[i + 1] - hostDRowPtr_[i] + THREADSPERBLOCK
			 - 1) / THREADSPERBLOCK + 1;
    _delInfectionTimeIntegral<<<blocksPerGrid, THREADSPERBLOCK, THREADSPERBLOCK*sizeof(float)>>>
      (ii, T2P(*devInfecIdx_), data, p, devScratch_, T2P(*devWorkspace_));
    checkCudaError(hipGetLastError());
    if(blocksPerGrid > 1) {
      CUDPPResult res = cudppReduce(addReduce_, &devComponents_->integral,
				    thrust::raw_pointer_cast(&(*devWorkspace_)[0]), blocksPerGrid);
      if (res != CUDPP_SUCCESS)
	throw std::runtime_error(
				 "cudppReduce failed in GpuLikelihood::UpdateInfectionTime()");
    }
    else {
      checkCudaError(hipMemcpy(&devComponents_->integral, thrust::raw_pointer_cast(&(*devWorkspace_)[0]), sizeof(float), hipMemcpyDeviceToDevice));
#ifndef NDEBUG
      cerr << __FUNCTION__ << ": blocksPerGrid = " << blocksPerGrid << endl;
#endif
    }
    
    _delInfectionTimeProduct<<<blocksPerGrid, THREADSPERBLOCK, THREADSPERBLOCK*sizeof(float)>>>
      (ii, T2P(*devInfecIdx_), data, p, devScratch_, T2P(*devProduct_));
    checkCudaError(hipGetLastError());

    // Make the change to the population
    devInfecIdx_->erase(devInfecIdx_->begin() + ii);
    hostInfecIdx_->erase(hostInfecIdx_->begin() + ii);
    hostSuscOccults_->push_back(i);
    //eventTimesPtr[i] = notification;
    //(*devProduct_)[i] = 1.0f;

    if (i == I1Idx_)
      {
        UpdateI1();
        CalcBgIntegral();
#ifndef NDEBUG
        std::cerr << "New I1" << std::endl;
#endif
      }

    ReduceProductVector();

    // Collect results and update likelihood
    hipDeviceSynchronize();
    hostComponents_->integral = savedIntegral
      + hostComponents_->integral * *gamma1_;

    logLikelihood_ = hostComponents_->logProduct
      - (hostComponents_->integral + hostComponents_->bgIntegral);

#ifdef GPUTIMING
    gettimeofday(&end, NULL);
    std::cerr << "Time (" << __PRETTY_FUNCTION__ << "): "
	      << timeinseconds(start, end) << std::endl;
    std::cerr.precision(20);
    std::cerr << "Likelihood (" << __PRETTY_FUNCTION__ << "): " << logLikelihood_
	      << std::endl;
#endif

#ifndef NDEBUG
    cerr << __FUNCTION__ << " (likelihood)\n";
    PrintLikelihoodComponents();
    PrintParameters();
    PrintEventTimes();
    cerr << endl;
#endif

  }

  float
  GpuLikelihood::GetIN(const size_t index)
  {
    int i = (*hostInfecIdx_)[index].ptr;
    //thrust::device_vector<float> res(1);
    thrust::device_ptr<float> et(devEventTimes_);
    //thrust::transform(et + eventTimesPitch_ + i, et + eventTimesPitch_ + i + 1,
    //    et + i, &res[0], thrust::minus<float>());

    return et[eventTimesPitch_+i] - et[i];

    //return res[0];
  }

  float
  GpuLikelihood::GetLogLikelihood() const
  {
    return logLikelihood_;
  }

  float
  GpuLikelihood::GetN(const int idx) const
  {
    float rv;
    checkCudaError(
		   hipMemcpy(devEventTimes_+idx+eventTimesPitch_,&rv,sizeof(float), hipMemcpyDeviceToHost));
    return rv;
  }

  void
  GpuLikelihood::LazyAddInfecTime(const int idx, const float inTime)
  {
    thrust::device_ptr<float> eventTimePtr(devEventTimes_);
    eventTimePtr[idx] = eventTimePtr[idx + eventTimesPitch_] - inTime;
    devInfecIdx_->push_back(idx);
    devProduct_->push_back(0.0f);
    hipDeviceSynchronize();
  }

  void
  GpuLikelihood::GetSumInfectivityPow(float* result) const
  {
    for (size_t k = 0; k < numSpecies_; ++k)
      {
        result[k] = indirectedSum(thrust::raw_pointer_cast(&(*devInfecIdx_)[0]),
				  numKnownInfecs_, devAnimalsInfPow_ + animalsInfPowPitch_ * k);
      }
  }

  void
  GpuLikelihood::GetSumSusceptibilityPow(float* result) const
  {
    for (size_t k = 0; k < numSpecies_; ++k)
      {
        thrust::device_ptr<float> p(
				    devAnimalsSuscPow_ + animalsSuscPowPitch_ * k);
        result[k] = thrust::reduce(p, p + popSize_);
      }
  }

  float
  GpuLikelihood::GetMeanI2N() const
  {
    thrust::device_ptr<float> p(devEventTimes_);
    thrust::device_vector<float> buff(numKnownInfecs_);
    thrust::transform(p + eventTimesPitch_,
		      p + eventTimesPitch_ + numKnownInfecs_, p, buff.begin(),
		      thrust::minus<float>());
    return thrust::reduce(buff.begin(), buff.end()) / (float) numKnownInfecs_;
  }

  float
  GpuLikelihood::GetMeanOccI() const
  {
    size_t numOccs = GetNumOccults();
    if (numOccs == 0)
      return 0.0f;
    else
      {
        float sumI = indirectedSum(
				   thrust::raw_pointer_cast(&(*devInfecIdx_)[numKnownInfecs_]),
				   GetNumOccults(), devEventTimes_);
        float sumN = indirectedSum(
				   thrust::raw_pointer_cast(&(*devInfecIdx_)[numKnownInfecs_]),
				   GetNumOccults(), devEventTimes_ + eventTimesPitch_);
        return (sumN - sumI) / GetNumOccults();
      }
  }

  float
  GpuLikelihood::NonCentreInfecTimes(const float oldGamma, const float newGamma,
				     const float prob)
  {

    // Generate random numbers
    thrust::device_vector<float> seeds(GetNumKnownInfecs());
    hiprandStatus_t status = hiprandGenerateUniform(cuRand_,
						  thrust::raw_pointer_cast(&seeds[0]), GetNumKnownInfecs());
    if (status != HIPRAND_STATUS_SUCCESS)
      {
        throw std::runtime_error("hiprandGenerateUniform failed");
      }

    float logLikDiff = 0.0f;

    int dimGrid((GetNumKnownInfecs() + THREADSPERBLOCK - 1) / THREADSPERBLOCK);

    // Update the infection times
    _nonCentreInfecTimes<<<dimGrid, THREADSPERBLOCK>>>(thrust::raw_pointer_cast(&(*devInfecIdx_)[0]), GetNumKnownInfecs(), devEventTimes_, eventTimesPitch_, oldGamma/newGamma, thrust::raw_pointer_cast(&seeds[0]), prob);

    // Do known bit -- GPU in parallel with CPU
    _knownInfectionsLikelihoodPNC<<<dimGrid, THREADSPERBLOCK, THREADSPERBLOCK*sizeof(float)>>>(raw_pointer_cast(&(*devInfecIdx_)[0]), GetNumKnownInfecs(), devEventTimes_, eventTimesPitch_, *a_, oldGamma, newGamma,
											       thrust::raw_pointer_cast(&seeds[0]), prob, thrust::raw_pointer_cast(&(*devWorkspace_)[0]));
    checkCudaError(hipGetLastError());

    // Todo: GPU-ise this!
    for (size_t i = GetNumKnownInfecs(); i < GetNumInfecs(); ++i)
      {
        float Ii, Ni;
        checkCudaError(
		       hipMemcpyAsync(&Ii, devEventTimes_+(*hostInfecIdx_)[i].ptr, sizeof(float), hipMemcpyDeviceToHost));
        checkCudaError(
		       hipMemcpyAsync(&Ni, devEventTimes_+eventTimesPitch_+(*hostInfecIdx_)[i].ptr, sizeof(float), hipMemcpyDeviceToHost));
        hipDeviceSynchronize();
        logLikDiff += logf(gsl_cdf_gamma_Q(Ni - Ii, *a_, 1.0 / newGamma))
	  - logf(gsl_cdf_gamma_Q(Ni - Ii, *a_, 1.0 / oldGamma));
      }

    logLikDiff += thrust::reduce(devWorkspace_->begin(),
				 devWorkspace_->begin() + dimGrid);


    return logLikDiff;
  }

  void
  GpuLikelihood::GetInfectiousPeriods(std::vector<EpiRisk::IPTuple_t>& periods)
  {
    periods.resize(GetNumInfecs());

    thrust::device_vector<float> devOutputVec(GetNumInfecs());
    int blocksPerGrid((GetNumInfecs() + THREADSPERBLOCK - 1) / THREADSPERBLOCK);
    _collectInfectiousPeriods<<<blocksPerGrid, THREADSPERBLOCK>>>(thrust::raw_pointer_cast(&(*devInfecIdx_)[0]),
								  GetNumInfecs(),
								  devEventTimes_,
								  eventTimesPitch_,
								  thrust::raw_pointer_cast(&devOutputVec[0]));

    thrust::host_vector<float> outputVec(GetNumInfecs());
    outputVec = devOutputVec;
    for (size_t i = 0; i < GetNumInfecs(); ++i)
      {
        periods[i].idx = (*hostInfecIdx_)[i].ptr;
        periods[i].val = outputVec[i];
      }
  }

  void
  GpuLikelihood::PrintLikelihoodComponents() const
  {
    hipDeviceSynchronize();
    cerr << "Background: " << hostComponents_->bgIntegral << "\n";
    cerr << "Integral: " << hostComponents_->integral << "\n";
    cerr << "Product: " << hostComponents_->logProduct << "\n";
  }

  void GpuLikelihood::PrintParameters() const
  {
    cerr << "Epsilon1,2: " << *epsilon1_ << ", " << *epsilon2_ << "\n";
    cerr << "Gamma1,2: " << *gamma1_ << ", " << *gamma2_ << "\n";
    cerr << "Delta: " << *delta_ << "\n";
    cerr << "Omega: " << *omega_ << "\n";
    for(int i = 0; i<numSpecies_; ++i) cerr << "Xi,Zeta,Phi,Psi[" << i << "]: " << xi_[i] << ", " << zeta_[i] << ", " << phi_[i] << ", " << psi_[i] << "\n";
    cerr << "alpha: " << *alpha_ << "\n";
    cerr << "a: " << *a_ << "\n";
    cerr << "b: " << *b_ << endl;
    cerr << "ObsTime: " << obsTime_ << "\n";
    cerr << "I1Idx = " << I1Idx_ << "\n";
    cerr << "I1Time = " << I1Time_ << "\n";
  }

  void
  GpuLikelihood::PrintEventTimes() const
  {
   
    // Obtain event times from GPU
    hipDeviceSynchronize();
    float *events = new float[numKnownInfecs_*NUMEVENTS];
    checkCudaError(hipMemcpy2D(events, numKnownInfecs_*sizeof(float), devEventTimes_, eventTimesPitch_*sizeof(float), numKnownInfecs_*sizeof(float), NUMEVENTS, hipMemcpyDeviceToHost));

    hipDeviceSynchronize();
    std::vector<std::string> ids;
    GetIds(ids);
    cerr << "===EVENTS===\n";
    for(int i = 0; i<numKnownInfecs_; ++i)
      cerr << ids[i] << "\t" << events[i] << "\t" << events[i+numKnownInfecs_] << "\t" << events[i+2*numKnownInfecs_] << "\n";
    cerr << "============" << endl;

    delete[] events;
  }
  
  void
  GpuLikelihood::PrintDistMatrix() const
  {
    cerr << "======DIST MATRIX======";

    // Copy distance matrix to host
    CsrMatrix *myCSR = new CsrMatrix;
    *myCSR = *devD_;

    myCSR->rowPtr = new int[myCSR->n];
    myCSR->colInd = new int[myCSR->nnz];
    myCSR->val = new float[myCSR->nnz];

    checkCudaError(hipMemcpy(myCSR->rowPtr, devD_->rowPtr, myCSR->n*sizeof(int), hipMemcpyDeviceToHost));
    checkCudaError(hipMemcpy(myCSR->colInd, devD_->colInd, myCSR->nnz*sizeof(int), hipMemcpyDeviceToHost));
    checkCudaError(hipMemcpy(myCSR->val, devD_->val, myCSR->nnz*sizeof(int), hipMemcpyDeviceToHost));

    for(int i=0; i<numKnownInfecs_; ++i) {
      for(int j=0; j<numKnownInfecs_; ++j) {
	cerr << GetDistElement(myCSR, i, j) << "\t";
      }
      cerr << "\n";
    }

    cerr << "=======================";

    delete[] myCSR->rowPtr;
    delete[] myCSR->colInd;
    delete[] myCSR->val;
    delete myCSR;
  }



} // namespace EpiRisk

